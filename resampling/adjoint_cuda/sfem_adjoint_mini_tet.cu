#include "hip/hip_runtime.h"
#include <stdio.h>

#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>
#include <thrust/extrema.h>

#include "sfem_adjoint_mini_loc_tet.cuh"
#include "sfem_adjoint_mini_tet.cuh"
#include "sfem_resample_field_cuda_fun.cuh"

extern "C" void                                                                         //
call_sfem_adjoint_mini_tet_kernel_gpu(const ptrdiff_t             start_element,        // Mesh
                                      const ptrdiff_t             end_element,          //
                                      const ptrdiff_t             nelements,            //
                                      const ptrdiff_t             nnodes,               //
                                      const idx_t** const         elems,                //
                                      const geom_t** const        xyz,                  //
                                      const ptrdiff_t             n0,                   // SDF
                                      const ptrdiff_t             n1,                   //
                                      const ptrdiff_t             n2,                   //
                                      const ptrdiff_t             stride0,              // Stride
                                      const ptrdiff_t             stride1,              //
                                      const ptrdiff_t             stride2,              //
                                      const geom_t                origin0,              // Origin
                                      const geom_t                origin1,              //
                                      const geom_t                origin2,              //
                                      const geom_t                dx,                   // Delta
                                      const geom_t                dy,                   //
                                      const geom_t                dz,                   //
                                      const real_t* const         weighted_field,       // Input weighted field
                                      const mini_tet_parameters_t mini_tet_parameters,  // Threshold for alpha
                                      real_t* const               data) {
    //

    hipStream_t cuda_stream_alloc = NULL;  // default stream
    hipStreamCreate(&cuda_stream_alloc);

    real_t* data_device           = NULL;
    real_t* weighted_field_device = NULL;

    hipMallocAsync((void**)&data_device, (n0 * n1 * n2) * sizeof(real_t), cuda_stream_alloc);
    hipMallocAsync((void**)&weighted_field_device, nnodes * sizeof(real_t), cuda_stream_alloc);

    elems_tet4_device elements_device = make_elems_tet4_device();
    cuda_allocate_elems_tet4_device_async(&elements_device, nelements, cuda_stream_alloc);

    xyz_tet4_device xyz_device = make_xyz_tet4_device();
    cuda_allocate_xyz_tet4_device_async(&xyz_device, nnodes, cuda_stream_alloc);

    hipStreamSynchronize(cuda_stream_alloc);  /// Ensure allocations are done before proceeding further with copies

    hipMemcpyAsync((void*)weighted_field_device,
                    (void*)weighted_field,
                    nnodes * sizeof(real_t),
                    hipMemcpyHostToDevice,
                    cuda_stream_alloc);

    hipMemset((void*)data_device, 0, (n0 * n1 * n2) * sizeof(real_t));

    copy_elems_tet4_device_async(elems, nelements, &elements_device, cuda_stream_alloc);

    copy_xyz_tet4_device_async(xyz, nnodes, &xyz_device, cuda_stream_alloc);

    hipStreamSynchronize(cuda_stream_alloc);

    // Optional: check for errors
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA error: %s, at file:%s:%d \n", hipGetErrorString(error), __FILE__, __LINE__);
    }

    const unsigned int threads_per_block      = 256;
    const unsigned int total_threads_per_grid = (end_element - start_element + 1) * LANES_PER_TILE;
    const unsigned int blocks_per_grid        = (total_threads_per_grid + threads_per_block - 1) / threads_per_block;

    hipStream_t hip_stream = 0;  // default stream
    hipStreamCreate(&hip_stream);

#if SFEM_LOG_LEVEL >= 5
    printf("Kernel args: start_element: %ld, end_element: %ld, nelements: %ld, nnodes: %ld\n",
           start_element,
           end_element,
           nelements,
           nnodes);
    printf("Kernel launch: blocks_per_grid: %u, threads_per_block: %u, total_threads_per_grid: %u\n",
           blocks_per_grid,
           threads_per_block,
           total_threads_per_grid);
#endif

    hipEvent_t start_event, stop_event;
    hipEventCreate(&start_event);
    hipEventCreate(&stop_event);

    hipEventRecord(start_event, hip_stream);

    sfem_adjoint_mini_tet_kernel_gpu<real_t><<<blocks_per_grid,                       //
                                               threads_per_block,                     //
                                               0,                                     //
                                               hip_stream>>>(start_element,          // Mesh
                                                              end_element,            //
                                                              nnodes,                 //
                                                              elements_device,        //
                                                              xyz_device,             //
                                                              n0,                     // SDF
                                                              n1,                     //
                                                              n2,                     //
                                                              stride0,                // Stride
                                                              stride1,                //
                                                              stride2,                //
                                                              origin0,                // Origin
                                                              origin1,                //
                                                              origin2,                //
                                                              dx,                     // Delta
                                                              dy,                     //
                                                              dz,                     //
                                                              weighted_field_device,  // Input weighted field
                                                              mini_tet_parameters,    // Threshold for alpha
                                                              data_device);           //

    hipStreamSynchronize(hip_stream);

    // Optional: check for errors
    error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA error: %s, at file:%s:%d \n", hipGetErrorString(error), __FILE__, __LINE__);
    }

    hipEventRecord(stop_event, hip_stream);
    hipEventSynchronize(stop_event);

    float milliseconds = 0.0f;
    hipEventElapsedTime(&milliseconds, start_event, stop_event);

    if (SFEM_LOG_LEVEL >= 5) {
        printf("================= SFEM Adjoint Mini-Tet Kernel GPU ================\n");
        printf("Kernel execution time: %f ms\n", milliseconds);
        printf("===================================================================\n");
    }

    hipEventDestroy(start_event);
    hipEventDestroy(stop_event);

    hipStreamDestroy(hip_stream);

    hipMemcpy((void*)data, (void*)data_device, (n0 * n1 * n2) * sizeof(real_t), hipMemcpyDeviceToHost);

    hipFreeAsync((void*)weighted_field_device, cuda_stream_alloc);

    free_xyz_tet4_device_async(&xyz_device, cuda_stream_alloc);

    free_elems_tet4_device_async(&elements_device, cuda_stream_alloc);

    hipFreeAsync(data_device, cuda_stream_alloc);
    hipStreamDestroy(cuda_stream_alloc);

}  // END: call_sfem_adjoint_mini_tet_kernel_gpu
// ////////////////////////////////////////////////////////////////////////////////////////////////
// ////////////////////////////////////////////////////////////////////////////////////////////////
// ////////////////////////////////////////////////////////////////////////////////////////////////

// ////////////////////////////////////////////////////////////////////////////////////////////////
// ////////////////////////////////////////////////////////////////////////////////////////////////
// ////////////////////////////////////////////////////////////////////////////////////////////////
extern "C" void                                                                                     //
call_sfem_adjoint_mini_tet_shared_info_kernel_gpu(const ptrdiff_t             start_element,        // Mesh
                                                  const ptrdiff_t             end_element,          //
                                                  const ptrdiff_t             nelements,            //
                                                  const ptrdiff_t             nnodes,               //
                                                  const idx_t** const         elems,                //
                                                  const geom_t** const        xyz,                  //
                                                  const ptrdiff_t             n0,                   // SDF
                                                  const ptrdiff_t             n1,                   //
                                                  const ptrdiff_t             n2,                   //
                                                  const ptrdiff_t             stride0,              // Stride
                                                  const ptrdiff_t             stride1,              //
                                                  const ptrdiff_t             stride2,              //
                                                  const geom_t                origin0,              // Origin
                                                  const geom_t                origin1,              //
                                                  const geom_t                origin2,              //
                                                  const geom_t                dx,                   // Delta
                                                  const geom_t                dy,                   //
                                                  const geom_t                dz,                   //
                                                  const real_t* const         weighted_field,       // Input weighted field
                                                  const mini_tet_parameters_t mini_tet_parameters,  // Threshold for alpha
                                                  real_t* const               data) {
    //

    hipStream_t cuda_stream_alloc = NULL;  // default stream
    hipStreamCreate(&cuda_stream_alloc);

    tet_properties_info_t<real_t> tet_properties_info;
    tet_properties_info.alloc_async(nelements, cuda_stream_alloc);

    real_t* data_device           = NULL;
    real_t* weighted_field_device = NULL;

    hipMallocAsync((void**)&data_device, (n0 * n1 * n2) * sizeof(real_t), cuda_stream_alloc);
    hipMallocAsync((void**)&weighted_field_device, nnodes * sizeof(real_t), cuda_stream_alloc);

    elems_tet4_device elements_device = make_elems_tet4_device();
    cuda_allocate_elems_tet4_device_async(&elements_device, nelements, cuda_stream_alloc);

    xyz_tet4_device xyz_device = make_xyz_tet4_device();
    cuda_allocate_xyz_tet4_device_async(&xyz_device, nnodes, cuda_stream_alloc);

    hipStreamSynchronize(cuda_stream_alloc);  /// Ensure allocations are done before proceeding further with copies

    hipMemcpyAsync((void*)weighted_field_device,
                    (void*)weighted_field,
                    nnodes * sizeof(real_t),
                    hipMemcpyHostToDevice,
                    cuda_stream_alloc);

    hipMemset((void*)data_device, 0, (n0 * n1 * n2) * sizeof(real_t));

    copy_elems_tet4_device_async(elems, nelements, &elements_device, cuda_stream_alloc);

    copy_xyz_tet4_device_async(xyz, nnodes, &xyz_device, cuda_stream_alloc);

    hipStreamSynchronize(cuda_stream_alloc);

    // Optional: check for errors
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA error: %s, at file:%s:%d \n", hipGetErrorString(error), __FILE__, __LINE__);
    }

    const unsigned int tets_per_block = 8;
    hipStream_t       hip_stream    = 0;  // default stream
    hipStreamCreate(&hip_stream);

    // hipMemset((void*)tet_properties_info.total_size_local, 7777700087766, nelements * sizeof(ptrdiff_t));///////////

    hipEvent_t start_event, stop_event;
    hipEventCreate(&start_event);
    hipEventCreate(&stop_event);

    hipEventRecord(start_event, hip_stream);

    {  // BEGIN: Compute local grid sizes for each element
        const unsigned int threads_per_block           = LANES_PER_TILE * tets_per_block;
        const unsigned int total_threads_per_grid_prop = (end_element - start_element + 1);
        const unsigned int blocks_per_grid = (total_threads_per_grid_prop + threads_per_block - 1) / threads_per_block;

        sfem_make_local_data_tets_kernel_gpu<real_t><<<blocks_per_grid,                      //
                                                       threads_per_block,                    //
                                                       0,                                    //
                                                       hip_stream>>>(start_element,         // Mesh
                                                                      end_element,           //
                                                                      nnodes,                //
                                                                      elements_device,       //
                                                                      xyz_device,            //
                                                                      n0,                    // SDF
                                                                      n1,                    //
                                                                      n2,                    //
                                                                      stride0,               // Stride
                                                                      stride1,               //
                                                                      stride2,               //
                                                                      origin0,               // Origin
                                                                      origin1,               //
                                                                      origin2,               //
                                                                      dx,                    // Delta
                                                                      dy,                    //
                                                                      dz,                    //
                                                                      tet_properties_info);  //
    }  // END: Compute local grid sizes for each element

    hipStreamSynchronize(hip_stream);

    // Optional: check for errors
    error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA error: %s, at file:%s:%d \n", hipGetErrorString(error), __FILE__, __LINE__);
    }

    ptrdiff_t max_total_size_local = -1;
    ptrdiff_t max_idx_global       = -1;

    ptrdiff_t min_total_size_local = -1;
    ptrdiff_t min_idx_global       = -1;

    {  // Find max and min total_size_local across all elements
        const ptrdiff_t count = (end_element - start_element);

        auto d_begin = thrust::device_pointer_cast(tet_properties_info.total_size_local) + start_element;
        auto d_end   = d_begin + count;

        auto max_it          = thrust::max_element(d_begin, d_end);
        max_total_size_local = *max_it;
        max_idx_global       = (max_it - d_begin) + start_element;

        // auto min_it          = thrust::min_element(d_begin, d_end);
        // min_total_size_local = *min_it;

        // min_idx_global = (min_it - d_begin) + start_element;
    }

    {
        const ptrdiff_t shared_memory_size = max_total_size_local * tets_per_block;

        const unsigned int threads_per_block      = LANES_PER_TILE * tets_per_block;
        const unsigned int total_threads_per_grid = (end_element - start_element + 1) * LANES_PER_TILE;
        const unsigned int blocks_per_grid        = (total_threads_per_grid + threads_per_block - 1) / threads_per_block;

        sfem_adjoint_mini_tet_shared_loc_kernel_gpu<real_t><<<blocks_per_grid,                       //
                                                              threads_per_block,                     //
                                                              shared_memory_size,                    //
                                                              hip_stream>>>(shared_memory_size,     //
                                                                             start_element,          // Mesh
                                                                             end_element,            //
                                                                             nnodes,                 //
                                                                             elements_device,        //
                                                                             xyz_device,             //
                                                                             n0,                     // SDF
                                                                             n1,                     //
                                                                             n2,                     //
                                                                             stride0,                // Stride
                                                                             stride1,                //
                                                                             stride2,                //
                                                                             origin0,                // Origin
                                                                             origin1,                //
                                                                             origin2,                //
                                                                             dx,                     // Delta
                                                                             dy,                     //
                                                                             dz,                     //
                                                                             weighted_field_device,  // Input weighted field
                                                                             mini_tet_parameters,    // Threshold for alpha
                                                                             data_device);           //
    }

    hipEventRecord(stop_event, hip_stream);
    hipEventSynchronize(stop_event);

    float milliseconds = 0.0f;
    hipEventElapsedTime(&milliseconds, start_event, stop_event);

    if (SFEM_LOG_LEVEL >= 1) {
        printf("================= SFEM Adjoint Mini-Tet Kernel GPU ================\n");
        printf("Kernel execution time: %f ms\n", milliseconds);
        printf("===================================================================\n");

        printf("  Max total_size_local = %lld\n", (long long)max_total_size_local);
        printf("  Max idx global       = %lld\n", (long long)max_idx_global);

        printf("  Min total_size_local = %lld\n", (long long)min_total_size_local);
        printf("  Min idx global       = %lld\n", (long long)min_idx_global);
        printf("===================================================================\n");
    }

    hipEventDestroy(start_event);
    hipEventDestroy(stop_event);

    hipStreamDestroy(hip_stream);

    hipMemcpy((void*)data, (void*)data_device, (n0 * n1 * n2) * sizeof(real_t), hipMemcpyDeviceToHost);

    hipFreeAsync((void*)weighted_field_device, cuda_stream_alloc);

    free_xyz_tet4_device_async(&xyz_device, cuda_stream_alloc);

    free_elems_tet4_device_async(&elements_device, cuda_stream_alloc);

    hipFreeAsync(data_device, cuda_stream_alloc);
    hipStreamDestroy(cuda_stream_alloc);
}
