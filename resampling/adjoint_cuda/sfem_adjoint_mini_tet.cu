#include <stdio.h>

#include "sfem_adjoint_mini_tet.cuh"
#include "sfem_resample_field_cuda_fun.cuh"

extern "C" void                                                                         //
call_sfem_adjoint_mini_tet_kernel_gpu(const ptrdiff_t             start_element,        // Mesh
                                      const ptrdiff_t             end_element,          //
                                      const ptrdiff_t             nelements,            //
                                      const ptrdiff_t             nnodes,               //
                                      const idx_t** const         elems,                //
                                      const geom_t** const        xyz,                  //
                                      const ptrdiff_t             n0,                   // SDF
                                      const ptrdiff_t             n1,                   //
                                      const ptrdiff_t             n2,                   //
                                      const ptrdiff_t             stride0,              // Stride
                                      const ptrdiff_t             stride1,              //
                                      const ptrdiff_t             stride2,              //
                                      const geom_t                origin0,              // Origin
                                      const geom_t                origin1,              //
                                      const geom_t                origin2,              //
                                      const geom_t                dx,                   // Delta
                                      const geom_t                dy,                   //
                                      const geom_t                dz,                   //
                                      const real_t* const         weighted_field,       // Input weighted field
                                      const mini_tet_parameters_t mini_tet_parameters,  // Threshold for alpha
                                      real_t* const               data) {
    //

    hipStream_t cuda_stream_alloc = NULL;  // default stream
    hipStreamCreate(&cuda_stream_alloc);

    real_t* data_device           = NULL;
    real_t* weighted_field_device = NULL;

    hipMallocAsync((void**)&data_device, (n0 * n1 * n2) * sizeof(real_t), cuda_stream_alloc);
    hipMallocAsync((void**)&weighted_field_device, nnodes * sizeof(real_t), cuda_stream_alloc);

    elems_tet4_device elements_device = make_elems_tet4_device();
    cuda_allocate_elems_tet4_device_async(&elements_device, nelements, cuda_stream_alloc);

    xyz_tet4_device xyz_device = make_xyz_tet4_device();
    cuda_allocate_xyz_tet4_device_async(&xyz_device, nnodes, cuda_stream_alloc);

    hipStreamSynchronize(cuda_stream_alloc);  /// Ensure allocations are done before proceeding further with copies

    hipMemcpyAsync((void*)weighted_field_device,
                    (void*)weighted_field,
                    nnodes * sizeof(real_t),
                    hipMemcpyHostToDevice,
                    cuda_stream_alloc);

    hipMemset((void*)data_device, 0, (n0 * n1 * n2) * sizeof(real_t));

    copy_elems_tet4_device_async(elems, nelements, &elements_device, cuda_stream_alloc);

    copy_xyz_tet4_device_async(xyz, nnodes, &xyz_device, cuda_stream_alloc);

    hipStreamSynchronize(cuda_stream_alloc);

    // Optional: check for errors
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA error: %s, at file:%s:%d \n", hipGetErrorString(error), __FILE__, __LINE__);
    }

    const unsigned int threads_per_block      = 256;
    const unsigned int total_threads_per_grid = (end_element - start_element + 1) * LANES_PER_TILE;
    const unsigned int blocks_per_grid        = (total_threads_per_grid + threads_per_block - 1) / threads_per_block;

    hipStream_t hip_stream = 0;  // default stream
    hipStreamCreate(&hip_stream);

#if SFEM_LOG_LEVEL >= 5
    printf("Kernel args: start_element: %ld, end_element: %ld, nelements: %ld, nnodes: %ld\n",
           start_element,
           end_element,
           nelements,
           nnodes);
    printf("Kernel launch: blocks_per_grid: %u, threads_per_block: %u, total_threads_per_grid: %u\n",
           blocks_per_grid,
           threads_per_block,
           total_threads_per_grid);
#endif

    hipEvent_t start_event, stop_event;
    hipEventCreate(&start_event);
    hipEventCreate(&stop_event);

    hipEventRecord(start_event, hip_stream);

    sfem_adjoint_mini_tet_kernel_gpu<real_t><<<blocks_per_grid,                       //
                                               threads_per_block,                     //
                                               0,                                     //
                                               hip_stream>>>(start_element,          // Mesh
                                                              end_element,            //
                                                              nnodes,                 //
                                                              elements_device,        //
                                                              xyz_device,             //
                                                              n0,                     // SDF
                                                              n1,                     //
                                                              n2,                     //
                                                              stride0,                // Stride
                                                              stride1,                //
                                                              stride2,                //
                                                              origin0,                // Origin
                                                              origin1,                //
                                                              origin2,                //
                                                              dx,                     // Delta
                                                              dy,                     //
                                                              dz,                     //
                                                              weighted_field_device,  // Input weighted field
                                                              mini_tet_parameters,    // Threshold for alpha
                                                              data_device);           //

    hipStreamSynchronize(hip_stream);

    // Optional: check for errors
    error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA error: %s, at file:%s:%d \n", hipGetErrorString(error), __FILE__, __LINE__);
    }

    hipEventRecord(stop_event, hip_stream);
    hipEventSynchronize(stop_event);

    float milliseconds = 0.0f;
    hipEventElapsedTime(&milliseconds, start_event, stop_event);

    if (SFEM_LOG_LEVEL >= 1) {
        printf("================= SFEM Adjoint Mini-Tet Kernel GPU ================\n");
        printf("Kernel execution time: %f ms\n", milliseconds);
        printf("===================================================================\n");
    }

    hipEventDestroy(start_event);
    hipEventDestroy(stop_event);

    hipStreamDestroy(hip_stream);

    hipMemcpy((void*)data, (void*)data_device, (n0 * n1 * n2) * sizeof(real_t), hipMemcpyDeviceToHost);

    hipFreeAsync((void*)weighted_field_device, cuda_stream_alloc);

    free_xyz_tet4_device_async(&xyz_device, cuda_stream_alloc);

    free_elems_tet4_device_async(&elements_device, cuda_stream_alloc);

    hipFreeAsync(data_device, cuda_stream_alloc);
    hipStreamDestroy(cuda_stream_alloc);

}  // END: call_sfem_adjoint_mini_tet_kernel_gpu
   // ////////////////////////////////////////////////////////////////////////////////////////////////
   // ////////////////////////////////////////////////////////////////////////////////////////////////
   // ////////////////////////////////////////////////////////////////////////////////////////////////
