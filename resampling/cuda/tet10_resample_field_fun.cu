#include <sfem_base.h>
#include <stdio.h>

// #define real_type real_t

#include "quadratures_rule_cuda.cuh"
#include "tet10_resample_field.cuh"

//////////////////////////////////////////////////////////
// make_xyz_tet10_device
//////////////////////////////////////////////////////////
xyz_tet10_device make_xyz_tet10_device(const ptrdiff_t nnodes) {  //
    //
    xyz_tet10_device xyz;

    hipMalloc(&xyz.x, nnodes * sizeof(geom_t));
    hipMalloc(&xyz.y, nnodes * sizeof(geom_t));
    hipMalloc(&xyz.z, nnodes * sizeof(geom_t));
    return xyz;
}
// end make_xyz_tet10_device

xyz_tet10_device                                                            //
make_xyz_tet10_device_async(const ptrdiff_t nnodes, hipStream_t stream) {  //
    //
    xyz_tet10_device xyz;

    hipMallocAsync(&xyz.x, nnodes * sizeof(geom_t), stream);
    hipMallocAsync(&xyz.y, nnodes * sizeof(geom_t), stream);
    hipMallocAsync(&xyz.z, nnodes * sizeof(geom_t), stream);
    return xyz;
}  // end make_xyz_tet10_device_async

//////////////////////////////////////////////////////////
// copy_xyz_tet10_device
//////////////////////////////////////////////////////////
void copy_xyz_tet10_device(const ptrdiff_t   nnodes,   //
                           xyz_tet10_device* xyz,      //
                           const geom_t**    xyz_host) {  //

    hipError_t err0 = hipMemcpy(xyz->x, xyz_host[0], nnodes * sizeof(idx_t), hipMemcpyHostToDevice);
    hipError_t err1 = hipMemcpy(xyz->y, xyz_host[1], nnodes * sizeof(idx_t), hipMemcpyHostToDevice);
    hipError_t err2 = hipMemcpy(xyz->z, xyz_host[2], nnodes * sizeof(idx_t), hipMemcpyHostToDevice);

    if (err0 != hipSuccess || err1 != hipSuccess || err2 != hipSuccess) {
        printf("ERROR: copying xyz_tet10_device to device: %s\n", hipGetErrorString(err0));
        // Handle the error or exit the program
    }
}  // end copy_xyz_tet10_device

//////////////////////////////////////////////////////////
// free_xyz_tet10_device
//////////////////////////////////////////////////////////
void free_xyz_tet10_device(xyz_tet10_device* xyz) {
    hipFree(xyz->x);
    hipFree(xyz->y);
    hipFree(xyz->z);

    xyz->x = NULL;
    xyz->y = NULL;
    xyz->z = NULL;
}
// end free_xyz_tet10_device

//////////////////////////////////////////////////////////
// make_elems_tet10_device
//////////////////////////////////////////////////////////
elems_tet10_device                                     //
make_elems_tet10_managed(const ptrdiff_t nelements) {  //
    //
    elems_tet10_device elems;

    hipError_t err0 = hipMallocManaged(&elems.elems_v0, nelements * sizeof(idx_t));
    hipError_t err1 = hipMallocManaged(&elems.elems_v1, nelements * sizeof(idx_t));
    hipError_t err2 = hipMallocManaged(&elems.elems_v2, nelements * sizeof(idx_t));
    hipError_t err3 = hipMallocManaged(&elems.elems_v3, nelements * sizeof(idx_t));
    hipError_t err4 = hipMallocManaged(&elems.elems_v4, nelements * sizeof(idx_t));
    hipError_t err5 = hipMallocManaged(&elems.elems_v5, nelements * sizeof(idx_t));
    hipError_t err6 = hipMallocManaged(&elems.elems_v6, nelements * sizeof(idx_t));
    hipError_t err7 = hipMallocManaged(&elems.elems_v7, nelements * sizeof(idx_t));
    hipError_t err8 = hipMallocManaged(&elems.elems_v8, nelements * sizeof(idx_t));
    hipError_t err9 = hipMallocManaged(&elems.elems_v9, nelements * sizeof(idx_t));

    if (err0 != hipSuccess || err1 != hipSuccess || err2 != hipSuccess || err3 != hipSuccess || err4 != hipSuccess ||
        err5 != hipSuccess || err6 != hipSuccess || err7 != hipSuccess || err8 != hipSuccess || err9 != hipSuccess) {
        printf("ERROR: allocating memory for elems_tet10_device managed at %s:%d\n", __FILE__, __LINE__);
        // Handle error
    }

    return elems;
}

hipError_t                                              //
copy_elems_tet10_managed(const ptrdiff_t     nelements,  //
                         elems_tet10_device* elems,      //
                         const idx_t**       elems_host) {     //
                                                         //
    hipError_t err0 = hipMemcpy(elems->elems_v0, elems_host[0], nelements * sizeof(idx_t), hipMemcpyHostToDevice);
    hipError_t err1 = hipMemcpy(elems->elems_v1, elems_host[1], nelements * sizeof(idx_t), hipMemcpyHostToDevice);
    hipError_t err2 = hipMemcpy(elems->elems_v2, elems_host[2], nelements * sizeof(idx_t), hipMemcpyHostToDevice);
    hipError_t err3 = hipMemcpy(elems->elems_v3, elems_host[3], nelements * sizeof(idx_t), hipMemcpyHostToDevice);
    hipError_t err4 = hipMemcpy(elems->elems_v4, elems_host[4], nelements * sizeof(idx_t), hipMemcpyHostToDevice);
    hipError_t err5 = hipMemcpy(elems->elems_v5, elems_host[5], nelements * sizeof(idx_t), hipMemcpyHostToDevice);
    hipError_t err6 = hipMemcpy(elems->elems_v6, elems_host[6], nelements * sizeof(idx_t), hipMemcpyHostToDevice);
    hipError_t err7 = hipMemcpy(elems->elems_v7, elems_host[7], nelements * sizeof(idx_t), hipMemcpyHostToDevice);
    hipError_t err8 = hipMemcpy(elems->elems_v8, elems_host[8], nelements * sizeof(idx_t), hipMemcpyHostToDevice);
    hipError_t err9 = hipMemcpy(elems->elems_v9, elems_host[9], nelements * sizeof(idx_t), hipMemcpyHostToDevice);

    if (err0 != hipSuccess || err1 != hipSuccess || err2 != hipSuccess || err3 != hipSuccess || err4 != hipSuccess ||
        err5 != hipSuccess || err6 != hipSuccess || err7 != hipSuccess || err8 != hipSuccess || err9 != hipSuccess) {
        printf("ERROR: copying elements to device: %s\n", hipGetErrorString(hipGetLastError()));
        return hipGetLastError();
    }
}

void                                                   //
free_elems_tet10_managed(elems_tet10_device* elems) {  //

    hipError_t err0 = hipFree(elems->elems_v0);
    hipError_t err1 = hipFree(elems->elems_v1);
    hipError_t err2 = hipFree(elems->elems_v2);
    hipError_t err3 = hipFree(elems->elems_v3);
    hipError_t err4 = hipFree(elems->elems_v4);
    hipError_t err5 = hipFree(elems->elems_v5);
    hipError_t err6 = hipFree(elems->elems_v6);
    hipError_t err7 = hipFree(elems->elems_v7);
    hipError_t err8 = hipFree(elems->elems_v8);
    hipError_t err9 = hipFree(elems->elems_v9);

    if (err0 != hipSuccess || err1 != hipSuccess || err2 != hipSuccess || err3 != hipSuccess || err4 != hipSuccess ||
        err5 != hipSuccess || err6 != hipSuccess || err7 != hipSuccess || err8 != hipSuccess || err9 != hipSuccess) {
        printf("ERROR: freeing device memory for elems: %s\n", hipGetErrorString(hipGetLastError()));
    }

    elems->elems_v0 = NULL;
    elems->elems_v1 = NULL;
    elems->elems_v2 = NULL;
    elems->elems_v3 = NULL;
    elems->elems_v4 = NULL;
    elems->elems_v5 = NULL;
    elems->elems_v6 = NULL;
    elems->elems_v7 = NULL;
    elems->elems_v8 = NULL;
    elems->elems_v9 = NULL;
}

//////////////////////////////////////////////////////////
// make_xyz_tet10_device_unified
//////////////////////////////////////////////////////////
xyz_tet10_device                                         //
make_xyz_tet10_device_unified(const ptrdiff_t nnodes) {  //
                                                         //
    xyz_tet10_device xyz;

    xyz.x = NULL;
    xyz.y = NULL;
    xyz.z = NULL;

    return xyz;
}
// end make_xyz_tet10_device_unified

//////////////////////////////////////////////////////////
// copy_xyz_tet10_device_unified
//////////////////////////////////////////////////////////
void                                                      //
copy_xyz_tet10_device_unified(const ptrdiff_t   nnodes,   //
                              xyz_tet10_device* xyz,      //
                              const geom_t**    xyz_host) {  //
                                                          //
    xyz->x = (geom_t*)xyz_host[0];
    xyz->y = (geom_t*)xyz_host[1];
    xyz->z = (geom_t*)xyz_host[2];
}
// end copy_xyz_tet10_device_unified

//////////////////////////////////////////////////////////
// memory_hint_xyz_tet10_device_unified
//////////////////////////////////////////////////////////
void                                                                                   //
memory_hint_xyz_tet10_device_unified(const ptrdiff_t nnodes, xyz_tet10_device* xyz) {  //
                                                                                       //
    hipError_t err0 = hipMemAdvise(xyz->x, nnodes * sizeof(geom_t), hipMemAdviseSetReadMostly, 0);
    hipError_t err1 = hipMemAdvise(xyz->y, nnodes * sizeof(geom_t), hipMemAdviseSetReadMostly, 0);
    hipError_t err2 = hipMemAdvise(xyz->z, nnodes * sizeof(geom_t), hipMemAdviseSetReadMostly, 0);

    if (err0 != hipSuccess || err1 != hipSuccess || err2 != hipSuccess) {
        printf("ERROR: setting memory hint for xyz_tet10_device: %s at %s:%d\n", hipGetErrorString(err0), __FILE__, __LINE__);
        // Handle the error or exit the program
    }

    // prefetch the data to the GPU
    hipError_t err3 = hipMemPrefetchAsync(xyz->x, nnodes * sizeof(geom_t), 0, 0);
    hipError_t err4 = hipMemPrefetchAsync(xyz->y, nnodes * sizeof(geom_t), 0, 0);
    hipError_t err5 = hipMemPrefetchAsync(xyz->z, nnodes * sizeof(geom_t), 0, 0);

    if (err3 != hipSuccess || err4 != hipSuccess || err5 != hipSuccess) {
        printf("ERROR: prefetching data for xyz_tet10_device: %s at %s:%d\n", hipGetErrorString(err3), __FILE__, __LINE__);
        // Handle the error or exit the program
    }
}

//////////////////////////////////////////////////////////
// free_xyz_tet10_device_unified
//////////////////////////////////////////////////////////
void                                                    //
free_xyz_tet10_device_unified(xyz_tet10_device* xyz) {  //
                                                        //
    xyz->x = NULL;
    xyz->y = NULL;
    xyz->z = NULL;
}
// end free_xyz_tet10_device_unified

//////////////////////////////////////////////////////////
// make_xyz_tet10_managed
//////////////////////////////////////////////////////////
xyz_tet10_device make_xyz_tet10_managed(const ptrdiff_t nnodes) {
    xyz_tet10_device xyz;
    hipMallocManaged(&xyz.x, nnodes * sizeof(geom_t));
    hipMallocManaged(&xyz.y, nnodes * sizeof(geom_t));
    hipMallocManaged(&xyz.z, nnodes * sizeof(geom_t));
    return xyz;
}
// end make_xyz_tet10_managed

//////////////////////////////////////////////////////////
// copy_xyz_tet10_managed
//////////////////////////////////////////////////////////
void                                               //
copy_xyz_tet10_managed(const ptrdiff_t   nnodes,   //
                       xyz_tet10_device* xyz,      //
                       const geom_t**    xyz_host) {  //

    hipError_t err0 = hipMemcpy(xyz->x, xyz_host[0], nnodes * sizeof(geom_t), hipMemcpyHostToDevice);
    hipError_t err1 = hipMemcpy(xyz->y, xyz_host[1], nnodes * sizeof(geom_t), hipMemcpyHostToDevice);
    hipError_t err2 = hipMemcpy(xyz->z, xyz_host[2], nnodes * sizeof(geom_t), hipMemcpyHostToDevice);

    if (err0 != hipSuccess || err1 != hipSuccess || err2 != hipSuccess) {
        printf("Error copying xyz_tet10_device to managed memory: %s\n", hipGetErrorString(hipGetLastError()));
        // Handle the error or exit the program
    }
}  // end copy_xyz_tet10_managed

//////////////////////////////////////////////////////////
// free_xyz_tet10_managed
//////////////////////////////////////////////////////////
void                                             //
free_xyz_tet10_managed(xyz_tet10_device* xyz) {  //
                                                 //
    hipError_t err0 = hipFree(xyz->x);
    hipError_t err1 = hipFree(xyz->y);
    hipError_t err2 = hipFree(xyz->z);

    if (err0 != hipSuccess || err1 != hipSuccess || err2 != hipSuccess) {
        printf("Error freeing managed memory for xyz: %s\n", hipGetErrorString(hipGetLastError()));
    }

    xyz->x = NULL;
    xyz->y = NULL;
    xyz->z = NULL;
}  // end free_xyz_tet10_managed

//////////////////////////////////////////////////////////
// memory_hint_xyz_tet10_managed
//////////////////////////////////////////////////////////
void                                                     //
memory_hint_xyz_tet10_managed(const ptrdiff_t   nnodes,  //
                              xyz_tet10_device* xyz) {   //
                                                         //
    hipMemAdvise(xyz->x, nnodes * sizeof(geom_t), hipMemAdviseSetPreferredLocation, hipCpuDeviceId);
    hipMemAdvise(xyz->y, nnodes * sizeof(geom_t), hipMemAdviseSetPreferredLocation, hipCpuDeviceId);
    hipMemAdvise(xyz->z, nnodes * sizeof(geom_t), hipMemAdviseSetPreferredLocation, hipCpuDeviceId);
}  // end memory_hint_xyz_tet10_managed

//////////////////////////////////////////////////////////
// make_elems_tet10_device
//////////////////////////////////////////////////////////
elems_tet10_device                                    //
make_elems_tet10_device(const ptrdiff_t nelements) {  //
                                                      //
    elems_tet10_device elems;                         //

    hipError_t err0 = hipMalloc(&elems.elems_v0, nelements * sizeof(idx_t));
    hipError_t err1 = hipMalloc(&elems.elems_v1, nelements * sizeof(idx_t));
    hipError_t err2 = hipMalloc(&elems.elems_v2, nelements * sizeof(idx_t));
    hipError_t err3 = hipMalloc(&elems.elems_v3, nelements * sizeof(idx_t));
    hipError_t err4 = hipMalloc(&elems.elems_v4, nelements * sizeof(idx_t));
    hipError_t err5 = hipMalloc(&elems.elems_v5, nelements * sizeof(idx_t));
    hipError_t err6 = hipMalloc(&elems.elems_v6, nelements * sizeof(idx_t));
    hipError_t err7 = hipMalloc(&elems.elems_v7, nelements * sizeof(idx_t));
    hipError_t err8 = hipMalloc(&elems.elems_v8, nelements * sizeof(idx_t));
    hipError_t err9 = hipMalloc(&elems.elems_v9, nelements * sizeof(idx_t));

    if (err0 != hipSuccess || err1 != hipSuccess || err2 != hipSuccess || err3 != hipSuccess || err4 != hipSuccess ||
        err5 != hipSuccess || err6 != hipSuccess || err7 != hipSuccess || err8 != hipSuccess || err9 != hipSuccess) {
        printf("ERROR: allocating memory for elems_tet10_device\n");
        // Handle error
    }

    return elems;
}  // end make_elems_tet10_device

elems_tet10_device                                                               //
make_elems_tet10_device_async(const ptrdiff_t nelements, hipStream_t stream) {  //
    elems_tet10_device elems;

    hipError_t err0 = hipMallocAsync(&elems.elems_v0, nelements * sizeof(idx_t), stream);
    hipError_t err1 = hipMallocAsync(&elems.elems_v1, nelements * sizeof(idx_t), stream);
    hipError_t err2 = hipMallocAsync(&elems.elems_v2, nelements * sizeof(idx_t), stream);
    hipError_t err3 = hipMallocAsync(&elems.elems_v3, nelements * sizeof(idx_t), stream);
    hipError_t err4 = hipMallocAsync(&elems.elems_v4, nelements * sizeof(idx_t), stream);
    hipError_t err5 = hipMallocAsync(&elems.elems_v5, nelements * sizeof(idx_t), stream);
    hipError_t err6 = hipMallocAsync(&elems.elems_v6, nelements * sizeof(idx_t), stream);
    hipError_t err7 = hipMallocAsync(&elems.elems_v7, nelements * sizeof(idx_t), stream);
    hipError_t err8 = hipMallocAsync(&elems.elems_v8, nelements * sizeof(idx_t), stream);
    hipError_t err9 = hipMallocAsync(&elems.elems_v9, nelements * sizeof(idx_t), stream);

    if (err0 != hipSuccess || err1 != hipSuccess || err2 != hipSuccess || err3 != hipSuccess || err4 != hipSuccess ||
        err5 != hipSuccess || err6 != hipSuccess || err7 != hipSuccess || err8 != hipSuccess || err9 != hipSuccess) {
        printf("ERROR: allocating memory for elems_tet10_device async\n");
        // Handle error
    }

    return elems;
}

//////////////////////////////////////////////////////////
// copy_elems_tet10_device
//////////////////////////////////////////////////////////
hipError_t copy_elems_tet10_device(const ptrdiff_t     nelements,  //
                                    elems_tet10_device* elems,      //
                                    const idx_t**       elems_host) {     //

    hipError_t err0 = hipMemcpy(elems->elems_v0, elems_host[0], nelements * sizeof(idx_t), hipMemcpyHostToDevice);
    hipError_t err1 = hipMemcpy(elems->elems_v1, elems_host[1], nelements * sizeof(idx_t), hipMemcpyHostToDevice);
    hipError_t err2 = hipMemcpy(elems->elems_v2, elems_host[2], nelements * sizeof(idx_t), hipMemcpyHostToDevice);
    hipError_t err3 = hipMemcpy(elems->elems_v3, elems_host[3], nelements * sizeof(idx_t), hipMemcpyHostToDevice);
    hipError_t err4 = hipMemcpy(elems->elems_v4, elems_host[4], nelements * sizeof(idx_t), hipMemcpyHostToDevice);
    hipError_t err5 = hipMemcpy(elems->elems_v5, elems_host[5], nelements * sizeof(idx_t), hipMemcpyHostToDevice);
    hipError_t err6 = hipMemcpy(elems->elems_v6, elems_host[6], nelements * sizeof(idx_t), hipMemcpyHostToDevice);
    hipError_t err7 = hipMemcpy(elems->elems_v7, elems_host[7], nelements * sizeof(idx_t), hipMemcpyHostToDevice);
    hipError_t err8 = hipMemcpy(elems->elems_v8, elems_host[8], nelements * sizeof(idx_t), hipMemcpyHostToDevice);
    hipError_t err9 = hipMemcpy(elems->elems_v9, elems_host[9], nelements * sizeof(idx_t), hipMemcpyHostToDevice);

    if (err0 != hipSuccess || err1 != hipSuccess || err2 != hipSuccess || err3 != hipSuccess || err4 != hipSuccess ||
        err5 != hipSuccess || err6 != hipSuccess || err7 != hipSuccess || err8 != hipSuccess || err9 != hipSuccess) {
        printf("ERROR: copying elements to device: %s\n", hipGetErrorString(hipGetLastError()));
        return hipGetLastError();
    }

    return hipSuccess;
}  // end copy_elems_tet10_device

///////////////////////////////////////////////////////////
// copy_elems_tet10_device_async
///////////////////////////////////////////////////////////
hipError_t                                                    //
copy_elems_tet10_device_async(const ptrdiff_t     nelements,   //
                              elems_tet10_device* elems,       //
                              const idx_t**       elems_host,  //
                              hipStream_t        stream) {           //

    hipError_t err0 = hipMemcpyAsync(elems->elems_v0, elems_host[0], nelements * sizeof(idx_t), hipMemcpyHostToDevice, stream);
    hipError_t err1 = hipMemcpyAsync(elems->elems_v1, elems_host[1], nelements * sizeof(idx_t), hipMemcpyHostToDevice, stream);
    hipError_t err2 = hipMemcpyAsync(elems->elems_v2, elems_host[2], nelements * sizeof(idx_t), hipMemcpyHostToDevice, stream);
    hipError_t err3 = hipMemcpyAsync(elems->elems_v3, elems_host[3], nelements * sizeof(idx_t), hipMemcpyHostToDevice, stream);
    hipError_t err4 = hipMemcpyAsync(elems->elems_v4, elems_host[4], nelements * sizeof(idx_t), hipMemcpyHostToDevice, stream);
    hipError_t err5 = hipMemcpyAsync(elems->elems_v5, elems_host[5], nelements * sizeof(idx_t), hipMemcpyHostToDevice, stream);
    hipError_t err6 = hipMemcpyAsync(elems->elems_v6, elems_host[6], nelements * sizeof(idx_t), hipMemcpyHostToDevice, stream);
    hipError_t err7 = hipMemcpyAsync(elems->elems_v7, elems_host[7], nelements * sizeof(idx_t), hipMemcpyHostToDevice, stream);
    hipError_t err8 = hipMemcpyAsync(elems->elems_v8, elems_host[8], nelements * sizeof(idx_t), hipMemcpyHostToDevice, stream);
    hipError_t err9 = hipMemcpyAsync(elems->elems_v9, elems_host[9], nelements * sizeof(idx_t), hipMemcpyHostToDevice, stream);

    if (err0 != hipSuccess || err1 != hipSuccess || err2 != hipSuccess || err3 != hipSuccess || err4 != hipSuccess ||
        err5 != hipSuccess || err6 != hipSuccess || err7 != hipSuccess || err8 != hipSuccess || err9 != hipSuccess) {
        printf("ERROR: copying elements to device async: %s\n", hipGetErrorString(hipGetLastError()));
        return hipGetLastError();
    }

    return hipSuccess;
}

//////////////////////////////////////////////////////////
// free_elems_tet10_device
//////////////////////////////////////////////////////////
void free_elems_tet10_device(elems_tet10_device* elems) {  //
                                                           //
    hipError_t err0 = hipFree(elems->elems_v0);
    hipError_t err1 = hipFree(elems->elems_v1);
    hipError_t err2 = hipFree(elems->elems_v2);
    hipError_t err3 = hipFree(elems->elems_v3);
    hipError_t err4 = hipFree(elems->elems_v4);
    hipError_t err5 = hipFree(elems->elems_v5);
    hipError_t err6 = hipFree(elems->elems_v6);
    hipError_t err7 = hipFree(elems->elems_v7);
    hipError_t err8 = hipFree(elems->elems_v8);
    hipError_t err9 = hipFree(elems->elems_v9);

    if (err0 != hipSuccess || err1 != hipSuccess || err2 != hipSuccess || err3 != hipSuccess || err4 != hipSuccess ||
        err5 != hipSuccess || err6 != hipSuccess || err7 != hipSuccess || err8 != hipSuccess || err9 != hipSuccess) {
        printf("ERROR: freeing device memory for elems: %s\n", hipGetErrorString(hipGetLastError()));
    }

    elems->elems_v0 = NULL;
    elems->elems_v1 = NULL;
    elems->elems_v2 = NULL;
    elems->elems_v3 = NULL;
    elems->elems_v4 = NULL;
    elems->elems_v5 = NULL;
    elems->elems_v6 = NULL;
    elems->elems_v7 = NULL;
    elems->elems_v8 = NULL;
    elems->elems_v9 = NULL;
}  // end free_elems_tet10_device

//////////////////////////////////////////////////////////
// elems_tet10_device for unified memory
//////////////////////////////////////////////////////////

/**
 * @brief
 *
 * @param nelements
 * @return elems_tet10_device
 */
elems_tet10_device                                            //
make_elems_tet10_device_unified(const ptrdiff_t nelements) {  //
                                                              //
    elems_tet10_device elems;
    elems.elems_v0 = NULL;
    elems.elems_v1 = NULL;
    elems.elems_v2 = NULL;
    elems.elems_v3 = NULL;
    elems.elems_v4 = NULL;
    elems.elems_v5 = NULL;
    elems.elems_v6 = NULL;
    elems.elems_v7 = NULL;
    elems.elems_v8 = NULL;
    elems.elems_v9 = NULL;
    return elems;
}

/**
 * @brief
 *
 * @param nelements
 * @param elems
 * @param elems_host
 * @return hipError_t
 */
hipError_t                                                     //
copy_elems_tet10_device_unified(const ptrdiff_t     nelements,  //
                                elems_tet10_device* elems,      //
                                const idx_t**       elems_host) {
    elems->elems_v0 = (idx_t*)elems_host[0];
    elems->elems_v1 = (idx_t*)elems_host[1];
    elems->elems_v2 = (idx_t*)elems_host[2];
    elems->elems_v3 = (idx_t*)elems_host[3];
    elems->elems_v4 = (idx_t*)elems_host[4];
    elems->elems_v5 = (idx_t*)elems_host[5];
    elems->elems_v6 = (idx_t*)elems_host[6];
    elems->elems_v7 = (idx_t*)elems_host[7];
    elems->elems_v8 = (idx_t*)elems_host[8];
    elems->elems_v9 = (idx_t*)elems_host[9];

    return hipSuccess;
}

/**
 * @brief
 *
 * @param elems
 */
void                                                          //
free_elems_tet10_device_unified(elems_tet10_device* elems) {  //
    elems->elems_v0 = NULL;
    elems->elems_v1 = NULL;
    elems->elems_v2 = NULL;
    elems->elems_v3 = NULL;
    elems->elems_v4 = NULL;
    elems->elems_v5 = NULL;
    elems->elems_v6 = NULL;
    elems->elems_v7 = NULL;
    elems->elems_v8 = NULL;
    elems->elems_v9 = NULL;

}  //

/**
 * @brief
 *
 * @param nelements
 * @param elems
 */
void                                                                   //
memory_hint_elems_tet10_device_unified(ptrdiff_t           nelements,  //
                                       elems_tet10_device* elems) {    //

    int  device_id;
    auto error = hipGetDevice(&device_id);

    hipError_t err0 = hipMemAdvise(elems->elems_v0, nelements * sizeof(idx_t), hipMemAdviseSetReadMostly, device_id);
    hipError_t err1 = hipMemAdvise(elems->elems_v1, nelements * sizeof(idx_t), hipMemAdviseSetReadMostly, device_id);
    hipError_t err2 = hipMemAdvise(elems->elems_v2, nelements * sizeof(idx_t), hipMemAdviseSetReadMostly, device_id);
    hipError_t err3 = hipMemAdvise(elems->elems_v3, nelements * sizeof(idx_t), hipMemAdviseSetReadMostly, device_id);
    hipError_t err4 = hipMemAdvise(elems->elems_v4, nelements * sizeof(idx_t), hipMemAdviseSetReadMostly, device_id);
    hipError_t err5 = hipMemAdvise(elems->elems_v5, nelements * sizeof(idx_t), hipMemAdviseSetReadMostly, device_id);
    hipError_t err6 = hipMemAdvise(elems->elems_v6, nelements * sizeof(idx_t), hipMemAdviseSetReadMostly, device_id);
    hipError_t err7 = hipMemAdvise(elems->elems_v7, nelements * sizeof(idx_t), hipMemAdviseSetReadMostly, device_id);
    hipError_t err8 = hipMemAdvise(elems->elems_v8, nelements * sizeof(idx_t), hipMemAdviseSetReadMostly, device_id);
    hipError_t err9 = hipMemAdvise(elems->elems_v9, nelements * sizeof(idx_t), hipMemAdviseSetReadMostly, device_id);

    if (err0 != hipSuccess || err1 != hipSuccess || err2 != hipSuccess || err3 != hipSuccess || err4 != hipSuccess ||
        err5 != hipSuccess || err6 != hipSuccess || err7 != hipSuccess || err8 != hipSuccess || err9 != hipSuccess) {
        printf("ERROR: setting memory hint for elems_tet10_device: %s at %s:%d\n", hipGetErrorString(err0), __FILE__, __LINE__);
        // Handle the error or exit the program
    }

    // prefetch the data to the GPU
    hipError_t err10 = hipMemPrefetchAsync(elems->elems_v0, nelements * sizeof(idx_t), device_id);
    hipError_t err11 = hipMemPrefetchAsync(elems->elems_v1, nelements * sizeof(idx_t), device_id);
    hipError_t err12 = hipMemPrefetchAsync(elems->elems_v2, nelements * sizeof(idx_t), device_id);
    hipError_t err13 = hipMemPrefetchAsync(elems->elems_v3, nelements * sizeof(idx_t), device_id);
    hipError_t err14 = hipMemPrefetchAsync(elems->elems_v4, nelements * sizeof(idx_t), device_id);
    hipError_t err15 = hipMemPrefetchAsync(elems->elems_v5, nelements * sizeof(idx_t), device_id);
    hipError_t err16 = hipMemPrefetchAsync(elems->elems_v6, nelements * sizeof(idx_t), device_id);
    hipError_t err17 = hipMemPrefetchAsync(elems->elems_v7, nelements * sizeof(idx_t), device_id);
    hipError_t err18 = hipMemPrefetchAsync(elems->elems_v8, nelements * sizeof(idx_t), device_id);
    hipError_t err19 = hipMemPrefetchAsync(elems->elems_v9, nelements * sizeof(idx_t), device_id);

    if (err10 != hipSuccess || err11 != hipSuccess || err12 != hipSuccess || err13 != hipSuccess || err14 != hipSuccess ||
        err15 != hipSuccess || err16 != hipSuccess || err17 != hipSuccess || err18 != hipSuccess || err19 != hipSuccess) {
        printf("ERROR: prefetching data for elems_tet10_device: %s at %s:%d\n", hipGetErrorString(err10), __FILE__, __LINE__);
        // Handle the error or exit the program
    }
}

//////////////////////////////////////////////////////////
// memory_hint_elems_tet10_device
//////////////////////////////////////////////////////////
void                                                  //
memory_hint_read_mostly(const ptrdiff_t array_size,   //
                        const ptrdiff_t sizeof_type,  //
                        void*           ptr) {
    int  device_id = 0;
    auto error     = hipGetDevice(&device_id);

    hipError_t err = hipMemAdvise(ptr, array_size * sizeof_type, hipMemAdviseSetReadMostly, device_id);

    if (err != hipSuccess) {
        printf("ERROR: setting memory hint for elems_tet10_device: %s at %s:%d\n", hipGetErrorString(err), __FILE__, __LINE__);
        // Handle the error or exit the program
    }

    // prefetch the data to the GPU
    hipError_t err2 = hipMemPrefetchAsync(ptr, array_size * sizeof_type, device_id);

    if (err2 != hipSuccess) {
        printf("ERROR: prefetching data for elems_tet10_device: %s at %s:%d\n", hipGetErrorString(err2), __FILE__, __LINE__);
        // Handle the error or exit the program
    }
}

//////////////////////////////////////////////////////////
// memory_hint_write_mostly
//////////////////////////////////////////////////////////
void                                                   //
memory_hint_write_mostly(const ptrdiff_t array_size,   //
                         const ptrdiff_t sizeof_type,  //
                         void*           ptr) {
    int  device_id = 0;
    auto error     = hipGetDevice(&device_id);

    hipError_t err = hipMemAdvise(ptr, array_size * sizeof_type, hipMemAdviseSetAccessedBy, device_id);

    if (err != hipSuccess) {
        printf("ERROR: setting memory hint for elems_tet10_device: %s at %s:%d\n", hipGetErrorString(err), __FILE__, __LINE__);
        // Handle the error or exit the program
    }

    // prefetch the data to the GPU
    hipError_t err2 = hipMemPrefetchAsync(ptr, array_size * sizeof_type, device_id);

    if (err2 != hipSuccess) {
        printf("ERROR: prefetching data for elems_tet10_device: %s at %s:%d\n", hipGetErrorString(err2), __FILE__, __LINE__);
        // Handle the error or exit the program
    }
}