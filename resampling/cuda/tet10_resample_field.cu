#include "hip/hip_runtime.h"
#include <hip/hip_cooperative_groups.h>
#include <hip/hip_runtime_api.h>
#include <sfem_base.h>
#include <stdio.h>

#define real_type real_t

#include "quadratures_rule_cuda.h"

#define MY_RESTRICT __restrict__

#define __WARP_SIZE__ 32

/////////////////////////////////////////////////////////////////
// Struct for xyz
/////////////////////////////////////////////////////////////////
typedef struct {
    float* x = NULL;
    float* y = NULL;
    float* z = NULL;
} xyz_tet10_device;
// end struct xyz_tet10_device

xyz_tet10_device make_xyz_tet10_device(const ptrdiff_t nnodes) {
    xyz_tet10_device xyz;
    hipMalloc(&xyz.x, nnodes * sizeof(float));
    hipMalloc(&xyz.y, nnodes * sizeof(float));
    hipMalloc(&xyz.z, nnodes * sizeof(float));
    return xyz;
}
// end make_xyz_tet10_device

void copy_xyz_tet10_device(const ptrdiff_t nnodes,    //
                           xyz_tet10_device* xyz,     //
                           const float** xyz_host) {  //
    hipError_t err0 =
            hipMemcpy(xyz->x, xyz_host[0], nnodes * sizeof(float), hipMemcpyHostToDevice);
    hipError_t err1 =
            hipMemcpy(xyz->y, xyz_host[1], nnodes * sizeof(float), hipMemcpyHostToDevice);
    hipError_t err2 =
            hipMemcpy(xyz->z, xyz_host[2], nnodes * sizeof(float), hipMemcpyHostToDevice);

    if (err0 != hipSuccess || err1 != hipSuccess || err2 != hipSuccess) {
        printf("Error copying xyz_tet10_device to device: %s\n", hipGetErrorString(err0));
        // Handle the error or exit the program
    }
}  // end copy_xyz_tet10_device

void free_xyz_tet10_device(xyz_tet10_device xyz) {
    hipFree(xyz.x);
    hipFree(xyz.y);
    hipFree(xyz.z);

    xyz.x = NULL;
    xyz.y = NULL;
    xyz.z = NULL;
}
// end free_xyz_tet10_device

/////////////////////////////////////////////////////////////////
// Struct for elems
/////////////////////////////////////////////////////////////////
typedef struct {
    int* elems_v0 = NULL;
    int* elems_v1 = NULL;
    int* elems_v2 = NULL;
    int* elems_v3 = NULL;
    int* elems_v4 = NULL;
    int* elems_v5 = NULL;
    int* elems_v6 = NULL;
    int* elems_v7 = NULL;
    int* elems_v8 = NULL;
    int* elems_v9 = NULL;
} elems_tet10_device;
// end struct elems_tet10_device

elems_tet10_device make_elems_tet10_device(const ptrdiff_t nelements) {
    elems_tet10_device elems;

    hipError_t err0 = hipMalloc(&elems.elems_v0, nelements * sizeof(int));
    hipError_t err1 = hipMalloc(&elems.elems_v1, nelements * sizeof(int));
    hipError_t err2 = hipMalloc(&elems.elems_v2, nelements * sizeof(int));
    hipError_t err3 = hipMalloc(&elems.elems_v3, nelements * sizeof(int));
    hipError_t err4 = hipMalloc(&elems.elems_v4, nelements * sizeof(int));
    hipError_t err5 = hipMalloc(&elems.elems_v5, nelements * sizeof(int));
    hipError_t err6 = hipMalloc(&elems.elems_v6, nelements * sizeof(int));
    hipError_t err7 = hipMalloc(&elems.elems_v7, nelements * sizeof(int));
    hipError_t err8 = hipMalloc(&elems.elems_v8, nelements * sizeof(int));
    hipError_t err9 = hipMalloc(&elems.elems_v9, nelements * sizeof(int));

    if (err0 != hipSuccess || err1 != hipSuccess || err2 != hipSuccess || err3 != hipSuccess ||
        err4 != hipSuccess || err5 != hipSuccess || err6 != hipSuccess || err7 != hipSuccess ||
        err8 != hipSuccess || err9 != hipSuccess) {
        printf("Error allocating memory for elems_tet10_device\n");
        // Handle error
    }

    return elems;
}  // end make_elems_tet10_device

hipError_t copy_elems_tet10_device(const ptrdiff_t nelements,   //
                                    elems_tet10_device* elems,   //
                                    const idx_t** elems_host) {  //
    hipError_t err0 = hipMemcpy(
            elems->elems_v0, elems_host[0], nelements * sizeof(int), hipMemcpyHostToDevice);
    hipError_t err1 = hipMemcpy(
            elems->elems_v1, elems_host[1], nelements * sizeof(int), hipMemcpyHostToDevice);
    hipError_t err2 = hipMemcpy(
            elems->elems_v2, elems_host[2], nelements * sizeof(int), hipMemcpyHostToDevice);
    hipError_t err3 = hipMemcpy(
            elems->elems_v3, elems_host[3], nelements * sizeof(int), hipMemcpyHostToDevice);
    hipError_t err4 = hipMemcpy(
            elems->elems_v4, elems_host[4], nelements * sizeof(int), hipMemcpyHostToDevice);
    hipError_t err5 = hipMemcpy(
            elems->elems_v5, elems_host[5], nelements * sizeof(int), hipMemcpyHostToDevice);
    hipError_t err6 = hipMemcpy(
            elems->elems_v6, elems_host[6], nelements * sizeof(int), hipMemcpyHostToDevice);
    hipError_t err7 = hipMemcpy(
            elems->elems_v7, elems_host[7], nelements * sizeof(int), hipMemcpyHostToDevice);
    hipError_t err8 = hipMemcpy(
            elems->elems_v8, elems_host[8], nelements * sizeof(int), hipMemcpyHostToDevice);
    hipError_t err9 = hipMemcpy(
            elems->elems_v9, elems_host[9], nelements * sizeof(int), hipMemcpyHostToDevice);

    if (err0 != hipSuccess || err1 != hipSuccess || err2 != hipSuccess || err3 != hipSuccess ||
        err4 != hipSuccess || err5 != hipSuccess || err6 != hipSuccess || err7 != hipSuccess ||
        err8 != hipSuccess || err9 != hipSuccess) {
        printf("Error copying elements to device: %s\n", hipGetErrorString(hipGetLastError()));
        return hipGetLastError();
    }

    return hipSuccess;
}  // end copy_elems_tet10_device

void free_elems_tet10_device(elems_tet10_device elems) {
    hipError_t err0 = hipFree(elems.elems_v0);
    hipError_t err1 = hipFree(elems.elems_v1);
    hipError_t err2 = hipFree(elems.elems_v2);
    hipError_t err3 = hipFree(elems.elems_v3);
    hipError_t err4 = hipFree(elems.elems_v4);
    hipError_t err5 = hipFree(elems.elems_v5);
    hipError_t err6 = hipFree(elems.elems_v6);
    hipError_t err7 = hipFree(elems.elems_v7);
    hipError_t err8 = hipFree(elems.elems_v8);
    hipError_t err9 = hipFree(elems.elems_v9);

    if (err0 != hipSuccess || err1 != hipSuccess || err2 != hipSuccess || err3 != hipSuccess ||
        err4 != hipSuccess || err5 != hipSuccess || err6 != hipSuccess || err7 != hipSuccess ||
        err8 != hipSuccess || err9 != hipSuccess) {
        printf("Error freeing device memory for elems: %s\n",
               hipGetErrorString(hipGetLastError()));
    }

    elems.elems_v0 = NULL;
    elems.elems_v1 = NULL;
    elems.elems_v2 = NULL;
    elems.elems_v3 = NULL;
    elems.elems_v4 = NULL;
    elems.elems_v5 = NULL;
    elems.elems_v6 = NULL;
    elems.elems_v7 = NULL;
    elems.elems_v8 = NULL;
    elems.elems_v9 = NULL;
}  // end free_elems_tet10_device

//-------------------------------------------
/// iso-parametric version
//-------------------------------------------
__device__ real_t                                    //
tet10_measure_cu(const double* const MY_RESTRICT x,  //
                 const double* const MY_RESTRICT y,  //
                 const double* const MY_RESTRICT z,  //

                 // Quadrature point //
                 const double qx,    //
                 const double qy,    //
                 const double qz) {  //
    //
    const real_t x0 = 4 * qz;
    const real_t x1 = x0 - 1;
    const real_t x2 = 4 * qy;
    const real_t x3 = 4 * qx;
    const real_t x4 = x3 - 4;
    const real_t x5 = -8 * qz - x2 - x4;
    const real_t x6 = -x3 * y[4];
    const real_t x7 = x0 + x2;
    const real_t x8 = x3 + x7 - 3;
    const real_t x9 = x8 * y[0];
    const real_t x10 = -x2 * y[6] + x9;
    const real_t x11 = x1 * y[3] + x10 + x2 * y[9] + x3 * y[8] + x5 * y[7] + x6;
    const real_t x12 = -x2 * z[6];
    const real_t x13 = -x0 * z[7];
    const real_t x14 = x3 - 1;
    const real_t x15 = x8 * z[0];
    const real_t x16 = -8 * qx - x7 + 4;
    const real_t x17 = x0 * z[8] + x12 + x13 + x14 * z[1] + x15 + x16 * z[4] + x2 * z[5];
    const real_t x18 = x2 - 1;
    const real_t x19 = -8 * qy - x0 - x4;
    const real_t x20 = -x3 * x[4];
    const real_t x21 = x8 * x[0];
    const real_t x22 = -x0 * x[7] + x21;
    const real_t x23 = (1.0 / 6.0) * x0 * x[9] + (1.0 / 6.0) * x18 * x[2] +
                       (1.0 / 6.0) * x19 * x[6] + (1.0 / 6.0) * x20 + (1.0 / 6.0) * x22 +
                       (1.0 / 6.0) * x3 * x[5];
    const real_t x24 = -x0 * y[7];
    const real_t x25 = x0 * y[8] + x10 + x14 * y[1] + x16 * y[4] + x2 * y[5] + x24;
    const real_t x26 = x15 - x3 * z[4];
    const real_t x27 = x1 * z[3] + x12 + x2 * z[9] + x26 + x3 * z[8] + x5 * z[7];
    const real_t x28 = x0 * y[9] + x18 * y[2] + x19 * y[6] + x24 + x3 * y[5] + x6 + x9;
    const real_t x29 = -x2 * x[6];
    const real_t x30 = (1.0 / 6.0) * x1 * x[3] + (1.0 / 6.0) * x2 * x[9] + (1.0 / 6.0) * x20 +
                       (1.0 / 6.0) * x21 + (1.0 / 6.0) * x29 + (1.0 / 6.0) * x3 * x[8] +
                       (1.0 / 6.0) * x5 * x[7];
    const real_t x31 = x0 * z[9] + x13 + x18 * z[2] + x19 * z[6] + x26 + x3 * z[5];
    const real_t x32 = (1.0 / 6.0) * x0 * x[8] + (1.0 / 6.0) * x14 * x[1] +
                       (1.0 / 6.0) * x16 * x[4] + (1.0 / 6.0) * x2 * x[5] + (1.0 / 6.0) * x22 +
                       (1.0 / 6.0) * x29;
    //
    return x11 * x17 * x23 - x11 * x31 * x32 - x17 * x28 * x30 - x23 * x25 * x27 + x25 * x30 * x31 +
           x27 * x28 * x32;
}  // end tet10_measure_cu

__device__ void tet10_transform_cu(const real_t* const MY_RESTRICT x,
                                   const real_t* const MY_RESTRICT y,
                                   const real_t* const MY_RESTRICT z,
                                   // Quadrature point
                                   const real_t qx, const real_t qy, const real_t qz,
                                   // Output
                                   real_t* const MY_RESTRICT out_x, real_t* const MY_RESTRICT out_y,
                                   real_t* const MY_RESTRICT out_z) {
    const real_t x0 = 4 * qx;
    const real_t x1 = qy * x0;
    const real_t x2 = qz * x0;
    const real_t x3 = 4 * qy;
    const real_t x4 = qz * x3;
    const real_t x5 = 2 * qx - 1;
    const real_t x6 = qx * x5;
    const real_t x7 = 2 * qy;
    const real_t x8 = qy * (x7 - 1);
    const real_t x9 = 2 * qz;
    const real_t x10 = qz * (x9 - 1);
    const real_t x11 = -4 * qz - x0 - x3 + 4;
    const real_t x12 = qx * x11;
    const real_t x13 = qy * x11;
    const real_t x14 = qz * x11;
    const real_t x15 = (-x5 - x7 - x9) * (-qx - qy - qz + 1);

    *out_x = x[0] * x15 + x[1] * x6 + x[2] * x8 + x[3] * x10 + x[4] * x12 + x[5] * x1 + x[6] * x13 +
             x[7] * x14 + x[8] * x2 + x[9] * x4;
    *out_y = y[0] * x15 + y[1] * x6 + y[2] * x8 + y[3] * x10 + y[4] * x12 + y[5] * x1 + y[6] * x13 +
             y[7] * x14 + y[8] * x2 + y[9] * x4;
    *out_z = z[0] * x15 + z[1] * x6 + z[2] * x8 + z[3] * x10 + z[4] * x12 + z[5] * x1 + z[6] * x13 +
             z[7] * x14 + z[8] * x2 + z[9] * x4;
}  // end tet10_transform_cu

__device__ void tet10_dual_basis_hrt_cu(const real_t qx, const real_t qy, const real_t qz,
                                        real_t* const f) {
    const real_t x0 = 2 * qy;
    const real_t x1 = 2 * qz;
    const real_t x2 = 2 * qx - 1;
    const real_t x3 = (-x0 - x1 - x2) * (-qx - qy - qz + 1);
    const real_t x4 = x0 - 1;
    const real_t x5 = (5.0 / 18.0) * qy;
    const real_t x6 = x4 * x5;
    const real_t x7 = x1 - 1;
    const real_t x8 = (5.0 / 18.0) * qz;
    const real_t x9 = x7 * x8;
    const real_t x10 = -4 * qx - 4 * qy - 4 * qz + 4;
    const real_t x11 = (5.0 / 72.0) * x10;
    const real_t x12 = qy * qz;
    const real_t x13 = qx * x11 + (10.0 / 9.0) * x12 + x6 + x9;
    const real_t x14 = (5.0 / 18.0) * qx;
    const real_t x15 = x14 * x2;
    const real_t x16 = (10.0 / 9.0) * qx;
    const real_t x17 = qy * x11 + qz * x16 + x15;
    const real_t x18 = qy * x16 + qz * x11;
    const real_t x19 = qx * x2;
    const real_t x20 = (5.0 / 18.0) * x3;
    const real_t x21 = qy * x14 + x10 * x8 + x20;
    const real_t x22 = qz * x14 + x10 * x5;
    const real_t x23 = qy * x4;
    const real_t x24 = qz * x5 + x10 * x14;
    const real_t x25 = qz * x7;
    const real_t x26 = (40.0 / 27.0) * x23;
    const real_t x27 = (115.0 / 27.0) * x10;
    const real_t x28 = (110.0 / 27.0) * qx;
    const real_t x29 = -qz * x28;
    const real_t x30 = (55.0 / 54.0) * x10;
    const real_t x31 = -qy * x30;
    const real_t x32 = (10.0 / 27.0) * x19;
    const real_t x33 = (40.0 / 27.0) * x25;
    const real_t x34 = x29 + x31 + x32 + x33;
    const real_t x35 = -qy * x28;
    const real_t x36 = -qz * x30;
    const real_t x37 = (10.0 / 27.0) * x3;
    const real_t x38 = x35 + x36 + x37;
    const real_t x39 = (40.0 / 27.0) * x10;
    const real_t x40 = qx * qy;
    const real_t x41 = -qx * x30 - 110.0 / 27.0 * x12;
    const real_t x42 = (10.0 / 27.0) * x23;
    const real_t x43 = (40.0 / 27.0) * x3;
    const real_t x44 = x42 + x43;
    const real_t x45 = qx * qz;
    const real_t x46 = (40.0 / 27.0) * x19;
    const real_t x47 = x41 + x46;
    const real_t x48 = (10.0 / 27.0) * x25;
    const real_t x49 = x26 + x48;
    const real_t x50 = x29 + x31;
    const real_t x51 = x35 + x36;

    f[0] = x13 + x17 + x18 + (25.0 / 9.0) * x3;
    f[1] = x13 + (25.0 / 9.0) * x19 + x21 + x22;
    f[2] = x17 + x21 + (25.0 / 9.0) * x23 + x24 + x9;
    f[3] = x15 + x18 + x20 + x22 + x24 + (25.0 / 9.0) * x25 + x6;
    f[4] = qx * x27 + (160.0 / 27.0) * x12 + x26 + x34 + x38;
    f[5] = qz * x39 + x34 + (460.0 / 27.0) * x40 + x41 + x44;
    f[6] = qy * x27 + x33 + x38 + x42 + (160.0 / 27.0) * x45 + x47;
    f[7] = qz * x27 + x37 + (160.0 / 27.0) * x40 + x47 + x49 + x50;
    f[8] = qy * x39 + x32 + x41 + x43 + (460.0 / 27.0) * x45 + x49 + x51;
    f[9] = qx * x39 + (460.0 / 27.0) * x12 + x44 + x46 + x48 + x50 + x51;
}  //    end tet10_dual_basis_hrt_cu

__device__ void hex_aa_8_eval_fun_cu(
        // Quadrature point (local coordinates)
        // With respect to the hat functions of a cube element
        // In a local coordinate system
        const real_t x, const real_t y, const real_t z,
        // Output
        real_t* const MY_RESTRICT f) {
    //
    f[0] = (1.0 - x) * (1.0 - y) * (1.0 - z);
    f[1] = x * (1.0 - y) * (1.0 - z);
    f[2] = x * y * (1.0 - z);
    f[3] = (1.0 - x) * y * (1.0 - z);
    f[4] = (1.0 - x) * (1.0 - y) * z;
    f[5] = x * (1.0 - y) * z;
    f[6] = x * y * z;
    f[7] = (1.0 - x) * y * z;
}  // end hex_aa_8_eval_fun_cu

__device__ void hex_aa_8_collect_coeffs_cu(
        const ptrdiff_t stride0, const ptrdiff_t stride1, const ptrdiff_t stride2,

        const ptrdiff_t i, const ptrdiff_t j, const ptrdiff_t k,
        // Attention this is geometric data transformed to solver data!
        const real_t* MY_RESTRICT data, real_t* MY_RESTRICT out) {
    //
    const ptrdiff_t i0 = i * stride0 + j * stride1 + k * stride2;
    const ptrdiff_t i1 = (i + 1) * stride0 + j * stride1 + k * stride2;
    const ptrdiff_t i2 = (i + 1) * stride0 + (j + 1) * stride1 + k * stride2;
    const ptrdiff_t i3 = i * stride0 + (j + 1) * stride1 + k * stride2;
    const ptrdiff_t i4 = i * stride0 + j * stride1 + (k + 1) * stride2;
    const ptrdiff_t i5 = (i + 1) * stride0 + j * stride1 + (k + 1) * stride2;
    const ptrdiff_t i6 = (i + 1) * stride0 + (j + 1) * stride1 + (k + 1) * stride2;
    const ptrdiff_t i7 = i * stride0 + (j + 1) * stride1 + (k + 1) * stride2;

    out[0] = data[i0];
    out[1] = data[i1];
    out[2] = data[i2];
    out[3] = data[i3];
    out[4] = data[i4];
    out[5] = data[i5];
    out[6] = data[i6];
    out[7] = data[i7];
}  // end hex_aa_8_collect_coeffs_cu

/**
 * @brief Resample a field from a hex8 mesh to a tet10 mesh
 *
 */
__global__ void hex8_to_isoparametric_tet10_resample_field_local_reduce_kernel(
        // Mesh
        const ptrdiff_t start_element,  // start element
        const ptrdiff_t end_element,    // end element
        const ptrdiff_t nnodes,         // number of nodes

        elems_tet10_device elems,  // connectivity
        xyz_tet10_device xyz,      // coordinates
        // SDF
        const ptrdiff_t nx,  // number of nodes in each direction x
        const ptrdiff_t ny,  // number of nodes in each direction y
        const ptrdiff_t nz,  // number of nodes in each direction z

        const ptrdiff_t stride0,  // stride of the data
        const ptrdiff_t stride1,  // stride of the data
        const ptrdiff_t stride2,  // stride of the data

        const geom_t originx,  // origin of the domain
        const geom_t originy,  // origin of the domain
        const geom_t originz,  // origin of the domain

        const geom_t deltax,  // delta of the domain
        const geom_t deltay,  // delta of the domain
        const geom_t deltaz,  // delta of the domain

        const real_t* const MY_RESTRICT data,  // SDF
        // Output
        real_t* const MY_RESTRICT weighted_field) {
    //
    // printf("============================================================\n");
    // printf("Start: hex8_to_isoparametric_tet10_resample_field_local_reduce_kernel\n");
    // printf("============================================================\n");

    const real_t ox = (real_t)originx;
    const real_t oy = (real_t)originy;
    const real_t oz = (real_t)originz;

    const real_t dx = (real_t)deltax;
    const real_t dy = (real_t)deltay;
    const real_t dz = (real_t)deltaz;

    ////////////////////////////////////////
    // Kernel specific variables

    namespace cg = cooperative_groups;

    cg::thread_block g = cg::this_thread_block();

    const ptrdiff_t element_i = (blockIdx.x * blockDim.x + threadIdx.x) / __WARP_SIZE__;

    if (element_i < start_element or element_i >= end_element) return;

    auto tile = cg::tiled_partition<__WARP_SIZE__>(g);
    const unsigned tile_rank = tile.thread_rank();

    ////////////////////////////////////////
    // Quadrature points
    ptrdiff_t ev[10];

    // ISOPARAMETRIC
    real_t x[10], y[10], z[10];

    real_t hex8_f[8];
    real_t coeffs[8];

    real_t tet10_f[10];

    // loop over the ndes of the element
    ev[0] = elems.elems_v0[element_i];
    ev[1] = elems.elems_v1[element_i];
    ev[2] = elems.elems_v2[element_i];
    ev[3] = elems.elems_v3[element_i];
    ev[4] = elems.elems_v4[element_i];
    ev[5] = elems.elems_v5[element_i];
    ev[6] = elems.elems_v6[element_i];
    ev[7] = elems.elems_v7[element_i];
    ev[8] = elems.elems_v8[element_i];
    ev[9] = elems.elems_v9[element_i];

    // ISOPARAMETRIC
    for (int v = 0; v < 10; ++v) {
        x[v] = xyz.x[ev[v]];  // x-coordinates
        y[v] = xyz.y[ev[v]];  // y-coordinates
        z[v] = xyz.z[ev[v]];  // z-coordinates
    }

    // SUBPARAMETRIC (for iso-parametric tassellation of tet10 might be necessary)

    real_t element_field_v0_reduce = 0.0;
    real_t element_field_v1_reduce = 0.0;
    real_t element_field_v2_reduce = 0.0;
    real_t element_field_v3_reduce = 0.0;
    real_t element_field_v4_reduce = 0.0;
    real_t element_field_v5_reduce = 0.0;
    real_t element_field_v6_reduce = 0.0;
    real_t element_field_v7_reduce = 0.0;
    real_t element_field_v8_reduce = 0.0;
    real_t element_field_v9_reduce = 0.0;

    const size_t nr_warp_loop = (TET4_NQP / __WARP_SIZE__) +                //
                                ((TET4_NQP % __WARP_SIZE__) == 0 ? 0 : 1);  //

    for (size_t warp_i = 0; warp_i < nr_warp_loop; warp_i++) {
        //
        const size_t q_i = warp_i * size_t(__WARP_SIZE__) + tile_rank;

        const real_type tet4_qx_v = (q_i < TET4_NQP) ? tet4_qx[q_i] : tet4_qx[0];
        const real_type tet4_qy_v = (q_i < TET4_NQP) ? tet4_qy[q_i] : tet4_qy[0];
        const real_type tet4_qz_v = (q_i < TET4_NQP) ? tet4_qz[q_i] : tet4_qz[0];
        const real_type tet4_qw_v = (q_i < TET4_NQP) ? tet4_qw[q_i] : 0.0;

        const real_t measure = tet10_measure_cu(x, y, z, tet4_qx_v, tet4_qy_v, tet4_qz_v);

        // assert(measure > 0);
        const real_t dV = measure * tet4_qw_v;
        // printf("dV[%d]: %e\n", q, dV);

        real_t g_qx, g_qy, g_qz;
        // Transform quadrature point to physical space
        // g_qx, g_qy, g_qz are the coordinates of the quadrature point in the physical
        // space
        tet10_transform_cu(x,
                           y,
                           z,  //
                           tet4_qx_v,
                           tet4_qy_v,
                           tet4_qz_v,
                           &g_qx,
                           &g_qy,
                           &g_qz);

        tet10_dual_basis_hrt_cu(tet4_qx_v, tet4_qy_v, tet4_qz_v, tet10_f);

        ///// ======================================================

        const real_t grid_x = (g_qx - ox) / dx;
        const real_t grid_y = (g_qy - oy) / dy;
        const real_t grid_z = (g_qz - oz) / dz;

        const ptrdiff_t i = floor(grid_x);
        const ptrdiff_t j = floor(grid_y);
        const ptrdiff_t k = floor(grid_z);

        // If outside
        // if (i < 0 || j < 0 || k < 0 || (i + 1 >= n[0]) || (j + 1 >= n[1]) || (k + 1 >= n[2])) {
        //     fprintf(stderr,
        //             "warning (%g, %g, %g) (%ld, %ld, %ld) outside domain  (%ld, %ld, "
        //             "%ld)!\n",
        //             g_qx,
        //             g_qy,
        //             g_qz,
        //             i,
        //             j,
        //             k,
        //             n[0],
        //             n[1],
        //             n[2]);
        //     continue;
        // }

        // Get the reminder [0, 1]
        real_t l_x = (grid_x - (real_t)(i));
        real_t l_y = (grid_y - (real_t)(j));
        real_t l_z = (grid_z - (real_t)(k));

        // assert(l_x >= -1e-8);
        // assert(l_y >= -1e-8);
        // assert(l_z >= -1e-8);

        // assert(l_x <= 1 + 1e-8);
        // assert(l_y <= 1 + 1e-8);
        // assert(l_z <= 1 + 1e-8);

        hex_aa_8_eval_fun_cu(l_x, l_y, l_z, hex8_f);
        hex_aa_8_collect_coeffs_cu(stride0, stride1, stride2, i, j, k, data, coeffs);

        // Integrate field
        {
            real_t eval_field = 0.0;
            // UNROLL_ZERO?
            for (int edof_j = 0; edof_j < 8; edof_j++) {
                eval_field += hex8_f[edof_j] * coeffs[edof_j];
            }

            // // UNROLL_ZERO?
            // for (int edof_i = 0; edof_i < 10; edof_i++) {
            //     element_field[edof_i] += eval_field * tet10_f[edof_i] * dV;
            // }  // end edof_i loop

            element_field_v0_reduce += eval_field * tet10_f[0] * dV;
            element_field_v1_reduce += eval_field * tet10_f[1] * dV;
            element_field_v2_reduce += eval_field * tet10_f[2] * dV;
            element_field_v3_reduce += eval_field * tet10_f[3] * dV;
            element_field_v4_reduce += eval_field * tet10_f[4] * dV;
            element_field_v5_reduce += eval_field * tet10_f[5] * dV;
            element_field_v6_reduce += eval_field * tet10_f[6] * dV;
            element_field_v7_reduce += eval_field * tet10_f[7] * dV;
            element_field_v8_reduce += eval_field * tet10_f[8] * dV;
            element_field_v9_reduce += eval_field * tet10_f[9] * dV;
        }
    }  // end quadrature loop

    for (int i = tile.size() / 2; i > 0; i /= 2) {
        element_field_v0_reduce += tile.shfl_down(element_field_v0_reduce, i);
        element_field_v1_reduce += tile.shfl_down(element_field_v1_reduce, i);
        element_field_v2_reduce += tile.shfl_down(element_field_v2_reduce, i);
        element_field_v3_reduce += tile.shfl_down(element_field_v3_reduce, i);
        element_field_v4_reduce += tile.shfl_down(element_field_v4_reduce, i);
        element_field_v5_reduce += tile.shfl_down(element_field_v5_reduce, i);
        element_field_v6_reduce += tile.shfl_down(element_field_v6_reduce, i);
        element_field_v7_reduce += tile.shfl_down(element_field_v7_reduce, i);
        element_field_v8_reduce += tile.shfl_down(element_field_v8_reduce, i);
        element_field_v9_reduce += tile.shfl_down(element_field_v9_reduce, i);
    }

    // UNROLL_ZERO?

    if (tile_rank == 0) {
        atomicAdd(&weighted_field[ev[0]], element_field_v0_reduce);
        atomicAdd(&weighted_field[ev[1]], element_field_v1_reduce);
        atomicAdd(&weighted_field[ev[2]], element_field_v2_reduce);
        atomicAdd(&weighted_field[ev[3]], element_field_v3_reduce);
        atomicAdd(&weighted_field[ev[4]], element_field_v4_reduce);
        atomicAdd(&weighted_field[ev[5]], element_field_v5_reduce);
        atomicAdd(&weighted_field[ev[6]], element_field_v6_reduce);
        atomicAdd(&weighted_field[ev[7]], element_field_v7_reduce);
        atomicAdd(&weighted_field[ev[8]], element_field_v8_reduce);
        atomicAdd(&weighted_field[ev[9]], element_field_v9_reduce);
    }

}  // end kernel hex8_to_isoparametric_tet10_resample_field_local_reduce_kernel

extern "C" int hex8_to_tet10_resample_field_local_CUDA(
        // Mesh
        const ptrdiff_t nelements,  // number of elements
        const ptrdiff_t nnodes,     // number of nodes
        const idx_t** const elems,  // connectivity
        const geom_t** const xyz,   // coordinates
        // SDF
        const ptrdiff_t* const SFEM_RESTRICT n,       // number of nodes in each direction
        const ptrdiff_t* const SFEM_RESTRICT stride,  // stride of the data

        const geom_t* const SFEM_RESTRICT origin,  // origin of the domain
        const geom_t* const SFEM_RESTRICT delta,   // delta of the domain
        const real_t* const SFEM_RESTRICT data,    // SDF
        // Output //
        real_t* const SFEM_RESTRICT weighted_field) {  //

    // Device memory
    real_t* data_device = NULL;
    int size_data = n[0] * n[1] * n[2];
    hipMalloc(&data_device, size_data * sizeof(real_t));
    hipMemcpy(data_device, data, size_data * sizeof(real_t), hipMemcpyHostToDevice);

    elems_tet10_device elems_device = make_elems_tet10_device(nelements);
    copy_elems_tet10_device(nelements, &elems_device, elems);

    xyz_tet10_device xyz_device = make_xyz_tet10_device(nnodes);
    copy_xyz_tet10_device(nnodes, &xyz_device, xyz);

    // Number of threads
    const ptrdiff_t warp_per_block = 2;
    const ptrdiff_t threadsPerBlock = warp_per_block * __WARP_SIZE__;

    // Number of blocks
    const ptrdiff_t numBlocks = (nelements / warp_per_block) + (nelements % warp_per_block) + 1;

    real_t* weighted_field_device = NULL;
    hipError_t errwf = hipMalloc(&weighted_field_device, nnodes * sizeof(real_t));
    if (errwf != hipSuccess) {
        printf("Error allocating device memory for weighted_field_device: %s\n",
               hipGetErrorString(errwf));
    }

    printf("============================================================================\n");
    printf("GPU:    Launching the kernel hex8_to_tet10_resample_field_local_CUDA \n");
    printf("GPU:    Number of blocks:            %ld\n", numBlocks);
    printf("GPU:    Number of threads per block: %ld\n", threadsPerBlock);
    printf("GPU:    Total number of threads:     %ld\n", (numBlocks * threadsPerBlock));
    printf("GPU:    Number of elements:          %ld\n", nelements);
    printf("============================================================================\n");

    hipDeviceSynchronize();

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    {
        hex8_to_isoparametric_tet10_resample_field_local_reduce_kernel<<<numBlocks,
                                                                         threadsPerBlock>>>(
                0,                       //
                nelements,               //
                nnodes,                  //
                                         //
                elems_device,            //
                xyz_device,              //
                                         //
                n[0],                    //
                n[1],                    //
                n[2],                    //
                                         //
                stride[0],               //
                stride[1],               //
                stride[2],               //
                                         //
                origin[0],               //
                origin[1],               //
                origin[2],               //
                                         //
                delta[0],                //
                delta[1],                //
                delta[2],                //
                                         //
                data_device,             //
                weighted_field_device);  //
    }

    // get cuda error
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!\n");
        printf("!!!!!! Error in hex8_to_tet10_resample_field_local_CUDA: %s\n",
               hipGetErrorString(err));
        printf("!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!\n");
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    const double seconds = milliseconds / 1000.0;

    printf("============================================================================\n");
    printf("GPU:    Time for the kernel "
           "(hex8_to_isoparametric_tet10_resample_field_local_reduce_kernel): %f seconds\n",
           seconds);
    const double elements_per_second = (double)(nelements) / seconds;
    printf("GPU:    Throughput for the kernel: %e elements/second\n", elements_per_second);
    printf("============================================================================\n");

    {
        hipError_t errdd = hipFree(data_device);
        if (errdd != hipSuccess)
            printf("Error freeing device memory for data_device: %s\n", hipGetErrorString(errdd));
    }

    free_elems_tet10_device(elems_device);

    hipMemcpy(weighted_field,
               weighted_field_device,  //
               nnodes * sizeof(real_t),
               hipMemcpyDeviceToHost);

    hipError_t errwf2 = hipFree(weighted_field_device);
    if (errwf2 != hipSuccess) {
        printf("Error freeing device memory for weighted_field_device: %s\n",
               hipGetErrorString(errwf2));
    }
    weighted_field_device = NULL;

    return 0;
}
