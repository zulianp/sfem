#include <hip/hip_cooperative_groups.h>
#include <hip/hip_runtime_api.h>
#include <sfem_base.h>
#include <stdio.h>

// #define real_type real_t

#include "mesh_aura.h"
#include "sfem_mesh.h"

#include "tet10_weno_cuda.cuh"

#include "quadratures_rule_cuda.h"
#include "tet10_resample_field.cuh"

#include "tet10_resample_field_kernels.cuh"

#define MY_RESTRICT __restrict__

#define __WARP_SIZE__ 32
#define WENO_CUDA 1

#if SFEM_TET10_WENO == ON
#define CUBE1 1
#else
#define CUBE1 0
#endif

/**
 * @brief lanches the kernels sequentially to
 * resample the field from hex8 to tet10 by applying all the necessary steps
 *
 * @param numBlocks
 * @param threadsPerBlock
 * @param nelements
 * @param nnodes
 * @param elems_device
 * @param xyz_device
 * @param n
 * @param stride
 * @param origin
 * @param delta
 * @param data_device
 * @param weighted_field_device
 * @param mass_vector
 * @param g_device
 * @return int
 */
int                                                                                               //
launch_kernels_hex8_to_tet10_resample_field_local_CUDA(const int numBlocks,                       //
                                                       const int threadsPerBlock,                 //
                                                       const int bool_assemble_dual_mass_vector,  // assemble dual mass vector
                                                       int       nelements,                       //
                                                       ptrdiff_t nnodes,                          //
                                                       elems_tet10_device                   elems_device,  //
                                                       xyz_tet10_device                     xyz_device,    //
                                                       const ptrdiff_t* const SFEM_RESTRICT n,             //
                                                       const ptrdiff_t* const SFEM_RESTRICT stride,        //
                                                       const geom_t* const SFEM_RESTRICT    origin,        //
                                                       const geom_t* const SFEM_RESTRICT    delta,         //
                                                       const real_t*                        data_device,   //
                                                       real_t*                              mass_vector,   //
                                                       real_t*                              g_device) {                                 //
    //
    PRINT_CURRENT_FUNCTION;

    // Set to zero the mass vector
    hipMemset(mass_vector, 0, nnodes * sizeof(real_t));

    // Launch the appropriate resample field kernel based on CUBE1
#if CUBE1 == 1  // WENO
    hex8_to_isoparametric_tet10_resample_field_local_cube1_kernel
#else
    hex8_to_isoparametric_tet10_resample_field_local_reduce_kernel
#endif
            <<<numBlocks, threadsPerBlock>>>(0,
                                             nelements,
                                             nnodes,
                                             elems_device,
                                             xyz_device,
                                             n[0],
                                             n[1],
                                             n[2],
                                             stride[0],
                                             stride[1],
                                             stride[2],
                                             origin[0],
                                             origin[1],
                                             origin[2],
                                             delta[0],
                                             delta[1],
                                             delta[2],
                                             data_device,
                                             g_device);

    // Synchronize device
    hipDeviceSynchronize();

    if (bool_assemble_dual_mass_vector == 1) {
        // Launch isoparametric_tet10_assemble_dual_mass_vector_kernel
        isoparametric_tet10_assemble_dual_mass_vector_kernel<<<numBlocks, threadsPerBlock>>>(
                0, nelements, nnodes, elems_device, xyz_device, mass_vector);

        // Synchronize device
        hipDeviceSynchronize();

        compute_g_kernel_v2<<<(nnodes / threadsPerBlock) + 1, threadsPerBlock>>>(nnodes, mass_vector, g_device);

        // Synchronize device
        hipDeviceSynchronize();
    }

    RETURN_FROM_FUNCTION(0);
}

/**
 * @brief lanches the kernels sequentially to
 * resample the field from hex8 to tet10 by applying all the necessary steps for the cases where unified and Managed memory is
 * used so that the MPI communication is handled directely from / to the device.
 *
 * @param numBlocks
 * @param threadsPerBlock
 * @param nelements
 * @param nnodes
 * @param elems_device
 * @param xyz_device
 * @param n
 * @param stride
 * @param origin
 * @param delta
 * @param data_device
 * @param weighted_field_device
 * @param mass_vector
 * @param g_device
 * @return int
 */
int                                                                           //
launch_kernels_hex8_to_tet10_resample_field_local_CUDA_unified(               //
        const int                            mpi_size,                        //
        const int                            mpi_rank,                        //
        const int                            numBlocks,                       //
        const int                            threadsPerBlock,                 //
        mesh_t*                              mesh,                            // Mesh
        const int                            bool_assemble_dual_mass_vector,  // assemble dual mass vector
        int                                  nelements,                       //
        ptrdiff_t                            nnodes,                          //
        elems_tet10_device                   elems_device,                    //
        xyz_tet10_device                     xyz_device,                      //
        const ptrdiff_t* const SFEM_RESTRICT n,                               //
        const ptrdiff_t* const SFEM_RESTRICT stride,                          //
        const geom_t* const SFEM_RESTRICT    origin,                          //
        const geom_t* const SFEM_RESTRICT    delta,                           //
        const real_t*                        data_device,                     //
        real_t*                              mass_vector,                     //
        real_t*                              g_device) {
    //
    PRINT_CURRENT_FUNCTION;

    // Set to zero the mass vector
    hipMemset(mass_vector, 0, nnodes * sizeof(real_t));

    // Launch the appropriate resample field kernel based on CUBE1
#if CUBE1 == 1  // WENO
    hex8_to_isoparametric_tet10_resample_field_local_cube1_kernel
#else
    hex8_to_isoparametric_tet10_resample_field_local_reduce_kernel
#endif
            <<<numBlocks, threadsPerBlock>>>(0,
                                             nelements,
                                             nnodes,
                                             elems_device,
                                             xyz_device,
                                             n[0],
                                             n[1],
                                             n[2],
                                             stride[0],
                                             stride[1],
                                             stride[2],
                                             origin[0],
                                             origin[1],
                                             origin[2],
                                             delta[0],
                                             delta[1],
                                             delta[2],
                                             data_device,
                                             g_device);

    // Synchronize device
    hipDeviceSynchronize();

    if (bool_assemble_dual_mass_vector == 1) {
        // Launch isoparametric_tet10_assemble_dual_mass_vector_kernel
        isoparametric_tet10_assemble_dual_mass_vector_kernel<<<numBlocks, threadsPerBlock>>>(
                0, nelements, nnodes, elems_device, xyz_device, mass_vector);

        // Synchronize device
        hipDeviceSynchronize();

        if (mpi_size > 1) {
            printf("MPI:    Launching the exchange, %s:%d\n", __FILE__, __LINE__);
            send_recv_t slave_to_master;
            mesh_create_nodal_send_recv(mesh, &slave_to_master);

            ptrdiff_t count       = mesh_exchange_master_buffer_count(&slave_to_master);
            real_t*   real_buffer = (real_t*)malloc(count * sizeof(real_t));

            exchange_add(mesh, &slave_to_master, mass_vector, real_buffer);
            exchange_add(mesh, &slave_to_master, g_device, real_buffer);

            free(real_buffer);
            send_recv_destroy(&slave_to_master);
        }

        // // Launch compute_g_kernel
        // compute_g_kernel<<<(nnodes / threadsPerBlock) + 1, threadsPerBlock>>>(
        //         nnodes, weighted_field_device, mass_vector, g_device);

        compute_g_kernel_v2<<<(nnodes / threadsPerBlock) + 1, threadsPerBlock>>>(nnodes, mass_vector, g_device);

        // Synchronize device
        hipDeviceSynchronize();
    }

    RETURN_FROM_FUNCTION(0);
}

////////////////////////////////////////////////////////////////////////
// calculate_threads_and_blocks
// Function to calculate the number of threads and blocks
// based on the number of elements and the number of warps per block
////////////////////////////////////////////////////////////////////////
void                                                      //
calculate_threads_and_blocks(ptrdiff_t  nelements,        //
                             ptrdiff_t  warp_per_block,   //
                             ptrdiff_t* threadsPerBlock,  //
                             ptrdiff_t* numBlocks) {      //

    *threadsPerBlock = warp_per_block * __WARP_SIZE__;
    *numBlocks       = (nelements / warp_per_block) + (nelements % warp_per_block) + 1;
}

////////////////////////////////////////////////////////////////////////
// hex8_to_tet10_resample_field_local_CUDA_unified
////////////////////////////////////////////////////////////////////////
extern "C" int                                                        //
hex8_to_tet10_resample_field_local_CUDA_unified(                      //
                                                                      // Mesh
        const ptrdiff_t              nelements,                       // number of elements
        const ptrdiff_t              nnodes,                          // number of nodes
        const int                    bool_assemble_dual_mass_vector,  // assemble dual mass vector
        idx_t** const SFEM_RESTRICT  elems,                           // connectivity
        geom_t** const SFEM_RESTRICT xyz,                             // coordinates
        // SDF
        const ptrdiff_t* const SFEM_RESTRICT n,       // number of nodes in each direction
        const ptrdiff_t* const SFEM_RESTRICT stride,  // stride of the data

        const geom_t* const SFEM_RESTRICT origin,  // origin of the domain
        const geom_t* const SFEM_RESTRICT delta,   // delta of the domain
        const real_t* const SFEM_RESTRICT data,    // SDF
        // Output //
        real_t* const SFEM_RESTRICT g_host) {  //
                                               //
    PRINT_CURRENT_FUNCTION;

    int size_data = n[0] * n[1] * n[2];

    // Device memory
    const real_t* data_device = data;
    real_t*       mass_vector = NULL;
    real_t*       g_device    = g_host;

    memory_hint_write_mostly(nelements, sizeof(real_t), (void*)g_device);
    memory_hint_read_mostly(size_data, sizeof(real_t), (void*)data_device);

    mass_vector = (real_t*)malloc(nnodes * sizeof(real_t));

    memory_hint_write_mostly(nnodes, sizeof(real_t), (void*)mass_vector);

    //// Initialize the data on the device
    elems_tet10_device elems_device =                    //
            make_elems_tet10_device_unified(nelements);  //

    copy_elems_tet10_device_unified(nelements, &elems_device, (const idx_t**)elems);
    memory_hint_elems_tet10_device_unified(nelements, &elems_device);

    xyz_tet10_device xyz_device =                   //
            make_xyz_tet10_device_unified(nnodes);  //

    copy_xyz_tet10_device_unified(nnodes, &xyz_device, (const float**)xyz);
    memory_hint_xyz_tet10_device_unified(nnodes, &xyz_device);

    const ptrdiff_t warp_per_block  = 8;  /// 8 warps per block /////
    ptrdiff_t       threadsPerBlock = 0;
    ptrdiff_t       numBlocks       = 0;

    calculate_threads_and_blocks(nelements, warp_per_block, &threadsPerBlock, &numBlocks);

#if CUBE1 == 0  // WENO ..
    char kernel_name[] = "hex8_to_isoparametric_tet10_resample_field_local_reduce_kernel";
#else
    char kernel_name[] = "hex8_to_isoparametric_tet10_resample_field_local_cube1_kernel";
#endif

    printf("============================================================================\n");
    printf("GPU:    Unified Memory Model\n");
    printf("GPU:    Launching the kernel %s \n", kernel_name);
    printf("GPU:    Number of blocks:            %ld\n", numBlocks);
    printf("GPU:    Number of threads per block: %ld\n", threadsPerBlock);
    printf("GPU:    Total number of threads:     %ld\n", (numBlocks * threadsPerBlock));
    printf("GPU:    Number of elements:          %ld\n", nelements);
    printf("GPU:    Use WENO:                    %s\n", (WENO_CUDA == 1 & CUBE1 == 1) ? "Yes" : "No");
    printf("============================================================================\n");

    hipDeviceSynchronize();

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    // Launch the kernels
    launch_kernels_hex8_to_tet10_resample_field_local_CUDA(numBlocks,
                                                           threadsPerBlock,
                                                           bool_assemble_dual_mass_vector,
                                                           nelements,
                                                           nnodes,
                                                           elems_device,
                                                           xyz_device,
                                                           n,
                                                           stride,
                                                           origin,
                                                           delta,
                                                           data_device,
                                                           mass_vector,
                                                           g_device);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    const double seconds = milliseconds / 1000.0;

    printf("============================================================================\n");
    printf("GPU:    Time for the kernel (%s):\n"  //
           "GPU:    %f seconds\n",                //
           kernel_name,
           seconds);
    const double elements_per_second = (double)(nelements) / seconds;
    printf("GPU:    Number of elements: %d.\n", nelements);
    printf("GPU:    Throughput for the kernel: %e elements/second\n", elements_per_second);
    printf("GPU:    %d, %f   (CSV friendly) \n", nelements, elements_per_second);
    printf("============================================================================\n");

    ////////////////////////////////////////
    /// Finalize the memory allocation
    free(mass_vector);
    mass_vector = NULL;  // free the memory allocated for mass_vector

    // The g device is already allocated in the unified memory
    // and managed by the main program
    g_device = NULL;

    free_elems_tet10_device_unified(&elems_device);
    free_xyz_tet10_device_unified(&xyz_device);

    RETURN_FROM_FUNCTION(0);
}

////////////////////////////////////////////////////////////////////////
// hex8_to_tet10_resample_field_local_CUDA_unified
////////////////////////////////////////////////////////////////////////
extern "C" int                                       //
hex8_to_tet10_resample_field_local_CUDA_unified_v2(  //
        const int mpi_size,                          //
        const int mpi_rank,                          //
        mesh_t*   mesh,                              // Mesh
        const int bool_assemble_dual_mass_vector,    // assemble dual mass vector
        // SDF
        const ptrdiff_t* const SFEM_RESTRICT n,       // number of nodes in each direction
        const ptrdiff_t* const SFEM_RESTRICT stride,  // stride of the data
        // Geometry
        const geom_t* const SFEM_RESTRICT origin,  // origin of the domain
        const geom_t* const SFEM_RESTRICT delta,   // delta of the domain
        // Data
        const real_t* const SFEM_RESTRICT data,  // SDF
        // Output //
        real_t* const SFEM_RESTRICT g_host) {  //
                                               //
    PRINT_CURRENT_FUNCTION;

    int size_data = n[0] * n[1] * n[2];

    // Device memory
    const real_t* data_device = data;
    real_t*       mass_vector = NULL;
    real_t*       g_device    = g_host;

    memory_hint_write_mostly(mesh->nelements, sizeof(real_t), (void*)g_device);
    memory_hint_read_mostly(size_data, sizeof(real_t), (void*)data_device);

    mass_vector = (real_t*)malloc(mesh->nnodes * sizeof(real_t));
    memory_hint_write_mostly(mesh->nnodes, sizeof(real_t), (void*)mass_vector);

    //// Initialize the data on the device
    elems_tet10_device elems_device =                          //
            make_elems_tet10_device_unified(mesh->nelements);  //

    copy_elems_tet10_device_unified(mesh->nelements, &elems_device, (const idx_t**)mesh->elements);
    memory_hint_elems_tet10_device_unified(mesh->nelements, &elems_device);

    xyz_tet10_device xyz_device =                         //
            make_xyz_tet10_device_unified(mesh->nnodes);  //

    copy_xyz_tet10_device_unified(mesh->nnodes, &xyz_device, (const float**)mesh->points);
    memory_hint_xyz_tet10_device_unified(mesh->nnodes, &xyz_device);

    const ptrdiff_t warp_per_block  = 8;  /// 8 warps per block /////
    ptrdiff_t       threadsPerBlock = 0;
    ptrdiff_t       numBlocks       = 0;

    calculate_threads_and_blocks(mesh->nelements, warp_per_block, &threadsPerBlock, &numBlocks);

#if CUBE1 == 0  // WENO ..
    char kernel_name[] = "hex8_to_isoparametric_tet10_resample_field_local_reduce_kernel";
#else
    char kernel_name[] = "hex8_to_isoparametric_tet10_resample_field_local_cube1_kernel";
#endif

    printf("============================================================================\n");
    printf("GPU:    Unified Memory Model V2 %s:%d \n", __FILE__, __LINE__);
    printf("GPU:    Mpi size:                    %d\n", mpi_size);
    printf("GPU:    Mpi rank:                    %d\n", mpi_rank);
    printf("GPU:    Launching the kernel %s \n", kernel_name);
    printf("GPU:    Number of blocks:            %ld\n", numBlocks);
    printf("GPU:    Number of threads per block: %ld\n", threadsPerBlock);
    printf("GPU:    Total number of threads:     %ld\n", (numBlocks * threadsPerBlock));
    printf("GPU:    Number of elements:          %ld\n", mesh->nelements);
    printf("GPU:    Use WENO:                    %s\n", (WENO_CUDA == 1 & CUBE1 == 1) ? "Yes" : "No");
    printf("============================================================================\n");

    hipDeviceSynchronize();

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    ////  Launch the kernel
    launch_kernels_hex8_to_tet10_resample_field_local_CUDA_unified(mpi_size,
                                                                   mpi_rank,
                                                                   numBlocks,
                                                                   threadsPerBlock,
                                                                   mesh,
                                                                   bool_assemble_dual_mass_vector,
                                                                   mesh->nelements,
                                                                   mesh->nnodes,
                                                                   elems_device,
                                                                   xyz_device,
                                                                   n,
                                                                   stride,
                                                                   origin,
                                                                   delta,
                                                                   data_device,
                                                                   mass_vector,
                                                                   g_device);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    const double seconds = milliseconds / 1000.0;

    printf("============================================================================\n");
    printf("GPU:    Time for the kernel (%s):\n"  //
           "GPU:    %f seconds\n",                //
           kernel_name,
           seconds);
    const double elements_per_second = (double)(mesh->nelements) / seconds;
    printf("GPU:    Number of elements: %d.\n", mesh->nelements);
    printf("GPU:    Throughput for the kernel: %e elements/second\n", elements_per_second);
    printf("GPU:    %d, %f   (CSV friendly) \n", mesh->nelements, elements_per_second);
    printf("============================================================================\n");

    ////////////////////////////////////////
    /// Finalize the memory allocation
    free(mass_vector);
    mass_vector = NULL;  // free the memory allocated for mass_vector

    // The g device is already allocated in the unified memory
    // and managed by the main program
    g_device = NULL;

    free_elems_tet10_device_unified(&elems_device);
    free_xyz_tet10_device_unified(&xyz_device);

    RETURN_FROM_FUNCTION(0);
}

////////////////////////////////////////////////////////////////////////
// hex8_to_tet10_resample_field_local_CUDA
////////////////////////////////////////////////////////////////////////
extern "C" int                                                        //
hex8_to_tet10_resample_field_local_CUDA(                              //
                                                                      // Mesh
        const ptrdiff_t              nelements,                       // number of elements
        const ptrdiff_t              nnodes,                          // number of nodes
        const int                    bool_assemble_dual_mass_vector,  // assemble dual mass vector
        idx_t** const SFEM_RESTRICT  elems,                           // connectivity
        geom_t** const SFEM_RESTRICT xyz,                             // coordinates
        // SDF
        const ptrdiff_t* const SFEM_RESTRICT n,       // number of nodes in each direction
        const ptrdiff_t* const SFEM_RESTRICT stride,  // stride of the data

        const geom_t* const SFEM_RESTRICT origin,  // origin of the domain
        const geom_t* const SFEM_RESTRICT delta,   // delta of the domain
        const real_t* const SFEM_RESTRICT data,    // SDF
        // Output //
        real_t* const SFEM_RESTRICT g_host) {  //
                                               // geom_t** const SFEM_RESTRICT xyz

#if SFEM_CUDA_MEMORY_MODEL == CUDA_UNIFIED_MEMORY
#pragma message "CUDA_UNIFIED_MEMORY is enabled"
    return hex8_to_tet10_resample_field_local_CUDA_unified(nelements,  //
                                                           nnodes,
                                                           bool_assemble_dual_mass_vector,
                                                           elems,
                                                           xyz,
                                                           n,
                                                           stride,
                                                           origin,
                                                           delta,
                                                           data,
                                                           g_host);
#elif SFEM_CUDA_MEMORY_MODEL == CUDA_MEMORY_MANAGED
#pragma message "CUDA_MEMORY_MANAGED is enabled: Not implemented yet"
    return -1;
#endif

    PRINT_CURRENT_FUNCTION;

    // Device memory
    real_t* data_device = NULL;
    real_t* mass_vector = NULL;
    real_t* g_device    = NULL;

    int size_data = n[0] * n[1] * n[2];
    hipMalloc(&data_device, size_data * sizeof(real_t));
    hipMemcpy(data_device, data, size_data * sizeof(real_t), hipMemcpyHostToDevice);

    elems_tet10_device elems_device = make_elems_tet10_device(nelements);
    copy_elems_tet10_device(nelements, &elems_device, (const idx_t**)elems);

    xyz_tet10_device xyz_device = make_xyz_tet10_device(nnodes);
    copy_xyz_tet10_device(nnodes, &xyz_device, (const float**)xyz);

    // Number of threads
    const ptrdiff_t warp_per_block  = 8;
    const ptrdiff_t threadsPerBlock = warp_per_block * __WARP_SIZE__;

    // Number of blocks
    const ptrdiff_t numBlocks = (nelements / warp_per_block) + (nelements % warp_per_block) + 1;

    hipMalloc(&mass_vector, nnodes * sizeof(real_t));
    hipMalloc(&g_device, nnodes * sizeof(real_t));

#if CUBE1 == 0  // WENO ..
    char kernel_name[] = "hex8_to_isoparametric_tet10_resample_field_local_reduce_kernel";
#else
    char kernel_name[] = "hex8_to_isoparametric_tet10_resample_field_local_cube1_kernel";
#endif

    printf("============================================================================\n");
    printf("GPU:    Host Memory Model [Default] \n");
    printf("GPU:    Launching the kernel %s \n", kernel_name);
    printf("GPU:    Number of blocks:            %ld\n", numBlocks);
    printf("GPU:    Number of threads per block: %ld\n", threadsPerBlock);
    printf("GPU:    Total number of threads:     %ld\n", (numBlocks * threadsPerBlock));
    printf("GPU:    Number of elements:          %ld\n", nelements);
    printf("GPU:    Use WENO:                    %s\n", (WENO_CUDA == 1 & CUBE1 == 1) ? "Yes" : "No");
    printf("============================================================================\n");

    hipDeviceSynchronize();

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    // Launch the kernels
    launch_kernels_hex8_to_tet10_resample_field_local_CUDA(numBlocks,
                                                           threadsPerBlock,
                                                           bool_assemble_dual_mass_vector,
                                                           nelements,
                                                           nnodes,
                                                           elems_device,
                                                           xyz_device,
                                                           n,
                                                           stride,
                                                           origin,
                                                           delta,
                                                           data_device,
                                                           //    weighted_field_device,
                                                           mass_vector,
                                                           g_device);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    const double seconds = milliseconds / 1000.0;

    printf("============================================================================\n");
    printf("GPU:    Time for the kernel (%s):\n"  //
           "GPU:    %f seconds\n",                //
           kernel_name,
           seconds);
    const double elements_per_second = (double)(nelements) / seconds;
    printf("GPU:    Number of elements: %d.\n", nelements);
    printf("GPU:    Throughput for the kernel: %e elements/second\n", elements_per_second);
    printf("GPU:    %d, %f   (CSV friendly) \n", nelements, elements_per_second);
    printf("============================================================================\n");

    {
        hipError_t errdd = hipFree(data_device);
        if (errdd != hipSuccess) printf("Error freeing device memory for data_device: %s\n", hipGetErrorString(errdd));
    }

    free_elems_tet10_device(&elems_device);
    free_xyz_tet10_device(&xyz_device);

    hipMemcpy(g_host,                   //
               g_device,                 //
               nnodes * sizeof(real_t),  //
               hipMemcpyDeviceToHost);  //

    hipError_t errg = hipFree(g_device);
    if (errg != hipSuccess) {
        printf("Error freeing device memory for g_device: %s\n", hipGetErrorString(errg));
    }
    g_device = NULL;

    hipError_t errmv = hipFree(mass_vector);
    if (errmv != hipSuccess) {
        printf("Error freeing device memory for mass_vector: %s\n", hipGetErrorString(errmv));
    }
    mass_vector = NULL;

    RETURN_FROM_FUNCTION(0);
    // return 0;
}

////////////////////////////////////////////////////////////////////////
// hex8_to_tet10_resample_field_local_CUDA_wrapper
////////////////////////////////////////////////////////////////////////
extern "C" int                                     //
hex8_to_tet10_resample_field_local_CUDA_wrapper(   //
        const int mpi_size,                        // MPI size
        const int mpi_rank,                        // MPI rank
                                                   // Mesh
        mesh_t*   mesh,                            // Mesh
        const int bool_assemble_dual_mass_vector,  // assemble dual mass vector
        // SDF
        const ptrdiff_t* const SFEM_RESTRICT n,       // number of nodes in each direction
        const ptrdiff_t* const SFEM_RESTRICT stride,  // stride of the data

        const geom_t* const SFEM_RESTRICT origin,  // origin of the domain
        const geom_t* const SFEM_RESTRICT delta,   // delta of the domain
        const real_t* const SFEM_RESTRICT data,    // SDF
        // Output //
        real_t* const SFEM_RESTRICT g_host) {  //

#if SFEM_CUDA_MEMORY_MODEL == CUDA_UNIFIED_MEMORY

#pragma message "CUDA_UNIFIED_MEMORY is enabled"
    return hex8_to_tet10_resample_field_local_CUDA_unified_v2(mpi_size,                        //
                                                              mpi_rank,                        //
                                                              mesh,                            //
                                                              bool_assemble_dual_mass_vector,  //
                                                              n,
                                                              stride,
                                                              origin,
                                                              delta,
                                                              data,
                                                              g_host);
#elif SFEM_CUDA_MEMORY_MODEL == CUDA_MANAGED_MEMORY

#pragma message "CUDA_MEMORY_MANAGED is enabled: Not implemented yet"
    return -1;

#elif SFEM_CUDA_MEMORY_MODEL == CUDA_HOST_MEMORY

    // Default memory model is CUDA_HOST_MEMORY.

    return hex8_to_tet10_resample_field_local_CUDA(mesh->nelements,                 //
                                                   mesh->nnodes,                    //
                                                   bool_assemble_dual_mass_vector,  //
                                                   mesh->elements,                  //
                                                   mesh->points,                    //
                                                   n,                               //
                                                   stride,                          //
                                                   origin,                          //
                                                   delta,                           //
                                                   data,                            //
                                                   g_host);                         //

#endif
}
