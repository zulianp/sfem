#include "sfem_resample_field_cuda_fun.cuh"

#include <hip/hip_runtime.h>
#include <stdio.h>
#include "sfem_base.h"

/**
 * @brief Make the elements struct
 *
 * @return elems_tet4_device
 */
elems_tet4_device           //
make_elems_tet4_device() {  //

    elems_tet4_device elems_device;

    elems_device.elems_v0 = NULL;
    elems_device.elems_v1 = NULL;
    elems_device.elems_v2 = NULL;
    elems_device.elems_v3 = NULL;

    return elems_device;
}

/**
 * @brief Functions for the elements struct
 *
 * @param elems_device
 * @param nelements
 */
void                                                              //
cuda_allocate_elems_tet4_device(elems_tet4_device* elems_device,  //
                                const ptrdiff_t    nelements) {      //

    hipMalloc((void**)&elems_device->elems_v0, nelements * sizeof(idx_t));
    hipMalloc((void**)&elems_device->elems_v1, nelements * sizeof(idx_t));
    hipMalloc((void**)&elems_device->elems_v2, nelements * sizeof(idx_t));
    hipMalloc((void**)&elems_device->elems_v3, nelements * sizeof(idx_t));
}

void                                                                    //
cuda_allocate_elems_tet4_device_async(elems_tet4_device* elems_device,  //
                                      const ptrdiff_t    nelements,     //
                                      hipStream_t       stream) {            //

    hipMallocAsync((void**)&elems_device->elems_v0, nelements * sizeof(idx_t), stream);
    hipMallocAsync((void**)&elems_device->elems_v1, nelements * sizeof(idx_t), stream);
    hipMallocAsync((void**)&elems_device->elems_v2, nelements * sizeof(idx_t), stream);
    hipMallocAsync((void**)&elems_device->elems_v3, nelements * sizeof(idx_t), stream);
}

/**
 * @brief Functions for the elements struct
 *
 * @param elems_device
 * @param nelements
 */
void                                                                      //
cuda_allocate_elems_tet4_device_managed(elems_tet4_device* elems_device,  //
                                        const ptrdiff_t    nelements) {      //

    hipMallocManaged((void**)&elems_device->elems_v0, nelements * sizeof(idx_t));
    hipMallocManaged((void**)&elems_device->elems_v1, nelements * sizeof(idx_t));
    hipMallocManaged((void**)&elems_device->elems_v2, nelements * sizeof(idx_t));
    hipMallocManaged((void**)&elems_device->elems_v3, nelements * sizeof(idx_t));
}

/**
 * @brief
 *
 * @param elems
 * @param nelements
 * @param elems_device
 */
void                                                       //
copy_elems_tet4_device(const idx_t**      elems,           // elements from host
                       const ptrdiff_t    nelements,       // number of elements
                       elems_tet4_device* elems_device) {  // to device

    hipMemcpy(elems_device->elems_v0, elems[0], nelements * sizeof(idx_t), hipMemcpyHostToDevice);
    hipMemcpy(elems_device->elems_v1, elems[1], nelements * sizeof(idx_t), hipMemcpyHostToDevice);
    hipMemcpy(elems_device->elems_v2, elems[2], nelements * sizeof(idx_t), hipMemcpyHostToDevice);
    hipMemcpy(elems_device->elems_v3, elems[3], nelements * sizeof(idx_t), hipMemcpyHostToDevice);
}

void                                                           //
copy_elems_tet4_device_async(const idx_t**      elems,         // elements from host
                             const ptrdiff_t    nelements,     // number of elements
                             elems_tet4_device* elems_device,  // to device
                             hipStream_t       stream) {            // stream

    hipMemcpyAsync(elems_device->elems_v0, elems[0], nelements * sizeof(idx_t), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(elems_device->elems_v1, elems[1], nelements * sizeof(idx_t), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(elems_device->elems_v2, elems[2], nelements * sizeof(idx_t), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(elems_device->elems_v3, elems[3], nelements * sizeof(idx_t), hipMemcpyHostToDevice, stream);
}

/**
 * @brief
 *
 * @param elems
 * @param nelements
 * @param elems_device
 */
void                                                               //
copy_elems_tet4_device_unified(const idx_t**      elems,           // elements from host
                               const ptrdiff_t    nelements,       // number of elements
                               elems_tet4_device* elems_device) {  // to device

    elems_device->elems_v0 = (idx_t*)elems[0];
    elems_device->elems_v1 = (idx_t*)elems[1];
    elems_device->elems_v2 = (idx_t*)elems[2];
    elems_device->elems_v3 = (idx_t*)elems[3];
}

/**
 * @brief Free memory for the elements struct
 *
 * @param elems_device
 */
void                                                       //
free_elems_tet4_device(elems_tet4_device* elems_device) {  //

    hipFree(elems_device->elems_v0);
    hipFree(elems_device->elems_v1);
    hipFree(elems_device->elems_v2);
    hipFree(elems_device->elems_v3);

    elems_device->elems_v0 = NULL;
    elems_device->elems_v1 = NULL;
    elems_device->elems_v2 = NULL;
    elems_device->elems_v3 = NULL;
}

void                                                           //
free_elems_tet4_device_async(elems_tet4_device* elems_device,  //
                             hipStream_t       stream) {            //
    hipFreeAsync(elems_device->elems_v0, stream);
    hipFreeAsync(elems_device->elems_v1, stream);
    hipFreeAsync(elems_device->elems_v2, stream);
    hipFreeAsync(elems_device->elems_v3, stream);
    elems_device->elems_v0 = NULL;
    elems_device->elems_v1 = NULL;
    elems_device->elems_v2 = NULL;
    elems_device->elems_v3 = NULL;
}

/**
 * @brief Free memory for the elements struct
 * @brief This function is used when the memory is allocated in the unified memory
 * It simply sets the pointers to NULL (since the memory is not managed by CUDA)
 *
 * @param elems_device
 */
void                                                               //
free_elems_tet4_device_unified(elems_tet4_device* elems_device) {  //
    elems_device->elems_v0 = NULL;
    elems_device->elems_v1 = NULL;
    elems_device->elems_v2 = NULL;
    elems_device->elems_v3 = NULL;
}

///////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////
/// Struct for xyz
///////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////

/**
 * @brief Allocate memory for the xyz struct
 *
 * @param xyz_device
 * @param nnodes
 */
void                                                        //
cuda_allocate_xyz_tet4_device(xyz_tet4_device* xyz_device,  //
                              const ptrdiff_t  nnodes) {     //

    hipMalloc((void**)&xyz_device->x, nnodes * sizeof(geom_t));
    hipMalloc((void**)&xyz_device->y, nnodes * sizeof(geom_t));
    hipMalloc((void**)&xyz_device->z, nnodes * sizeof(geom_t));
}

void                                                              //
cuda_allocate_xyz_tet4_device_async(xyz_tet4_device* xyz_device,  //
                                    const ptrdiff_t  nnodes,      //
                                    hipStream_t     stream) {        //

    hipMallocAsync((void**)&xyz_device->x, nnodes * sizeof(geom_t), stream);
    hipMallocAsync((void**)&xyz_device->y, nnodes * sizeof(geom_t), stream);
    hipMallocAsync((void**)&xyz_device->z, nnodes * sizeof(geom_t), stream);
}

/**
 * @brief Allocate managed memory for the xyz struct
 *
 * @param xyz_device
 * @param nnodes
 */
void                                                                //
cuda_allocate_xyz_tet4_device_managed(xyz_tet4_device* xyz_device,  //
                                      const ptrdiff_t  nnodes) {     //

    hipError_t err0 = hipMallocManaged((void**)&xyz_device->x, nnodes * sizeof(geom_t));
    hipError_t err1 = hipMallocManaged((void**)&xyz_device->y, nnodes * sizeof(geom_t));
    hipError_t err2 = hipMallocManaged((void**)&xyz_device->z, nnodes * sizeof(geom_t));

    if (err0 != hipSuccess || err1 != hipSuccess || err2 != hipSuccess) {
        fprintf(stderr,
                "Failed to allocate managed memory (error codes: %s, %s, %s)!\n",
                hipGetErrorString(err0),
                hipGetErrorString(err1),
                hipGetErrorString(err2));
        exit(EXIT_FAILURE);
    }
}

/**
 * @brief   Make the xyz struct
 *
 * @return xyz_tet4_device
 */
xyz_tet4_device           //
make_xyz_tet4_device() {  //

    xyz_tet4_device xyz_device;

    xyz_device.x = NULL;
    xyz_device.y = NULL;
    xyz_device.z = NULL;

    return xyz_device;
}

/**
 * @brief Free memory for the xyz struct
 *
 * @param xyz_device
 */
void                                                 //
free_xyz_tet4_device(xyz_tet4_device* xyz_device) {  //
    hipFree(xyz_device->x);
    hipFree(xyz_device->y);
    hipFree(xyz_device->z);

    xyz_device->x = NULL;
    xyz_device->y = NULL;
    xyz_device->z = NULL;
}

void                                                     //
free_xyz_tet4_device_async(xyz_tet4_device* xyz_device,  //
                           hipStream_t     stream) {        //

    hipFreeAsync(xyz_device->x, stream);
    hipFreeAsync(xyz_device->y, stream);
    hipFreeAsync(xyz_device->z, stream);

    xyz_device->x = NULL;
    xyz_device->y = NULL;
    xyz_device->z = NULL;
}

/**
 * @brief Free memory for the xyz struct
 * @brief This function is used when the memory is allocated in the unified memory
 * It simply sets the pointers to NULL (since the memory is not managed by CUDA)
 *
 * @param xyz_device
 */
void                                                         //
free_xyz_tet4_device_unified(xyz_tet4_device* xyz_device) {  //
    xyz_device->x = NULL;
    xyz_device->y = NULL;
    xyz_device->z = NULL;
}

/**
 * @brief Copy the xyz struct from host to device
 *
 * @param xyz
 * @param nnodes
 * @param xyz_device
 */
void                                                 //
copy_xyz_tet4_device(const geom_t**   xyz,           //
                     const ptrdiff_t  nnodes,        //
                     xyz_tet4_device* xyz_device) {  //
                                                     //
    hipMemcpy(xyz_device->x, xyz[0], nnodes * sizeof(geom_t), hipMemcpyHostToDevice);
    hipMemcpy(xyz_device->y, xyz[1], nnodes * sizeof(geom_t), hipMemcpyHostToDevice);
    hipMemcpy(xyz_device->z, xyz[2], nnodes * sizeof(geom_t), hipMemcpyHostToDevice);
}

void                                                     //
copy_xyz_tet4_device_async(const geom_t**   xyz,         //
                           const ptrdiff_t  nnodes,      //
                           xyz_tet4_device* xyz_device,  //
                           hipStream_t     stream) {        //

    hipMemcpyAsync(xyz_device->x, xyz[0], nnodes * sizeof(geom_t), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(xyz_device->y, xyz[1], nnodes * sizeof(geom_t), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(xyz_device->z, xyz[2], nnodes * sizeof(geom_t), hipMemcpyHostToDevice, stream);
}

/**
 * @brief Copy the xyz struct from host to device
 *
 * @param xyz
 * @param nnodes
 * @param xyz_device
 */
void                                                         //
copy_xyz_tet4_device_unified(const geom_t**   xyz,           //
                             const ptrdiff_t  nnodes,        //
                             xyz_tet4_device* xyz_device) {  //
                                                             //
    xyz_device->x = (geom_t*)xyz[0];
    xyz_device->y = (geom_t*)xyz[1];
    xyz_device->z = (geom_t*)xyz[2];
}