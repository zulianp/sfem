#include "hip/hip_runtime.h"
// #include "laplacian.h"

#include <cassert>
#include <cmath>
// #include <cstdio>
#include <algorithm>
#include <cstddef>

extern "C" {
#include "sfem_base.h"

#include "crs_graph.h"
#include "cuda_crs.h"
#include "sfem_base.h"
#include "sfem_vec.h"
#include "sortreduce.h"
}

#include "sfem_cuda_base.h"

#if 1
#include "nvToolsExt.h"
#define SFEM_RANGE_PUSH(name_) \
    do {                       \
        nvtxRangePushA(name_); \
    } while (0)
#define SFEM_RANGE_POP() \
    do {                 \
        nvtxRangePop();  \
    } while (0)
#else
#define SFEM_RANGE_PUSH(name_)
#define SFEM_RANGE_POP()
#endif

#define MIN(a, b) ((a) < (b) ? (a) : (b))
#define POW2(a) ((a) * (a))

static inline __device__ void laplacian(const real_t *SFEM_RESTRICT jac_inv,
                                        const count_t stride,
                                        real_t *SFEM_RESTRICT element_matrix)

{
    real_t dv;
    {
        dv =
            0.16666666666666666 / (jac_inv[0 * stride] * jac_inv[4 * stride] * jac_inv[8 * stride] -
                                   jac_inv[0 * stride] * jac_inv[5 * stride] * jac_inv[7 * stride] -
                                   jac_inv[1 * stride] * jac_inv[3 * stride] * jac_inv[8 * stride] +
                                   jac_inv[1 * stride] * jac_inv[5 * stride] * jac_inv[6 * stride] +
                                   jac_inv[2 * stride] * jac_inv[3 * stride] * jac_inv[7 * stride] -
                                   jac_inv[2 * stride] * jac_inv[4 * stride] * jac_inv[6 * stride]);

        assert(dv == dv);
    }

    {
        // FLOATING POINT OPS!
        //      - Result: 4*ADD + 16*ASSIGNMENT + 4*MUL + 12*POW
        //      - Subexpressions: 12*ADD + 24*MUL + 3*NEG + 6*SUB
        const real_t x0 = -jac_inv[0 * stride] - jac_inv[3 * stride] - jac_inv[6 * stride];
        const real_t x1 = -jac_inv[1 * stride] - jac_inv[4 * stride] - jac_inv[7 * stride];
        const real_t x2 = -jac_inv[2 * stride] - jac_inv[5 * stride] - jac_inv[8 * stride];
        const real_t x3 =
            dv * (jac_inv[0 * stride] * x0 + jac_inv[1 * stride] * x1 + jac_inv[2 * stride] * x2);
        const real_t x4 =
            dv * (jac_inv[3 * stride] * x0 + jac_inv[4 * stride] * x1 + jac_inv[5 * stride] * x2);
        const real_t x5 =
            dv * (jac_inv[6 * stride] * x0 + jac_inv[7 * stride] * x1 + jac_inv[8 * stride] * x2);
        const real_t x6 = dv * (jac_inv[0 * stride] * jac_inv[3 * stride] +
                                jac_inv[1 * stride] * jac_inv[4 * stride] +
                                jac_inv[2 * stride] * jac_inv[5 * stride]);
        const real_t x7 = dv * (jac_inv[0 * stride] * jac_inv[6 * stride] +
                                jac_inv[1 * stride] * jac_inv[7 * stride] +
                                jac_inv[2 * stride] * jac_inv[8 * stride]);
        const real_t x8 = dv * (jac_inv[3 * stride] * jac_inv[6 * stride] +
                                jac_inv[4 * stride] * jac_inv[7 * stride] +
                                jac_inv[5 * stride] * jac_inv[8 * stride]);
        element_matrix[0 * stride] = dv * (POW2(x0) + POW2(x1) + POW2(x2));
        element_matrix[1 * stride] = x3;
        element_matrix[2 * stride] = x4;
        element_matrix[3 * stride] = x5;
        element_matrix[4 * stride] = x3;
        element_matrix[5 * stride] = dv * (POW2(jac_inv[0 * stride]) + POW2(jac_inv[1 * stride]) +
                                           POW2(jac_inv[2 * stride]));
        element_matrix[6 * stride] = x6;
        element_matrix[7 * stride] = x7;
        element_matrix[8 * stride] = x4;
        element_matrix[9 * stride] = x6;
        element_matrix[10 * stride] = dv * (POW2(jac_inv[3 * stride]) + POW2(jac_inv[4 * stride]) +
                                            POW2(jac_inv[5 * stride]));
        element_matrix[11 * stride] = x8;
        element_matrix[12 * stride] = x5;
        element_matrix[13 * stride] = x7;
        element_matrix[14 * stride] = x8;
        element_matrix[15 * stride] = dv * (POW2(jac_inv[6 * stride]) + POW2(jac_inv[7 * stride]) +
                                            POW2(jac_inv[8 * stride]));
    }

    // printf("[%g %g %g\n%g %g %g\n%g %g %g]\n",
    //        jac_inv[0 * stride],
    //        jac_inv[1 * stride],
    //        jac_inv[2 * stride],
    //        jac_inv[3 * stride],
    //        jac_inv[4 * stride],
    //        jac_inv[5 * stride],
    //        jac_inv[6 * stride],
    //        jac_inv[7 * stride],
    //        jac_inv[8 * stride]);

    // printf("[%g %g %g %g\n%g %g %g %g\n%g %g %g %g\n%g %g %g %g]\n",
    //        element_matrix[0 * stride],
    //        element_matrix[1 * stride],
    //        element_matrix[2 * stride],
    //        element_matrix[3 * stride],
    //        element_matrix[4 * stride],
    //        element_matrix[5 * stride],
    //        element_matrix[6 * stride],
    //        element_matrix[7 * stride],
    //        element_matrix[8 * stride],
    //        element_matrix[9 * stride],
    //        element_matrix[10 * stride],
    //        element_matrix[11 * stride],
    //        element_matrix[12 * stride],
    //        element_matrix[13 * stride],
    //        element_matrix[14 * stride],
    //        element_matrix[15 * stride]);
}

static inline __device__ __host__ int linear_search(const idx_t target,
                                                    const idx_t *const arr,
                                                    const int size) {
    int i;
    for (i = 0; i < size - 4; i += 4) {
        if (arr[i] == target) return i;
        if (arr[i + 1] == target) return i + 1;
        if (arr[i + 2] == target) return i + 2;
        if (arr[i + 3] == target) return i + 3;
    }
    for (; i < size; i++) {
        if (arr[i] == target) return i;
    }
    return -1;
}

static inline __device__ __host__ int find_col(const idx_t key,
                                               const idx_t *const row,
                                               const int lenrow) {
    // if (lenrow <= 32)
    // {
    return linear_search(key, row, lenrow);

    // Using sentinel (potentially dangerous if matrix is buggy and column does not exist)
    // while (key > row[++k]) {
    //     // Hi
    // }
    // assert(k < lenrow);
    // assert(key == row[k]);
    // } else {
    //     // Use this for larger number of dofs per row
    //     return find_idx_binary_search(key, row, lenrow);
    // }
}

static inline __device__ __host__ void find_cols4(const idx_t *targets,
                                                  const idx_t *const row,
                                                  const int lenrow,
                                                  int *ks) {
    if (lenrow > 32) {
        for (int d = 0; d < 4; ++d) {
            ks[d] = find_col(targets[d], row, lenrow);
        }
    } else {
#pragma unroll(4)
        for (int d = 0; d < 4; ++d) {
            ks[d] = 0;
        }

        for (int i = 0; i < lenrow; ++i) {
#pragma unroll(4)
            for (int d = 0; d < 4; ++d) {
                ks[d] += row[i] < targets[d];
            }
        }
    }
}

static inline __device__ __host__ void jacobian_inverse_micro_kernel(const real_t px0,
                                                                     const real_t px1,
                                                                     const real_t px2,
                                                                     const real_t px3,
                                                                     const real_t py0,
                                                                     const real_t py1,
                                                                     const real_t py2,
                                                                     const real_t py3,
                                                                     const real_t pz0,
                                                                     const real_t pz1,
                                                                     const real_t pz2,
                                                                     const real_t pz3,
                                                                     const count_t stride,
                                                                     real_t *jac_inv) {
    // printf("[%g %g %g] [%g %g %g] [%g %g %g] [%g %g %g] %ld\n",
    //        px0,
    //        py0,
    //        pz0,
    //        px1,
    //        py1,
    //        pz1,
    //        px2,
    //        py2,
    //        pz2,
    //        px3,
    //        py3,
    //        pz3,
    //        (long)stride);

    // FLOATING POINT OPS!
    //       - Result: 9*ADD + 9*ASSIGNMENT + 25*MUL
    //       - Subexpressions: 2*ADD + DIV + 12*MUL + 12*SUB
    const real_t x0 = -py0 + py2;
    const real_t x1 = -pz0 + pz3;
    const real_t x2 = x0 * x1;
    const real_t x3 = -py0 + py3;
    const real_t x4 = -pz0 + pz2;
    const real_t x5 = x3 * x4;
    const real_t x6 = -px0 + px1;
    const real_t x7 = -pz0 + pz1;
    const real_t x8 = -px0 + px2;
    const real_t x9 = x3 * x8;
    const real_t x10 = -py0 + py1;
    const real_t x11 = -px0 + px3;
    const real_t x12 = x1 * x8;
    const real_t x13 = x0 * x11;
    const real_t x14 = 1.0 / (x10 * x11 * x4 - x10 * x12 - x13 * x7 + x2 * x6 - x5 * x6 + x7 * x9);
    jac_inv[0 * stride] = x14 * (x2 - x5);
    jac_inv[1 * stride] = x14 * (x11 * x4 - x12);
    jac_inv[2 * stride] = x14 * (-x13 + x9);
    jac_inv[3 * stride] = x14 * (-x1 * x10 + x3 * x7);
    jac_inv[4 * stride] = x14 * (x1 * x6 - x11 * x7);
    jac_inv[5 * stride] = x14 * (x10 * x11 - x3 * x6);
    jac_inv[6 * stride] = x14 * (-x0 * x7 + x10 * x4);
    jac_inv[7 * stride] = x14 * (-x4 * x6 + x7 * x8);
    jac_inv[8 * stride] = x14 * (x0 * x6 - x10 * x8);

    // printf("[%g %g %g\n%g %g %g\n%g %g %g]\n",
    //        jac_inv[0 * stride],
    //        jac_inv[1 * stride],
    //        jac_inv[2 * stride],
    //        jac_inv[3 * stride],
    //        jac_inv[4 * stride],
    //        jac_inv[5 * stride],
    //        jac_inv[6 * stride],
    //        jac_inv[7 * stride],
    //        jac_inv[8 * stride]);
}

__global__ void jacobian_inverse_kernel(const ptrdiff_t nelements,
                                        const geom_t *const SFEM_RESTRICT xyz,
                                        real_t *const SFEM_RESTRICT jacobian_inverse) {
    for (ptrdiff_t e = blockIdx.x * blockDim.x + threadIdx.x; e < nelements;
         e += blockDim.x * gridDim.x) {
        // Thy element coordinates and jacobian
        const geom_t *const this_xyz = &xyz[e];
        real_t *const this_jacobian_inverse = &jacobian_inverse[e];

        const ptrdiff_t xi = 0 * 4;
        const ptrdiff_t yi = 1 * 4;
        const ptrdiff_t zi = 2 * 4;

        jacobian_inverse_micro_kernel(
            // X-coordinates
            this_xyz[(xi + 0) * nelements],
            this_xyz[(xi + 1) * nelements],
            this_xyz[(xi + 2) * nelements],
            this_xyz[(xi + 3) * nelements],
            // Y-coordinates
            this_xyz[(yi + 0) * nelements],
            this_xyz[(yi + 1) * nelements],
            this_xyz[(yi + 2) * nelements],
            this_xyz[(yi + 3) * nelements],
            // Z-coordinates
            this_xyz[(zi + 0) * nelements],
            this_xyz[(zi + 1) * nelements],
            this_xyz[(zi + 2) * nelements],
            this_xyz[(zi + 3) * nelements],
            nelements,
            this_jacobian_inverse);
    }
}

__global__ void laplacian_crs_kernel(const ptrdiff_t nelements,
                                                  const real_t *const SFEM_RESTRICT
                                                      jacobian_inverse,
                                                  real_t *const SFEM_RESTRICT values) {
    for (ptrdiff_t e = blockIdx.x * blockDim.x + threadIdx.x; e < nelements;
         e += blockDim.x * gridDim.x) {
        laplacian(&jacobian_inverse[e], nelements, &values[e]);
    }
}

__global__ void local_to_global_kernel(const ptrdiff_t nelements,
                                       idx_t **const SFEM_RESTRICT elems,
                                       const real_t *const SFEM_RESTRICT element_matrix,
                                       const count_t *const SFEM_RESTRICT rowptr,
                                       const idx_t *const SFEM_RESTRICT colidx,
                                       real_t *const SFEM_RESTRICT values) {
    idx_t ev[4];
    idx_t ks[4];
    for (ptrdiff_t e = blockIdx.x * blockDim.x + threadIdx.x; e < nelements;
         e += blockDim.x * gridDim.x) {
#pragma unroll(4)
        for (int v = 0; v < 4; ++v) {
            ev[v] = elems[v][e];
        }

        // offsetted array for this element
        const real_t *const this_matrix = &element_matrix[e];

        // printf("%d)\n", (int)e);

        for (int edof_i = 0; edof_i < 4; ++edof_i) {
            const idx_t dof_i = ev[edof_i];
            const idx_t lenrow = rowptr[dof_i + 1] - rowptr[dof_i];

            const idx_t *const row = &colidx[rowptr[dof_i]];

            find_cols4(ev, row, lenrow, ks);

            real_t *const rowvalues = &values[rowptr[dof_i]];

            // #pragma unroll(4)
            for (int edof_j = 0; edof_j < 4; ++edof_j) {
                ptrdiff_t idx = (edof_i * 4 + edof_j) * nelements;
                const real_t v = this_matrix[idx];

                // printf("(%d, %d) %g\n", dof_i, ev[edof_j], v);

                atomicAdd(&rowvalues[ks[edof_j]], v);
            }

            // printf("\n");
        }

        // printf("\n");

        // printf("[%g %g %g %g\n%g %g %g %g\n%g %g %g %g\n%g %g %g %g]\n",
        //        this_matrix[0 * nelements],
        //        this_matrix[1 * nelements],
        //        this_matrix[2 * nelements],
        //        this_matrix[3 * nelements],
        //        this_matrix[4 * nelements],
        //        this_matrix[5 * nelements],
        //        this_matrix[6 * nelements],
        //        this_matrix[7 * nelements],
        //        this_matrix[8 * nelements],
        //        this_matrix[9 * nelements],
        //        this_matrix[10 * nelements],
        //        this_matrix[11 * nelements],
        //        this_matrix[12 * nelements],
        //        this_matrix[13 * nelements],
        //        this_matrix[14 * nelements],
        //        this_matrix[15 * nelements]);
    }
}

__global__ void print_elem_kernel(const ptrdiff_t nelements, idx_t **const elems) {
    const int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= nelements) return;

    printf("%d %d %d %d\n", elems[0][i], elems[1][i], elems[2][i], elems[3][i]);
}

#if 0

extern "C" void laplacian_crs(const ptrdiff_t nelements,
                                           const ptrdiff_t nnodes,
                                           idx_t **const SFEM_RESTRICT elems,
                                           geom_t **const SFEM_RESTRICT xyz,
                                           const count_t *const SFEM_RESTRICT rowptr,
                                           const idx_t *const SFEM_RESTRICT colidx,
                                           real_t *const SFEM_RESTRICT values) {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    static int block_size = 128;
    const ptrdiff_t nbatch = MIN(block_size * 1000, nelements);

    ptrdiff_t n_blocks = std::max(ptrdiff_t(1), (nbatch + block_size - 1) / block_size);

    geom_t *he_xyz = nullptr;
    SFEM_CUDA_CHECK(hipHostMalloc(&he_xyz, 3 * 4 * nbatch * sizeof(geom_t)));
    geom_t *de_xyz = nullptr;
    SFEM_CUDA_CHECK(hipMalloc(&de_xyz, 3 * 4 * nbatch * sizeof(geom_t)));

    real_t *d_jacobian_inverse = nullptr;
    SFEM_CUDA_CHECK(hipMalloc(&d_jacobian_inverse, 3 * 3 * nbatch * sizeof(real_t)));

    real_t *he_matrix = nullptr;
    hipHostMalloc(&he_matrix, 4 * 4 * nbatch * sizeof(real_t));
    real_t *de_matrix = nullptr;
    SFEM_CUDA_CHECK(hipMalloc(&de_matrix, 4 * 4 * nbatch * sizeof(real_t)));

    idx_t **hd_elems[4];
    idx_t **d_elems = nullptr;

    count_t *d_rowptr = nullptr;
    idx_t *d_colidx = nullptr;
    real_t *d_values = nullptr;

    // Allocate space for indices
    for (int d = 0; d < 4; d++) {
        SFEM_CUDA_CHECK(hipMalloc(&hd_elems[d], nbatch * sizeof(idx_t)));
    }

    SFEM_CUDA_CHECK(hipMalloc(&d_elems, 4 * sizeof(idx_t *)));
    hipMemcpy(d_elems, hd_elems, 4 * sizeof(idx_t *), hipMemcpyHostToDevice);

    // Copy crs-matrix
    crs_device_create(nnodes, rowptr[nnodes], &d_rowptr, &d_colidx, &d_values);
    crs_graph_host_to_device(nnodes, rowptr[nnodes], rowptr, colidx, d_rowptr, d_colidx);

    ptrdiff_t last_n = 0;
    for (ptrdiff_t element_offset = 0; element_offset < nelements; element_offset += nbatch) {
        ptrdiff_t n = MIN(nbatch, nelements - element_offset);

        {
            // #pragma omp parallel
            {
                // #pragma omp parallel for collapse(2)
                for (int d = 0; d < 3; ++d) {
                    for (int e_node = 0; e_node < 4; e_node++) {
                        // printf("%d %d\n", d, e_node)
                        const geom_t *const x = xyz[d];
                        ptrdiff_t offset = (d * 4 + e_node) * n;
                        const idx_t *const nodes = &elems[e_node][element_offset];

                        geom_t *buff = &he_xyz[offset];
                        // #pragma omp parallel for
                        for (ptrdiff_t k = 0; k < n; k++) {
                            buff[k] = x[nodes[k]];
                        }
                    }
                }
            }
        }

        if (last_n) {
            // Do this here to let the main kernel overlap with the packing
            local_to_global_kernel<<<n_blocks, block_size>>>(last_n, d_elems, de_matrix, d_rowptr, d_colidx, d_values);
        }

        SFEM_CUDA_CHECK(hipMemcpy(de_xyz, he_xyz, 3 * 4 * n * sizeof(geom_t), hipMemcpyHostToDevice));

        for (int e_node = 0; e_node < 4; e_node++) {
            SFEM_CUDA_CHECK(hipMemcpy(
                hd_elems[e_node], &elems[e_node][element_offset], n * sizeof(idx_t), hipMemcpyHostToDevice));
        }

        jacobian_inverse_kernel<<<n_blocks, block_size>>>(n, de_xyz, d_jacobian_inverse);
        laplacian_crs_kernel<<<n_blocks, block_size>>>(n, d_jacobian_inverse, de_matrix);
        last_n = n;
    }

    if (last_n) {
        local_to_global_kernel<<<n_blocks, block_size>>>(last_n, d_elems, de_matrix, d_rowptr, d_colidx, d_values);
    }

    SFEM_CUDA_CHECK(hipMemcpy(values, d_values, rowptr[nnodes] * sizeof(real_t), hipMemcpyDeviceToHost));

    {  // Free resources on CPU
        hipHostFree(he_xyz);
        hipHostFree(he_matrix);
    }

    {  // Free resources on GPU
        SFEM_CUDA_CHECK(hipFree(de_xyz));
        SFEM_CUDA_CHECK(hipFree(de_matrix));
        SFEM_CUDA_CHECK(hipFree(d_jacobian_inverse));

        for (int d = 0; d < 4; d++) {
            SFEM_CUDA_CHECK(hipFree(hd_elems[d]));
        }
        SFEM_CUDA_CHECK(hipFree(d_elems));

        crs_device_free(d_rowptr, d_colidx, d_values);
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    printf("cuda_laplacian_2.c: laplacian_crs\t%g seconds\nloops %d\n",
           milliseconds / 1000,
           int(nelements / nbatch));
}

#else

extern "C" void laplacian_crs(const ptrdiff_t nelements,
                                           const ptrdiff_t nnodes,
                                           idx_t **const SFEM_RESTRICT elems,
                                           geom_t **const SFEM_RESTRICT xyz,
                                           const count_t *const SFEM_RESTRICT rowptr,
                                           const idx_t *const SFEM_RESTRICT colidx,
                                           real_t *const SFEM_RESTRICT values) {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    SFEM_RANGE_PUSH("lapl-set-up");
    hipEventRecord(start);

    // static int block_size = 256;
    static int block_size = 128;
    const ptrdiff_t nbatch = MIN(block_size * 500, nelements);

    ptrdiff_t n_blocks = std::max(ptrdiff_t(1), (nbatch + block_size - 1) / block_size);

    geom_t *he_xyz = nullptr;
    SFEM_CUDA_CHECK(hipHostMalloc(&he_xyz, 3 * 4 * nbatch * sizeof(geom_t)));
    geom_t *de_xyz = nullptr;
    SFEM_CUDA_CHECK(hipMalloc(&de_xyz, 3 * 4 * nbatch * sizeof(geom_t)));

    real_t *d_jacobian_inverse = nullptr;
    SFEM_CUDA_CHECK(hipMalloc(&d_jacobian_inverse, 3 * 3 * nbatch * sizeof(real_t)));

    real_t *he_matrix = nullptr;
    hipHostMalloc(&he_matrix, 4 * 4 * nbatch * sizeof(real_t));
    real_t *de_matrix = nullptr;
    SFEM_CUDA_CHECK(hipMalloc(&de_matrix, 4 * 4 * nbatch * sizeof(real_t)));

    idx_t *hh_elems[4];
    for (int d = 0; d < 4; d++) {
        SFEM_CUDA_CHECK(hipHostMalloc(&hh_elems[d], nbatch * sizeof(idx_t)));
    }

    idx_t **hd_elems[4];
    idx_t **d_elems = nullptr;

    count_t *d_rowptr = nullptr;
    idx_t *d_colidx = nullptr;
    real_t *d_values = nullptr;

    static const int nstreams = 2;
    hipStream_t stream[nstreams];
    // hipEvent_t cu_event[nstreams];
    for (int s = 0; s < nstreams; s++) {
        hipStreamCreate(&stream[s]);
        // hipEventCreate(&cu_event[s]);
    }

    // Allocate space for indices
    for (int d = 0; d < 4; d++) {
        SFEM_CUDA_CHECK(hipMalloc(&hd_elems[d], nbatch * sizeof(idx_t)));
    }

    SFEM_CUDA_CHECK(hipMalloc(&d_elems, 4 * sizeof(idx_t *)));
    hipMemcpy(d_elems, hd_elems, 4 * sizeof(idx_t *), hipMemcpyHostToDevice);

    SFEM_RANGE_POP();

    SFEM_RANGE_PUSH("lapl-crs-host-to-device");
    // Copy crs-matrix
    crs_device_create(nnodes, rowptr[nnodes], &d_rowptr, &d_colidx, &d_values);
    crs_graph_host_to_device(nnodes, rowptr[nnodes], rowptr, colidx, d_rowptr, d_colidx);

    SFEM_RANGE_POP();

    ptrdiff_t last_n = 0;
    ptrdiff_t last_element_offset = 0;
    for (ptrdiff_t element_offset = 0; element_offset < nelements; element_offset += nbatch) {
        ptrdiff_t n = MIN(nbatch, nelements - element_offset);

        {
            SFEM_RANGE_PUSH("lapl-packing");
            {
                for (int d = 0; d < 3; ++d) {
                    for (int e_node = 0; e_node < 4; e_node++) {
                        const geom_t *const x = xyz[d];
                        ptrdiff_t offset = (d * 4 + e_node) * n;
                        const idx_t *const nodes = &elems[e_node][element_offset];

                        geom_t *buff = &he_xyz[offset];

#pragma omp parallel
                        {
#pragma omp for  // nowait
                            for (ptrdiff_t k = 0; k < n; k++) {
                                buff[k] = x[nodes[k]];
                            }
                        }
                    }
                }
            }

            SFEM_RANGE_POP();
        }

        if (last_n) {
            hipStreamSynchronize(stream[0]);
            // Do this here to let the main kernel overlap with the packing
            local_to_global_kernel<<<n_blocks, block_size, 0, stream[1]>>>(
                last_n, d_elems, de_matrix, d_rowptr, d_colidx, d_values);

            SFEM_DEBUG_SYNCHRONIZE();
        }

        SFEM_CUDA_CHECK(hipMemcpyAsync(
            de_xyz, he_xyz, 3 * 4 * n * sizeof(geom_t), hipMemcpyHostToDevice, stream[0]));

        if (last_n) {
            // make sure that the previous copy async and kernel from stream 1 is finished!
            hipStreamSynchronize(stream[1]);
        }

        SFEM_RANGE_PUSH("lapl-copy-host-to-host");
        //  Copy elements to host-pinned memory
        for (int e_node = 0; e_node < 4; e_node++) {
            memcpy(hh_elems[e_node], &elems[e_node][element_offset], n * sizeof(idx_t));
        }
        SFEM_RANGE_POP();

        for (int e_node = 0; e_node < 4; e_node++) {
            SFEM_CUDA_CHECK(hipMemcpyAsync(hd_elems[e_node],
                                            hh_elems[e_node],
                                            n * sizeof(idx_t),
                                            hipMemcpyHostToDevice,
                                            stream[1]));
        }

        jacobian_inverse_kernel<<<n_blocks, block_size, 0, stream[0]>>>(
            n, de_xyz, d_jacobian_inverse);

        SFEM_DEBUG_SYNCHRONIZE();

        laplacian_crs_kernel<<<n_blocks, block_size, 0, stream[0]>>>(
            n, d_jacobian_inverse, de_matrix);

        SFEM_DEBUG_SYNCHRONIZE();

        last_n = n;
        last_element_offset = element_offset;
    }

    if (last_n) {
        hipStreamSynchronize(stream[0]);
        // Do this here to let the main kernel overlap with the packing
        local_to_global_kernel<<<n_blocks, block_size, 0, stream[1]>>>(
            last_n, d_elems, de_matrix, d_rowptr, d_colidx, d_values);

        SFEM_DEBUG_SYNCHRONIZE();

        hipStreamSynchronize(stream[1]);
    }

    SFEM_RANGE_PUSH("lapl-values-device-to-host");

    SFEM_CUDA_CHECK(
        hipMemcpy(values, d_values, rowptr[nnodes] * sizeof(real_t), hipMemcpyDeviceToHost));

    SFEM_RANGE_POP();

    SFEM_RANGE_PUSH("lapl-tear-down");
    {  // Free resources on CPU
        hipHostFree(he_xyz);
        hipHostFree(he_matrix);

        for (int d = 0; d < 4; d++) {
            SFEM_CUDA_CHECK(hipHostFree(hh_elems[d]));
        }
    }

    {  // Free resources on GPU
        SFEM_CUDA_CHECK(hipFree(de_xyz));
        SFEM_CUDA_CHECK(hipFree(de_matrix));
        SFEM_CUDA_CHECK(hipFree(d_jacobian_inverse));

        for (int d = 0; d < 4; d++) {
            SFEM_CUDA_CHECK(hipFree(hd_elems[d]));
        }
        SFEM_CUDA_CHECK(hipFree(d_elems));

        crs_device_free(d_rowptr, d_colidx, d_values);

        for (int s = 0; s < nstreams; s++) {
            hipStreamDestroy(stream[s]);
            // hipEventDestroy(cu_event[s]);
        }
    }

    SFEM_RANGE_POP();

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    printf("cuda_laplacian_2.c: laplacian_crs\t%g seconds\nloops %d\n",
           milliseconds / 1000,
           int(nelements / nbatch));
}

#endif

extern "C" void tet4_laplacian_assemble_value(const ptrdiff_t nelements,
                                              const ptrdiff_t nnodes,
                                              idx_t **const SFEM_RESTRICT elems,
                                              geom_t **const SFEM_RESTRICT xyz,
                                              const real_t *const SFEM_RESTRICT u,
                                              real_t *const SFEM_RESTRICT value) {
    assert(false);
}
