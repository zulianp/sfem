#include "hip/hip_runtime.h"
#include <algorithm>
#include <cassert>
#include <cmath>
#include <cstddef>

#include "sfem_base.h"
#include "sfem_vec.h"
#include "sortreduce.h"

#include "cu_tet10_linear_elasticity.h"
#include "sfem_cuda_base.h"
#include "sfem_defs.h"

#include "cu_tet4_inline.hpp"

#define MIN(a, b) ((a) < (b) ? (a) : (b))
#define POW2(a) ((a) * (a))


static inline __device__ __host__ void ref_shape_grad_x(const scalar_t qx,
                                                        const scalar_t qy,
                                                        const scalar_t qz,
                                                        scalar_t *const out) {
    const scalar_t x0 = 4 * qx;
    const scalar_t x1 = 4 * qy;
    const scalar_t x2 = 4 * qz;
    const scalar_t x3 = x1 + x2;
    out[0] = x0 + x3 - 3;
    out[1] = x0 - 1;
    out[2] = 0;
    out[3] = 0;
    out[4] = -8 * qx - x3 + 4;
    out[5] = x1;
    out[6] = -x1;
    out[7] = -x2;
    out[8] = x2;
    out[9] = 0;
}

static inline __device__ __host__ void ref_shape_grad_y(const scalar_t qx,
                                                        const scalar_t qy,
                                                        const scalar_t qz,
                                                        scalar_t *const out) {
    const scalar_t x0 = 4 * qy;
    const scalar_t x1 = 4 * qx;
    const scalar_t x2 = 4 * qz;
    const scalar_t x3 = x1 + x2;
    out[0] = x0 + x3 - 3;
    out[1] = 0;
    out[2] = x0 - 1;
    out[3] = 0;
    out[4] = -x1;
    out[5] = x1;
    out[6] = -8 * qy - x3 + 4;
    out[7] = -x2;
    out[8] = 0;
    out[9] = x2;
}

static inline __device__ __host__ void ref_shape_grad_z(const scalar_t qx,
                                                        const scalar_t qy,
                                                        const scalar_t qz,
                                                        scalar_t *const out) {
    const scalar_t x0 = 4 * qz;
    const scalar_t x1 = 4 * qx;
    const scalar_t x2 = 4 * qy;
    const scalar_t x3 = x1 + x2;
    out[0] = x0 + x3 - 3;
    out[1] = 0;
    out[2] = 0;
    out[3] = x0 - 1;
    out[4] = -x1;
    out[5] = 0;
    out[6] = -x2;
    out[7] = -8 * qz - x3 + 4;
    out[8] = x1;
    out[9] = x2;
}

static inline __device__ __host__ void apply_micro_kernel(
        const scalar_t mu,
        const scalar_t lambda,
        const scalar_t *const SFEM_RESTRICT adjugate,
        const scalar_t jacobian_determinant,
        const scalar_t qx,
        const scalar_t qy,
        const scalar_t qz,
        const scalar_t qw,
        const scalar_t *const SFEM_RESTRICT u,
        accumulator_t *const SFEM_RESTRICT element_vector) {
    // This can be reduced with 1D products (ref_shape_grad_{x,y,z})
    scalar_t disp_grad[9] = {0};

#define MICRO_KERNEL_USE_CODEGEN 0

#if MICRO_KERNEL_USE_CODEGEN
    // Code-gen way

    const scalar_t denom = 1;
    {
        const scalar_t x0 = 1.0 / jacobian_determinant;
        const scalar_t x1 = 4 * qx;
        const scalar_t x2 = x1 - 1;
        const scalar_t x3 = 4 * qy;
        const scalar_t x4 = -u[6] * x3;
        const scalar_t x5 = qz - 1;
        const scalar_t x6 = 8 * qx + 4 * qy + 4 * x5;
        const scalar_t x7 = 4 * qz;
        const scalar_t x8 = x1 + x3 + x7 - 3;
        const scalar_t x9 = u[0] * x8;
        const scalar_t x10 = -u[7] * x7 + x9;
        const scalar_t x11 = u[1] * x2 - u[4] * x6 + u[5] * x3 + u[8] * x7 + x10 + x4;
        const scalar_t x12 = x3 - 1;
        const scalar_t x13 = -u[4] * x1;
        const scalar_t x14 = 4 * qx + 8 * qy + 4 * x5;
        const scalar_t x15 = u[2] * x12 + u[5] * x1 - u[6] * x14 + u[9] * x7 + x10 + x13;
        const scalar_t x16 = x7 - 1;
        const scalar_t x17 = 4 * qx + 4 * qy + 8 * qz - 4;
        const scalar_t x18 = u[3] * x16 - u[7] * x17 + u[8] * x1 + u[9] * x3 + x13 + x4 + x9;
        const scalar_t x19 = -u[16] * x3;
        const scalar_t x20 = u[10] * x8;
        const scalar_t x21 = -u[17] * x7 + x20;
        const scalar_t x22 = u[11] * x2 - u[14] * x6 + u[15] * x3 + u[18] * x7 + x19 + x21;
        const scalar_t x23 = -u[14] * x1;
        const scalar_t x24 = u[12] * x12 + u[15] * x1 - u[16] * x14 + u[19] * x7 + x21 + x23;
        const scalar_t x25 = u[13] * x16 - u[17] * x17 + u[18] * x1 + u[19] * x3 + x19 + x20 + x23;
        const scalar_t x26 = -u[26] * x3;
        const scalar_t x27 = u[20] * x8;
        const scalar_t x28 = -u[27] * x7 + x27;
        const scalar_t x29 = u[21] * x2 - u[24] * x6 + u[25] * x3 + u[28] * x7 + x26 + x28;
        const scalar_t x30 = -u[24] * x1;
        const scalar_t x31 = u[22] * x12 + u[25] * x1 - u[26] * x14 + u[29] * x7 + x28 + x30;
        const scalar_t x32 = u[23] * x16 - u[27] * x17 + u[28] * x1 + u[29] * x3 + x26 + x27 + x30;
        disp_grad[0] = x0 * (adjugate[0] * x11 + adjugate[3] * x15 + adjugate[6] * x18);
        disp_grad[1] = x0 * (adjugate[1] * x11 + adjugate[4] * x15 + adjugate[7] * x18);
        disp_grad[2] = x0 * (adjugate[2] * x11 + adjugate[5] * x15 + adjugate[8] * x18);
        disp_grad[3] = x0 * (adjugate[0] * x22 + adjugate[3] * x24 + adjugate[6] * x25);
        disp_grad[4] = x0 * (adjugate[1] * x22 + adjugate[4] * x24 + adjugate[7] * x25);
        disp_grad[5] = x0 * (adjugate[2] * x22 + adjugate[5] * x24 + adjugate[8] * x25);
        disp_grad[6] = x0 * (adjugate[0] * x29 + adjugate[3] * x31 + adjugate[6] * x32);
        disp_grad[7] = x0 * (adjugate[1] * x29 + adjugate[4] * x31 + adjugate[7] * x32);
        disp_grad[8] = x0 * (adjugate[2] * x29 + adjugate[5] * x31 + adjugate[8] * x32);
    }
#else
    // Programmatic way

    const scalar_t denom = jacobian_determinant;
    {
        scalar_t temp[9] = {0};
        scalar_t grad[10];

        ref_shape_grad_x(qx, qy, qz, grad);
#pragma unroll
        for (int i = 0; i < 10; i++) {
            const scalar_t g = grad[i];
            temp[0] += u[i] * g;
            temp[3] += u[10 + i] * g;
            temp[6] += u[20 + i] * g;
        }

        ref_shape_grad_y(qx, qy, qz, grad);
#pragma unroll
        for (int i = 0; i < 10; i++) {
            const scalar_t g = grad[i];
            temp[1] += u[i] * g;
            temp[4] += u[10 + i] * g;
            temp[7] += u[20 + i] * g;
        }

        ref_shape_grad_z(qx, qy, qz, grad);
#pragma unroll
        for (int i = 0; i < 10; i++) {
            const scalar_t g = grad[i];
            temp[2] += u[i] * g;
            temp[5] += u[10 + i] * g;
            temp[8] += u[20 + i] * g;
        }

        for (int i = 0; i < 3; i++) {
#pragma unroll
            for (int j = 0; j < 3; j++) {
#pragma unroll
                for (int k = 0; k < 3; k++) {
                    disp_grad[i * 3 + j] += temp[i * 3 + k] * adjugate[k * 3 + j];
                }
            }
        }
    }

#endif
    // Includes first Piola-Kirchoff stress: P^T * J^-T * det(J)

    scalar_t *PxJinv_t = disp_grad;
    {
        const scalar_t x0 = (1.0 / 6.0) * mu;
        const scalar_t x1 = x0 * (disp_grad[1] + disp_grad[3]);
        const scalar_t x2 = x0 * (disp_grad[2] + disp_grad[6]);
        const scalar_t x3 = 2 * mu;
        const scalar_t x4 = lambda * (disp_grad[0] + disp_grad[4] + disp_grad[8]);
        const scalar_t x5 = (1.0 / 6.0) * disp_grad[0] * x3 + (1.0 / 6.0) * x4;
        const scalar_t x6 = x0 * (disp_grad[5] + disp_grad[7]);
        const scalar_t x7 = (1.0 / 6.0) * disp_grad[4] * x3 + (1.0 / 6.0) * x4;
        const scalar_t x8 = (1.0 / 6.0) * disp_grad[8] * x3 + (1.0 / 6.0) * x4;
        PxJinv_t[0] = adjugate[0] * x5 + adjugate[1] * x1 + adjugate[2] * x2;
        PxJinv_t[1] = adjugate[3] * x5 + adjugate[4] * x1 + adjugate[5] * x2;
        PxJinv_t[2] = adjugate[6] * x5 + adjugate[7] * x1 + adjugate[8] * x2;
        PxJinv_t[3] = adjugate[0] * x1 + adjugate[1] * x7 + adjugate[2] * x6;
        PxJinv_t[4] = adjugate[3] * x1 + adjugate[4] * x7 + adjugate[5] * x6;
        PxJinv_t[5] = adjugate[6] * x1 + adjugate[7] * x7 + adjugate[8] * x6;
        PxJinv_t[6] = adjugate[0] * x2 + adjugate[1] * x6 + adjugate[2] * x8;
        PxJinv_t[7] = adjugate[3] * x2 + adjugate[4] * x6 + adjugate[5] * x8;
        PxJinv_t[8] = adjugate[6] * x2 + adjugate[7] * x6 + adjugate[8] * x8;
    }

    // Scale by quadrature weight
    for (int i = 0; i < 9; i++) {
        PxJinv_t[i] *= qw / denom;
    }

// On CPU both versions are equivalent
#if MICRO_KERNEL_USE_CODEGEN
    {
        const scalar_t x0 = 4 * qx;
        const scalar_t x1 = 4 * qy;
        const scalar_t x2 = 4 * qz;
        const scalar_t x3 = x0 + x1 + x2 - 3;
        const scalar_t x4 = x0 - 1;
        const scalar_t x5 = x1 - 1;
        const scalar_t x6 = x2 - 1;
        const scalar_t x7 = PxJinv_t[1] * x0;
        const scalar_t x8 = PxJinv_t[2] * x0;
        const scalar_t x9 = qz - 1;
        const scalar_t x10 = 8 * qx + 4 * qy + 4 * x9;
        const scalar_t x11 = PxJinv_t[0] * x1;
        const scalar_t x12 = PxJinv_t[2] * x1;
        const scalar_t x13 = 4 * qx + 8 * qy + 4 * x9;
        const scalar_t x14 = PxJinv_t[0] * x2;
        const scalar_t x15 = PxJinv_t[1] * x2;
        const scalar_t x16 = 4 * qx + 4 * qy + 8 * qz - 4;
        const scalar_t x17 = PxJinv_t[4] * x0;
        const scalar_t x18 = PxJinv_t[5] * x0;
        const scalar_t x19 = PxJinv_t[3] * x1;
        const scalar_t x20 = PxJinv_t[5] * x1;
        const scalar_t x21 = PxJinv_t[3] * x2;
        const scalar_t x22 = PxJinv_t[4] * x2;
        const scalar_t x23 = PxJinv_t[7] * x0;
        const scalar_t x24 = PxJinv_t[8] * x0;
        const scalar_t x25 = PxJinv_t[6] * x1;
        const scalar_t x26 = PxJinv_t[8] * x1;
        const scalar_t x27 = PxJinv_t[6] * x2;
        const scalar_t x28 = PxJinv_t[7] * x2;
        element_vector[0] += x3 * (PxJinv_t[0] + PxJinv_t[1] + PxJinv_t[2]);
        element_vector[1] += PxJinv_t[0] * x4;
        element_vector[2] += PxJinv_t[1] * x5;
        element_vector[3] += PxJinv_t[2] * x6;
        element_vector[4] += -PxJinv_t[0] * x10 - x7 - x8;
        element_vector[5] += x11 + x7;
        element_vector[6] += -PxJinv_t[1] * x13 - x11 - x12;
        element_vector[7] += -PxJinv_t[2] * x16 - x14 - x15;
        element_vector[8] += x14 + x8;
        element_vector[9] += x12 + x15;
        element_vector[10] += x3 * (PxJinv_t[3] + PxJinv_t[4] + PxJinv_t[5]);
        element_vector[11] += PxJinv_t[3] * x4;
        element_vector[12] += PxJinv_t[4] * x5;
        element_vector[13] += PxJinv_t[5] * x6;
        element_vector[14] += -PxJinv_t[3] * x10 - x17 - x18;
        element_vector[15] += x17 + x19;
        element_vector[16] += -PxJinv_t[4] * x13 - x19 - x20;
        element_vector[17] += -PxJinv_t[5] * x16 - x21 - x22;
        element_vector[18] += x18 + x21;
        element_vector[19] += x20 + x22;
        element_vector[20] += x3 * (PxJinv_t[6] + PxJinv_t[7] + PxJinv_t[8]);
        element_vector[21] += PxJinv_t[6] * x4;
        element_vector[22] += PxJinv_t[7] * x5;
        element_vector[23] += PxJinv_t[8] * x6;
        element_vector[24] += -PxJinv_t[6] * x10 - x23 - x24;
        element_vector[25] += x23 + x25;
        element_vector[26] += -PxJinv_t[7] * x13 - x25 - x26;
        element_vector[27] += -PxJinv_t[8] * x16 - x27 - x28;
        element_vector[28] += x24 + x27;
        element_vector[29] += x26 + x28;
    }

#else

    {
        scalar_t grad[10];
        ref_shape_grad_x(qx, qy, qz, grad);

#pragma unroll
        for (int i = 0; i < 10; i++) {
            scalar_t g = grad[i];
            element_vector[i] += PxJinv_t[0] * g;
            element_vector[10 + i] += PxJinv_t[3] * g;
            element_vector[20 + i] += PxJinv_t[6] * g;
        }

        ref_shape_grad_y(qx, qy, qz, grad);

#pragma unroll
        for (int i = 0; i < 10; i++) {
            scalar_t g = grad[i];
            element_vector[i] += PxJinv_t[1] * g;
            element_vector[10 + i] += PxJinv_t[4] * g;
            element_vector[20 + i] += PxJinv_t[7] * g;
        }

        ref_shape_grad_z(qx, qy, qz, grad);

#pragma unroll
        for (int i = 0; i < 10; i++) {
            scalar_t g = grad[i];
            element_vector[i] += PxJinv_t[2] * g;
            element_vector[10 + i] += PxJinv_t[5] * g;
            element_vector[20 + i] += PxJinv_t[8] * g;
        }
    }

#endif

#undef MICRO_KERNEL_USE_CODEGEN
}

static inline __device__ __host__ void diag_micro_kernel(const scalar_t mu,
                                                         const scalar_t lambda,
                                                         const scalar_t *const SFEM_RESTRICT
                                                                 adjugate,
                                                         const scalar_t jacobian_determinant,
                                                         const scalar_t qx,
                                                         const scalar_t qy,
                                                         const scalar_t qz,
                                                         const scalar_t qw,
                                                         accumulator_t *const SFEM_RESTRICT diag) {
    const scalar_t x0 = POW2(adjugate[1] + adjugate[4] + adjugate[7]);
    const scalar_t x1 = mu * x0;
    const scalar_t x2 = POW2(adjugate[2] + adjugate[5] + adjugate[8]);
    const scalar_t x3 = mu * x2;
    const scalar_t x4 = lambda + 2 * mu;
    const scalar_t x5 = POW2(adjugate[0] + adjugate[3] + adjugate[6]);
    const scalar_t x6 = 4 * qx;
    const scalar_t x7 = 4 * qy;
    const scalar_t x8 = 4 * qz;
    const scalar_t x9 = 1.0 / jacobian_determinant;
    const scalar_t x10 = (1.0 / 6.0) * x9;
    const scalar_t x11 = x10 * POW2(x6 + x7 + x8 - 3);
    const scalar_t x12 = POW2(adjugate[1]);
    const scalar_t x13 = mu * x12;
    const scalar_t x14 = POW2(adjugate[2]);
    const scalar_t x15 = mu * x14;
    const scalar_t x16 = POW2(adjugate[0]);
    const scalar_t x17 = x10 * POW2(x6 - 1);
    const scalar_t x18 = POW2(adjugate[4]);
    const scalar_t x19 = mu * x18;
    const scalar_t x20 = POW2(adjugate[5]);
    const scalar_t x21 = mu * x20;
    const scalar_t x22 = POW2(adjugate[3]);
    const scalar_t x23 = x10 * POW2(x7 - 1);
    const scalar_t x24 = POW2(adjugate[7]);
    const scalar_t x25 = mu * x24;
    const scalar_t x26 = POW2(adjugate[8]);
    const scalar_t x27 = mu * x26;
    const scalar_t x28 = POW2(adjugate[6]);
    const scalar_t x29 = x10 * POW2(x8 - 1);
    const scalar_t x30 = adjugate[4] * qx;
    const scalar_t x31 = adjugate[7] * qx;
    const scalar_t x32 = qz - 1;
    const scalar_t x33 = 2 * qx + qy + x32;
    const scalar_t x34 = POW2(adjugate[1] * x33 + x30 + x31);
    const scalar_t x35 = mu * x34;
    const scalar_t x36 = adjugate[5] * qx;
    const scalar_t x37 = adjugate[8] * qx;
    const scalar_t x38 = POW2(adjugate[2] * x33 + x36 + x37);
    const scalar_t x39 = mu * x38;
    const scalar_t x40 = adjugate[3] * qx;
    const scalar_t x41 = adjugate[6] * qx;
    const scalar_t x42 = POW2(adjugate[0] * x33 + x40 + x41);
    const scalar_t x43 = (8.0 / 3.0) * x9;
    const scalar_t x44 = adjugate[1] * qy;
    const scalar_t x45 = POW2(x30 + x44);
    const scalar_t x46 = mu * x45;
    const scalar_t x47 = adjugate[2] * qy;
    const scalar_t x48 = POW2(x36 + x47);
    const scalar_t x49 = mu * x48;
    const scalar_t x50 = adjugate[0] * qy;
    const scalar_t x51 = POW2(x40 + x50);
    const scalar_t x52 = adjugate[7] * qy;
    const scalar_t x53 = qx + 2 * qy + x32;
    const scalar_t x54 = POW2(adjugate[4] * x53 + x44 + x52);
    const scalar_t x55 = mu * x54;
    const scalar_t x56 = adjugate[8] * qy;
    const scalar_t x57 = POW2(adjugate[5] * x53 + x47 + x56);
    const scalar_t x58 = mu * x57;
    const scalar_t x59 = adjugate[6] * qy;
    const scalar_t x60 = POW2(adjugate[3] * x53 + x50 + x59);
    const scalar_t x61 = adjugate[1] * qz;
    const scalar_t x62 = adjugate[4] * qz;
    const scalar_t x63 = qx + qy + 2 * qz - 1;
    const scalar_t x64 = POW2(adjugate[7] * x63 + x61 + x62);
    const scalar_t x65 = mu * x64;
    const scalar_t x66 = adjugate[2] * qz;
    const scalar_t x67 = adjugate[5] * qz;
    const scalar_t x68 = POW2(adjugate[8] * x63 + x66 + x67);
    const scalar_t x69 = mu * x68;
    const scalar_t x70 = adjugate[0] * qz;
    const scalar_t x71 = adjugate[3] * qz;
    const scalar_t x72 = POW2(adjugate[6] * x63 + x70 + x71);
    const scalar_t x73 = POW2(x31 + x61);
    const scalar_t x74 = mu * x73;
    const scalar_t x75 = POW2(x37 + x66);
    const scalar_t x76 = mu * x75;
    const scalar_t x77 = POW2(x41 + x70);
    const scalar_t x78 = POW2(x52 + x62);
    const scalar_t x79 = mu * x78;
    const scalar_t x80 = POW2(x56 + x67);
    const scalar_t x81 = mu * x80;
    const scalar_t x82 = POW2(x59 + x71);
    const scalar_t x83 = mu * x5;
    const scalar_t x84 = mu * x16;
    const scalar_t x85 = mu * x22;
    const scalar_t x86 = mu * x28;
    const scalar_t x87 = mu * x42;
    const scalar_t x88 = mu * x51;
    const scalar_t x89 = mu * x60;
    const scalar_t x90 = mu * x72;
    const scalar_t x91 = mu * x77;
    const scalar_t x92 = mu * x82;
    diag[0] += qw * (x11 * (x1 + x3 + x4 * x5));
    diag[1] += qw * (x17 * (x13 + x15 + x16 * x4));
    diag[2] += qw * (x23 * (x19 + x21 + x22 * x4));
    diag[3] += qw * (x29 * (x25 + x27 + x28 * x4));
    diag[4] += qw * (x43 * (x35 + x39 + x4 * x42));
    diag[5] += qw * (x43 * (x4 * x51 + x46 + x49));
    diag[6] += qw * (x43 * (x4 * x60 + x55 + x58));
    diag[7] += qw * (x43 * (x4 * x72 + x65 + x69));
    diag[8] += qw * (x43 * (x4 * x77 + x74 + x76));
    diag[9] += qw * (x43 * (x4 * x82 + x79 + x81));
    diag[10] += qw * (x11 * (x0 * x4 + x3 + x83));
    diag[11] += qw * (x17 * (x12 * x4 + x15 + x84));
    diag[12] += qw * (x23 * (x18 * x4 + x21 + x85));
    diag[13] += qw * (x29 * (x24 * x4 + x27 + x86));
    diag[14] += qw * (x43 * (x34 * x4 + x39 + x87));
    diag[15] += qw * (x43 * (x4 * x45 + x49 + x88));
    diag[16] += qw * (x43 * (x4 * x54 + x58 + x89));
    diag[17] += qw * (x43 * (x4 * x64 + x69 + x90));
    diag[18] += qw * (x43 * (x4 * x73 + x76 + x91));
    diag[19] += qw * (x43 * (x4 * x78 + x81 + x92));
    diag[20] += qw * (x11 * (x1 + x2 * x4 + x83));
    diag[21] += qw * (x17 * (x13 + x14 * x4 + x84));
    diag[22] += qw * (x23 * (x19 + x20 * x4 + x85));
    diag[23] += qw * (x29 * (x25 + x26 * x4 + x86));
    diag[24] += qw * (x43 * (x35 + x38 * x4 + x87));
    diag[25] += qw * (x43 * (x4 * x48 + x46 + x88));
    diag[26] += qw * (x43 * (x4 * x57 + x55 + x89));
    diag[27] += qw * (x43 * (x4 * x68 + x65 + x90));
    diag[28] += qw * (x43 * (x4 * x75 + x74 + x91));
    diag[29] += qw * (x43 * (x4 * x80 + x79 + x92));
}

static const int n_qp = 8;
static const scalar_t h_qx[8] =
        {0.0, 1.0, 0.0, 0.0, 0.333333333333, 0.333333333333, 0.0, 0.333333333333};

static const scalar_t h_qy[8] =
        {0.0, 0.0, 1.0, 0.0, 0.333333333333, 0.0, 0.333333333333, 0.333333333333};

static const scalar_t h_qz[8] =
        {0.0, 0.0, 0.0, 1.0, 0.0, 0.333333333333, 0.333333333333, 0.333333333333};

static const scalar_t h_qw[8] = {0.025, 0.025, 0.025, 0.025, 0.225, 0.225, 0.225, 0.225};

__constant__ scalar_t qx[8];
__constant__ scalar_t qy[8];
__constant__ scalar_t qz[8];
__constant__ scalar_t qw[8];

static void init_quadrature() {
    static bool initialized = false;
    if (!initialized) {
        SFEM_CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(qx), h_qx, 8 * sizeof(scalar_t)));
        SFEM_CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(qy), h_qy, 8 * sizeof(scalar_t)));
        SFEM_CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(qz), h_qz, 8 * sizeof(scalar_t)));
        SFEM_CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(qw), h_qw, 8 * sizeof(scalar_t)));
        initialized = true;
    }
}

template <typename T>
__global__ void cu_tet10_linear_elasticity_apply_kernel(
        const ptrdiff_t nelements,
        const ptrdiff_t stride,  // Stride for elements and fff
        const idx_t *const SFEM_RESTRICT elements,
        const cu_jacobian_t *const SFEM_RESTRICT g_jacobian_adjugate,
        const cu_jacobian_t *const SFEM_RESTRICT g_jacobian_determinant,
        const real_t mu,
        const real_t lambda,
        const ptrdiff_t u_stride,
        const T *const SFEM_RESTRICT g_ux,
        const T *const SFEM_RESTRICT g_uy,
        const T *const SFEM_RESTRICT g_uz,
        const ptrdiff_t out_stride,
        T *const SFEM_RESTRICT g_outx,
        T *const SFEM_RESTRICT g_outy,
        T *const SFEM_RESTRICT g_outz) {
    for (ptrdiff_t e = blockIdx.x * blockDim.x + threadIdx.x; e < nelements;
         e += blockDim.x * gridDim.x) {
        idx_t ev[10];

        // Sub-geometry
        scalar_t adjugate[9];
        scalar_t element_u[30];
        accumulator_t element_vector[30] = {0};

        // Copy over jacobian adjugate
        {
            const cu_jacobian_t *const jacobian_adjugate = &g_jacobian_adjugate[e];
            for (int i = 0; i < 9; i++) {
                adjugate[i] = jacobian_adjugate[i * stride];
            }
        }

#ifdef SFEM_ENABLE_FP32_KERNELS
        const scalar_t jacobian_determinant = 1;
#else
        const scalar_t jacobian_determinant = g_jacobian_determinant[e];
#endif

#pragma unroll(10)
        for (int v = 0; v < 10; ++v) {
            ev[v] = elements[v * stride + e];
        }

        for (int v = 0; v < 10; ++v) {
            element_u[v] = g_ux[ev[v] * u_stride];
            element_u[10 + v] = g_uy[ev[v] * u_stride];
            element_u[20 + v] = g_uz[ev[v] * u_stride];
        }

        for (int k = 0; k < n_qp; k++) {
            apply_micro_kernel(mu,
                               lambda,
                               adjugate,
                               jacobian_determinant,
                               qx[k],
                               qy[k],
                               qz[k],
                               qw[k],
                               element_u,
                               element_vector);
        }

#ifdef SFEM_ENABLE_FP32_KERNELS
        //
        {
            // real_t use here instead of scalar_t to have division in full precision
            const real_t jacobian_determinant = g_jacobian_determinant[e];

            for (int v = 0; v < 10; v++) {
                atomicAdd(&g_outx[ev[v] * out_stride], element_vector[v] / jacobian_determinant);
            }

            for (int v = 0; v < 10; v++) {
                atomicAdd(&g_outy[ev[v] * out_stride],
                          element_vector[10 + v] / jacobian_determinant);
            }

            for (int v = 0; v < 10; v++) {
                atomicAdd(&g_outz[ev[v] * out_stride],
                          element_vector[20 + v] / jacobian_determinant);
            }
        }
#else
        for (int v = 0; v < 10; v++) {
            atomicAdd(&g_outx[ev[v] * out_stride], element_vector[v]);
        }

        for (int v = 0; v < 10; v++) {
            atomicAdd(&g_outy[ev[v] * out_stride], element_vector[10 + v]);
        }

        for (int v = 0; v < 10; v++) {
            atomicAdd(&g_outz[ev[v] * out_stride], element_vector[20 + v]);
        }
#endif
    }
}

template <typename T>
static int cu_tet10_linear_elasticity_apply_tpl(
        const ptrdiff_t nelements,
        const ptrdiff_t stride,  // Stride for elements and fff
        const idx_t *const SFEM_RESTRICT elements,
        const cu_jacobian_t *const SFEM_RESTRICT jacobian_adjugate,
        const cu_jacobian_t *const SFEM_RESTRICT jacobian_determinant,
        const real_t mu,
        const real_t lambda,
        const ptrdiff_t u_stride,
        const T *const SFEM_RESTRICT ux,
        const T *const SFEM_RESTRICT uy,
        const T *const SFEM_RESTRICT uz,
        const ptrdiff_t out_stride,
        T *const SFEM_RESTRICT outx,
        T *const SFEM_RESTRICT outy,
        T *const SFEM_RESTRICT outz,
        void *stream) {
    int block_size = 128;
#ifdef SFEM_USE_OCCUPANCY_MAX_POTENTIAL
    {
        int min_grid_size;
        hipOccupancyMaxPotentialBlockSize(
                &min_grid_size, &block_size, cu_tet10_linear_elasticity_apply_kernel<T>, 0, 0);
    }
#endif  // SFEM_USE_OCCUPANCY_MAX_POTENTIAL

    ptrdiff_t n_blocks = MAX(ptrdiff_t(1), (nelements + block_size - 1) / block_size);

    if (stream) {
        hipStream_t s = *static_cast<hipStream_t *>(stream);
        cu_tet10_linear_elasticity_apply_kernel<<<n_blocks, block_size, 0, s>>>(
                nelements,
                stride,
                elements,
                jacobian_adjugate,
                jacobian_determinant,
                mu,
                lambda,
                u_stride,
                ux,
                uy,
                uz,
                out_stride,
                outx,
                outy,
                outz);
    } else {
        cu_tet10_linear_elasticity_apply_kernel<<<n_blocks, block_size, 0>>>(nelements,
                                                                             stride,
                                                                             elements,
                                                                             jacobian_adjugate,
                                                                             jacobian_determinant,
                                                                             mu,
                                                                             lambda,
                                                                             u_stride,
                                                                             ux,
                                                                             uy,
                                                                             uz,
                                                                             out_stride,
                                                                             outx,
                                                                             outy,
                                                                             outz);
    }

    SFEM_DEBUG_SYNCHRONIZE();
    return SFEM_SUCCESS;
}

extern int cu_tet10_linear_elasticity_apply(
        const ptrdiff_t nelements,
        const ptrdiff_t stride,  // Stride for elements and jacobian
        const idx_t *const SFEM_RESTRICT elements,
        const void *const SFEM_RESTRICT jacobian_adjugate,
        const void *const SFEM_RESTRICT jacobian_determinant,
        const real_t mu,
        const real_t lambda,
        const enum RealType real_type,
        const ptrdiff_t u_stride,
        const void *const SFEM_RESTRICT ux,
        const void *const SFEM_RESTRICT uy,
        const void *const SFEM_RESTRICT uz,
        const ptrdiff_t out_stride,
        void *const SFEM_RESTRICT outx,
        void *const SFEM_RESTRICT outy,
        void *const SFEM_RESTRICT outz,
        void *stream) {
    init_quadrature();

    switch (real_type) {
        case SFEM_REAL_DEFAULT: {
            return cu_tet10_linear_elasticity_apply_tpl(nelements,
                                                        stride,
                                                        elements,
                                                        (cu_jacobian_t *)jacobian_adjugate,
                                                        (cu_jacobian_t *)jacobian_determinant,
                                                        mu,
                                                        lambda,
                                                        u_stride,
                                                        (real_t *)ux,
                                                        (real_t *)uy,
                                                        (real_t *)uz,
                                                        out_stride,
                                                        (real_t *)outx,
                                                        (real_t *)outy,
                                                        (real_t *)outz,
                                                        stream);
        }
        case SFEM_FLOAT32: {
            return cu_tet10_linear_elasticity_apply_tpl(nelements,
                                                        stride,
                                                        elements,
                                                        (cu_jacobian_t *)jacobian_adjugate,
                                                        (cu_jacobian_t *)jacobian_determinant,
                                                        mu,
                                                        lambda,
                                                        u_stride,
                                                        (float *)ux,
                                                        (float *)uy,
                                                        (float *)uz,
                                                        out_stride,
                                                        (float *)outx,
                                                        (float *)outy,
                                                        (float *)outz,
                                                        stream);
        }
        case SFEM_FLOAT64: {
            return cu_tet10_linear_elasticity_apply_tpl(nelements,
                                                        stride,
                                                        elements,
                                                        (cu_jacobian_t *)jacobian_adjugate,
                                                        (cu_jacobian_t *)jacobian_determinant,
                                                        mu,
                                                        lambda,
                                                        u_stride,
                                                        (double *)ux,
                                                        (double *)uy,
                                                        (double *)uz,
                                                        out_stride,
                                                        (double *)outx,
                                                        (double *)outy,
                                                        (double *)outz,
                                                        stream);
        }
        default: {
            fprintf(stderr,
                    "[Error] cu_tet10_linear_elasticity_apply: not implemented for type %s "
                    "(code %d)\n",
                    real_type_to_string(real_type),
                    real_type);
            assert(0);
            return SFEM_FAILURE;
        }
    }
}

/// --- DIAG

template <typename T>
__global__ void cu_tet10_linear_elasticity_diag_kernel(
        const ptrdiff_t nelements,
        const ptrdiff_t stride,  // Stride for elements and fff
        const idx_t *const SFEM_RESTRICT elements,
        const cu_jacobian_t *const SFEM_RESTRICT g_jacobian_adjugate,
        const cu_jacobian_t *const SFEM_RESTRICT g_jacobian_determinant,
        const real_t mu,
        const real_t lambda,
        const ptrdiff_t diag_stride,
        T *const SFEM_RESTRICT g_diagx,
        T *const SFEM_RESTRICT g_diagy,
        T *const SFEM_RESTRICT g_diagz) {
    for (ptrdiff_t e = blockIdx.x * blockDim.x + threadIdx.x; e < nelements;
         e += blockDim.x * gridDim.x) {
        idx_t ev[10];

        // Sub-geometry
        scalar_t adjugate[9];
        accumulator_t element_vector[30] = {0};

        // Copy over jacobian adjugate
        {
            const cu_jacobian_t *const jacobian_adjugate = &g_jacobian_adjugate[e];
            for (int i = 0; i < 9; i++) {
                adjugate[i] = jacobian_adjugate[i * stride];
            }
        }

#ifdef SFEM_ENABLE_FP32_KERNELS
        const scalar_t jacobian_determinant = 1;
#else
        const scalar_t jacobian_determinant = g_jacobian_determinant[e];
#endif

#pragma unroll(10)
        for (int v = 0; v < 10; ++v) {
            ev[v] = elements[v * stride + e];
        }

        for (int k = 0; k < n_qp; k++) {
            diag_micro_kernel(mu,
                              lambda,
                              adjugate,
                              jacobian_determinant,
                              qx[k],
                              qy[k],
                              qz[k],
                              qw[k],
                              element_vector);
        }

#ifdef SFEM_ENABLE_FP32_KERNELS
        //
        {
            // real_t use here instead of scalar_t to have division in full precision
            const real_t jacobian_determinant = g_jacobian_determinant[e];

            for (int v = 0; v < 10; v++) {
                atomicAdd(&g_diagx[ev[v] * diag_stride], element_vector[v] / jacobian_determinant);
            }

            for (int v = 0; v < 10; v++) {
                atomicAdd(&g_diagy[ev[v] * diag_stride],
                          element_vector[10 + v] / jacobian_determinant);
            }

            for (int v = 0; v < 10; v++) {
                atomicAdd(&g_diagz[ev[v] * diag_stride],
                          element_vector[20 + v] / jacobian_determinant);
            }
        }
#else

        for (int v = 0; v < 10; v++) {
            atomicAdd(&g_diagx[ev[v] * diag_stride], element_vector[v]);
        }

        for (int v = 0; v < 10; v++) {
            atomicAdd(&g_diagy[ev[v] * diag_stride], element_vector[10 + v]);
        }

        for (int v = 0; v < 10; v++) {
            atomicAdd(&g_diagz[ev[v] * diag_stride], element_vector[20 + v]);
        }
#endif
    }
}

template <typename T>
static int cu_tet10_linear_elasticity_diag_tpl(
        const ptrdiff_t nelements,
        const ptrdiff_t stride,  // Stride for elements and fff
        const idx_t *const SFEM_RESTRICT elements,
        const cu_jacobian_t *const SFEM_RESTRICT jacobian_adjugate,
        const cu_jacobian_t *const SFEM_RESTRICT jacobian_determinant,
        const real_t mu,
        const real_t lambda,
        const ptrdiff_t diag_stride,
        T *const SFEM_RESTRICT diagx,
        T *const SFEM_RESTRICT diagy,
        T *const SFEM_RESTRICT diagz,
        void *stream) {
    int block_size = 128;
#ifdef SFEM_USE_OCCUPANCY_MAX_POTENTIAL
    {
        int min_grid_size;
        hipOccupancyMaxPotentialBlockSize(
                &min_grid_size, &block_size, cu_tet10_linear_elasticity_diag_kernel<T>, 0, 0);
    }
#endif  // SFEM_USE_OCCUPANCY_MAX_POTENTIAL

    ptrdiff_t n_blocks = MAX(ptrdiff_t(1), (nelements + block_size - 1) / block_size);

    if (stream) {
        hipStream_t s = *static_cast<hipStream_t *>(stream);
        cu_tet10_linear_elasticity_diag_kernel<<<n_blocks, block_size, 0, s>>>(nelements,
                                                                               stride,
                                                                               elements,
                                                                               jacobian_adjugate,
                                                                               jacobian_determinant,
                                                                               mu,
                                                                               lambda,
                                                                               diag_stride,
                                                                               diagx,
                                                                               diagy,
                                                                               diagz);
    } else {
        cu_tet10_linear_elasticity_diag_kernel<<<n_blocks, block_size, 0>>>(nelements,
                                                                            stride,
                                                                            elements,
                                                                            jacobian_adjugate,
                                                                            jacobian_determinant,
                                                                            mu,
                                                                            lambda,
                                                                            diag_stride,
                                                                            diagx,
                                                                            diagy,
                                                                            diagz);
    }

    SFEM_DEBUG_SYNCHRONIZE();
    return SFEM_SUCCESS;
}

extern int cu_tet10_linear_elasticity_diag(const ptrdiff_t nelements,
                                           const ptrdiff_t stride,  // Stride for elements and fff
                                           const idx_t *const SFEM_RESTRICT elements,
                                           const void *const SFEM_RESTRICT jacobian_adjugate,
                                           const void *const SFEM_RESTRICT jacobian_determinant,
                                           const real_t mu,
                                           const real_t lambda,
                                           const enum RealType real_type,
                                           const ptrdiff_t diag_stride,
                                           void *const SFEM_RESTRICT diagx,
                                           void *const SFEM_RESTRICT diagy,
                                           void *const SFEM_RESTRICT diagz,
                                           void *stream) {
    init_quadrature();

    switch (real_type) {
        case SFEM_REAL_DEFAULT: {
            return cu_tet10_linear_elasticity_diag_tpl(nelements,
                                                       stride,
                                                       elements,
                                                       (cu_jacobian_t *)jacobian_adjugate,
                                                       (cu_jacobian_t *)jacobian_determinant,
                                                       mu,
                                                       lambda,
                                                       diag_stride,
                                                       (real_t *)diagx,
                                                       (real_t *)diagy,
                                                       (real_t *)diagz,
                                                       stream);
        }
        case SFEM_FLOAT32: {
            return cu_tet10_linear_elasticity_diag_tpl(nelements,
                                                       stride,
                                                       elements,
                                                       (cu_jacobian_t *)jacobian_adjugate,
                                                       (cu_jacobian_t *)jacobian_determinant,
                                                       mu,
                                                       lambda,
                                                       diag_stride,
                                                       (float *)diagx,
                                                       (float *)diagy,
                                                       (float *)diagz,
                                                       stream);
        }
        case SFEM_FLOAT64: {
            return cu_tet10_linear_elasticity_diag_tpl(nelements,
                                                       stride,
                                                       elements,
                                                       (cu_jacobian_t *)jacobian_adjugate,
                                                       (cu_jacobian_t *)jacobian_determinant,
                                                       mu,
                                                       lambda,
                                                       diag_stride,
                                                       (double *)diagx,
                                                       (double *)diagy,
                                                       (double *)diagz,
                                                       stream);
        }
        default: {
            fprintf(stderr,
                    "[Error] cu_tet10_linear_elasticity_diag: not implemented for type %s "
                    "(code %d)\n",
                    real_type_to_string(real_type),
                    real_type);
            assert(0);
            return SFEM_FAILURE;
        }
    }
}