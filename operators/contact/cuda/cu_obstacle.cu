#include "hip/hip_runtime.h"
#include "cu_obstacle.h"

#include "sfem_macros.h"
#include "sfem_cuda_base.h"

__global__ void obstacle_normal_project_kernel(const int                         dim,
                                               const ptrdiff_t                   n,
                                               const idx_t *const SFEM_RESTRICT  idx,
                                               real_t **const SFEM_RESTRICT      normals,
                                               const real_t *const SFEM_RESTRICT h,
                                               real_t *const SFEM_RESTRICT       out) {
    for (ptrdiff_t i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockIdx.x) {
        const ptrdiff_t     ii  = idx[i] * dim;
        const real_t *const hii = &h[ii];
        for (int d = 0; d < dim; d++) {
            out[i] += hii[d] * normals[d][i];
        }
    }
}

__global__ void obstacle_distribute_contact_forces_kernel(const int                         dim,
                                                          const ptrdiff_t                   n,
                                                          const idx_t *const SFEM_RESTRICT  idx,
                                                          real_t **const SFEM_RESTRICT      normals,
                                                          const real_t *const SFEM_RESTRICT m,
                                                          const real_t *const               f,
                                                          real_t *const                     out) {
    for (ptrdiff_t i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockIdx.x) {
        const ptrdiff_t ii  = idx[i] * dim;
        real_t *const   oii = &out[ii];
        const real_t    fi  = f[i] * m[i];
        for (int d = 0; d < dim; d++) {
            oii[d] += normals[d][i] * fi;
        }
    }
}

extern int cu_obstacle_normal_project(const int                         dim,
                                      const ptrdiff_t                   n,
                                      const idx_t *const SFEM_RESTRICT  idx,
                                      real_t **const SFEM_RESTRICT      normals,
                                      const real_t *const SFEM_RESTRICT h,
                                      real_t *const SFEM_RESTRICT       out) {
    SFEM_DEBUG_SYNCHRONIZE();

    int             block_size = 128;
    const ptrdiff_t n_blocks   = MAX(ptrdiff_t(1), (n + block_size - 1) / block_size);
    obstacle_normal_project_kernel<<<n_blocks, block_size, 0>>>(dim, n, idx, normals, h, out);

    SFEM_DEBUG_SYNCHRONIZE();
    return SFEM_SUCCESS;
}

extern int cu_obstacle_distribute_contact_forces(const int                         dim,
                                                 const ptrdiff_t                   n,
                                                 const idx_t *const SFEM_RESTRICT  idx,
                                                 real_t **const SFEM_RESTRICT      normals,
                                                 const real_t *const SFEM_RESTRICT m,
                                                 const real_t *const               f,
                                                 real_t *const                     out) {
    SFEM_DEBUG_SYNCHRONIZE();

    int             block_size = 128;
    const ptrdiff_t n_blocks   = MAX(ptrdiff_t(1), (n + block_size - 1) / block_size);
    obstacle_distribute_contact_forces_kernel<<<n_blocks, block_size, 0>>>(dim, n, idx, normals, m, f, out);

    SFEM_DEBUG_SYNCHRONIZE();
    return SFEM_SUCCESS;
}
