#include "hip/hip_runtime.h"
#include "cu_sshex8_laplacian.h"

#include "sfem_cuda_base.h"
#include "cu_sshex8_inline.hpp"
#include "cu_hex8_laplacian_inline.hpp"

template <typename real_t>
__global__ void cu_affine_sshex8_laplacian_apply_kernel(
        const int level,
        const ptrdiff_t nelements,
        const ptrdiff_t stride,  // Stride for elements and fff
        const idx_t *const SFEM_RESTRICT elements,
        const cu_jacobian_t *const SFEM_RESTRICT fff,
        const real_t *const SFEM_RESTRICT x,
        real_t *const SFEM_RESTRICT y) {
    scalar_t laplacian_matrix[8 * 8];
#ifndef NDEBUG
    const int nxe = cu_sshex8_nxe(level);
#endif

    for (ptrdiff_t e = blockIdx.x * blockDim.x + threadIdx.x; e < nelements;
         e += blockDim.x * gridDim.x) {
        // Build operator
        {
            scalar_t sub_fff[6];
            const scalar_t h = 1. / level;
            cu_hex8_sub_fff_0(stride, &fff[e], h, sub_fff);
            cu_hex8_laplacian_matrix_fff_integral(sub_fff, laplacian_matrix);
        }

        // Iterate over sub-elements
        for (int zi = 0; zi < level; zi++) {
            for (int yi = 0; yi < level; yi++) {
                for (int xi = 0; xi < level; xi++) {
                    assert(cu_sshex8_lidx(level, xi + 1, yi + 1, zi + 1) < nxe);

                    int ev[8] = {
                            // Bottom
                            elements[cu_sshex8_lidx(level, xi, yi, zi) * stride + e],
                            elements[cu_sshex8_lidx(level, xi + 1, yi, zi) * stride + e],
                            elements[cu_sshex8_lidx(level, xi + 1, yi + 1, zi) * stride + e],
                            elements[cu_sshex8_lidx(level, xi, yi + 1, zi) * stride + e],
                            // Top
                            elements[cu_sshex8_lidx(level, xi, yi, zi + 1) * stride + e],
                            elements[cu_sshex8_lidx(level, xi + 1, yi, zi + 1) * stride + e],
                            elements[cu_sshex8_lidx(level, xi + 1, yi + 1, zi + 1) * stride +
                                     e],
                            elements[cu_sshex8_lidx(level, xi, yi + 1, zi + 1) * stride + e]};

                    scalar_t element_u[8];

                    for (int d = 0; d < 8; d++) {
                        element_u[d] = x[ev[d]];
                    }

                    scalar_t element_vector[8];
                    for (int i = 0; i < 8; i++) {
                        element_vector[i] = 0;
                    }

                    for (int i = 0; i < 8; i++) {
                        const scalar_t *const row = &laplacian_matrix[i * 8];
                        const scalar_t ui = element_u[i];
                        assert(ui == ui);
                        for (int j = 0; j < 8; j++) {
                            assert(row[j] == row[j]);
                            element_vector[j] += ui * row[j];
                        }
                    }

                    for (int d = 0; d < 8; d++) {
                        assert(element_vector[d] == element_vector[d]);
                        atomicAdd(&y[ev[d]], element_vector[d]);
                    }
                }
            }
        }
    }
}

#define B_(x, y, z) ((z)*BLOCK_SIZE_2 + (y)*BLOCK_SIZE + (x))

template <typename real_t, int LEVEL>
__global__ void cu_affine_sshex8_laplacian_apply_kernel_fixed(
        const ptrdiff_t nelements,
        const ptrdiff_t stride,  // Stride for elements and fff
        const idx_t *const SFEM_RESTRICT elements,
        const cu_jacobian_t *const SFEM_RESTRICT fff,
        const real_t *const SFEM_RESTRICT x,
        real_t *const SFEM_RESTRICT y) {
#ifndef NDEBUG
    const int nxe = cu_sshex8_nxe(LEVEL);
#endif

    static const int BLOCK_SIZE = LEVEL + 1;
    static const int BLOCK_SIZE_2 = BLOCK_SIZE * BLOCK_SIZE;
    static const int BLOCK_SIZE_3 = BLOCK_SIZE_2 * BLOCK_SIZE;

    // Uses "local" memory
    scalar_t x_block[BLOCK_SIZE_3];
    scalar_t y_block[BLOCK_SIZE_3];
    scalar_t laplacian_matrix[8 * 8];

    for (ptrdiff_t e = blockIdx.x * blockDim.x + threadIdx.x; e < nelements;
         e += blockDim.x * gridDim.x) {
        // Build operator
        {
            scalar_t sub_fff[6];
            const scalar_t h = 1. / LEVEL;
            cu_hex8_sub_fff_0(stride, &fff[e], h, sub_fff);
            cu_hex8_laplacian_matrix_fff_integral(sub_fff, laplacian_matrix);
        }

        // Gather
        for (int zi = 0; zi < BLOCK_SIZE; zi++) {
            for (int yi = 0; yi < BLOCK_SIZE; yi++) {
                for (int xi = 0; xi < BLOCK_SIZE; xi++) {
                    const int lidx = cu_sshex8_lidx(LEVEL, xi, yi, zi);
                    assert(lidx < nxe);
                    const idx_t idx = elements[lidx * stride + e];
                    x_block[B_(xi, yi, zi)] = x[idx];
                }
            }
        }

        // Reset
        for (int i = 0; i < BLOCK_SIZE_3; i++) {
            y_block[i] = 0;
        }

        // Compute
        for (int zi = 0; zi < BLOCK_SIZE - 1; zi++) {
            for (int yi = 0; yi < BLOCK_SIZE - 1; yi++) {
                for (int xi = 0; xi < BLOCK_SIZE - 1; xi++) {
                    assert(B_(xi + 1, yi + 1, zi + 1) < BLOCK_SIZE_3);

                    scalar_t element_u[8] = {x_block[B_(xi, yi, zi)],
                                             x_block[B_(xi + 1, yi, zi)],
                                             x_block[B_(xi + 1, yi + 1, zi)],
                                             x_block[B_(xi, yi + 1, zi)],
                                             x_block[B_(xi, yi, zi + 1)],
                                             x_block[B_(xi + 1, yi, zi + 1)],
                                             x_block[B_(xi + 1, yi + 1, zi + 1)],
                                             x_block[B_(xi, yi + 1, zi + 1)]};

                    scalar_t element_vector[8] = {0};
                    for (int i = 0; i < 8; i++) {
                        const scalar_t *const row = &laplacian_matrix[i * 8];
                        const scalar_t ui = element_u[i];
                        assert(ui == ui);
                        for (int j = 0; j < 8; j++) {
                            assert(row[j] == row[j]);
                            element_vector[j] += ui * row[j];
                        }
                    }

                    y_block[B_(xi, yi, zi)] += element_vector[0];
                    y_block[B_(xi + 1, yi, zi)] += element_vector[1];
                    y_block[B_(xi + 1, yi + 1, zi)] += element_vector[2];
                    y_block[B_(xi, yi + 1, zi)] += element_vector[3];
                    y_block[B_(xi, yi, zi + 1)] += element_vector[4];
                    y_block[B_(xi + 1, yi, zi + 1)] += element_vector[5];
                    y_block[B_(xi + 1, yi + 1, zi + 1)] += element_vector[6];
                    y_block[B_(xi, yi + 1, zi + 1)] += element_vector[7];
                }
            }
        }

        // Scatter
        for (int zi = 0; zi < BLOCK_SIZE; zi++) {
            for (int yi = 0; yi < BLOCK_SIZE; yi++) {
                for (int xi = 0; xi < BLOCK_SIZE; xi++) {
                    const int lidx = cu_sshex8_lidx(LEVEL, xi, yi, zi);

                    assert(lidx < nxe);
                    const idx_t idx = elements[lidx * stride + e];
                    atomicAdd(&y[idx], y_block[B_(xi, yi, zi)]);
                }
            }
        }
    }
}

template <typename T, int LEVEL>
static int cu_affine_sshex8_laplacian_apply_fixed_tpl(
        const ptrdiff_t nelements,
        const ptrdiff_t stride,  // Stride for elements and fff
        const idx_t *const SFEM_RESTRICT elements,
        const cu_jacobian_t *const SFEM_RESTRICT fff,
        const T *const x,
        T *const y,
        void *stream) {
    int block_size = 128;
#ifdef SFEM_USE_OCCUPANCY_MAX_POTENTIAL
    {
        int min_grid_size;
        hipOccupancyMaxPotentialBlockSize(
                &min_grid_size,
                &block_size,
                cu_affine_sshex8_laplacian_apply_kernel_fixed<T, LEVEL>,
                0,
                0);
    }
#endif  // SFEM_USE_OCCUPANCY_MAX_POTENTIAL

    const ptrdiff_t n_blocks = MAX(ptrdiff_t(1), (nelements + block_size - 1) / block_size);

    if (stream) {
        hipStream_t s = *static_cast<hipStream_t *>(stream);
        cu_affine_sshex8_laplacian_apply_kernel_fixed<T, LEVEL>
                <<<n_blocks, block_size, 0, s>>>(nelements, stride, elements, fff, x, y);
    } else {
        cu_affine_sshex8_laplacian_apply_kernel_fixed<T, LEVEL>
                <<<n_blocks, block_size, 0>>>(nelements, stride, elements, fff, x, y);
    }

    SFEM_DEBUG_SYNCHRONIZE();
    return SFEM_SUCCESS;
}

#include "cu_sshex8_laplacian_variants.hpp"
#include "cu_sshex8_laplacian_warp.hpp"

#define my_kernel_ cu_affine_sshex8_laplacian_apply_kernel

template <typename T>
static int cu_affine_sshex8_laplacian_apply_tpl(
        const int level,
        const ptrdiff_t nelements,
        const ptrdiff_t stride,  // Stride for elements and fff
        const ptrdiff_t interior_start,
        const idx_t *const SFEM_RESTRICT elements,
        const cu_jacobian_t *const SFEM_RESTRICT fff,
        const T *const x,
        T *const y,
        void *stream) {
    SFEM_DEBUG_SYNCHRONIZE();

    int SFEM_HEX8_WARP_LEVEL_KERNEL=1;
    SFEM_READ_ENV(SFEM_HEX8_WARP_LEVEL_KERNEL, atoi);

    if(SFEM_HEX8_WARP_LEVEL_KERNEL) 
    {
        switch (level) {
            // case 2: {
            //     return cu_affine_sshex8_laplacian_apply_warp_tpl<T, 2>(
            //             nelements, stride, interior_start, elements, fff, x, y, stream);
            // }
            case 4: {
                return cu_affine_sshex8_laplacian_apply_warp_tpl<T, 4>(
                        nelements, stride, interior_start, elements, fff, x, y, stream);
            }
            case 6: {
                return cu_affine_sshex8_laplacian_apply_warp_tpl<T, 6>(
                        nelements, stride, interior_start, elements, fff, x, y, stream);
            }
            // case 7: {
            //     return cu_affine_sshex8_laplacian_apply_warp_tpl<T, 7>(
            //             nelements, stride, interior_start, elements, fff, x, y, stream);
            // }
            case 8: {
                return cu_affine_sshex8_laplacian_apply_warp_tpl<T, 8>(
                        nelements, stride, interior_start, elements, fff, x, y, stream);
            }
            // case 9: {
            //     return cu_affine_sshex8_laplacian_apply_warp_tpl<T, 9>(
            //             nelements, stride, interior_start, elements, fff, x, y, stream);
            // }
            case 10: {
                return cu_affine_sshex8_laplacian_apply_warp_tpl<T, 10>(
                        nelements, stride, interior_start, elements, fff, x, y, stream);
            }
            default:
                break;
        }

    }

#if 1
    switch (level) {
        // case 2: {
        //     return cu_affine_sshex8_laplacian_apply_volgen_tpl<T, 2>(
        //             nelements, stride, interior_start, elements, fff, x, y, stream);
        // }
        case 4: {
            return cu_affine_sshex8_laplacian_apply_volgen_tpl<T, 4>(
                    nelements, stride, interior_start, elements, fff, x, y, stream);
        }
        case 6: {
            return cu_affine_sshex8_laplacian_apply_volgen_tpl<T, 6>(
                    nelements, stride, interior_start, elements, fff, x, y, stream);
        }
        // case 7: {
        //     return cu_affine_sshex8_laplacian_apply_volgen_tpl<T, 7>(
        //             nelements, stride, interior_start, elements, fff, x, y, stream);
        // }
        case 8: {
            return cu_affine_sshex8_laplacian_apply_volgen_tpl<T, 8>(
                    nelements, stride, interior_start, elements, fff, x, y, stream);
        }
        // case 9: {
        //     return cu_affine_sshex8_laplacian_apply_volgen_tpl<T, 9>(
        //             nelements, stride, interior_start, elements, fff, x, y, stream);
        // }
        // case 10: {
        //     return cu_affine_sshex8_laplacian_apply_volgen_tpl<T, 10>(
        //             nelements, stride, interior_start, elements, fff, x, y, stream);
        // }
        default:
            break;
    }

#else

    switch (level) {
        case 2: {
            return cu_affine_sshex8_laplacian_apply_fixed_tpl<T, 2>(
                    nelements, stride, elements, fff, x, y, stream);
        }
        case 4: {
            return cu_affine_sshex8_laplacian_apply_fixed_tpl<T, 4>(
                    nelements, stride, elements, fff, x, y, stream);
        }
        case 6: {
            return cu_affine_sshex8_laplacian_apply_fixed_tpl<T, 6>(
                    nelements, stride, elements, fff, x, y, stream);
        }
        case 7: {
            return cu_affine_sshex8_laplacian_apply_fixed_tpl<T, 7>(
                    nelements, stride, elements, fff, x, y, stream);
        }
        case 8: {
            return cu_affine_sshex8_laplacian_apply_fixed_tpl<T, 8>(
                    nelements, stride, elements, fff, x, y, stream);
        }
        case 9: {
            return cu_affine_sshex8_laplacian_apply_fixed_tpl<T, 9>(
                    nelements, stride, elements, fff, x, y, stream);
        }
        case 10: {
            return cu_affine_sshex8_laplacian_apply_fixed_tpl<T, 10>(
                    nelements, stride, elements, fff, x, y, stream);
        }
        default:
            break;
    }
#endif

    // Hand tuned
    int block_size = 128;
#ifdef SFEM_USE_OCCUPANCY_MAX_POTENTIAL
    {
        int min_grid_size;
        hipOccupancyMaxPotentialBlockSize(&min_grid_size, &block_size, my_kernel_<T>, 0, 0);
    }
#endif  // SFEM_USE_OCCUPANCY_MAX_POTENTIAL

    const ptrdiff_t n_blocks = MAX(ptrdiff_t(1), (nelements + block_size - 1) / block_size);

    if (stream) {
        hipStream_t s = *static_cast<hipStream_t *>(stream);
        my_kernel_<<<n_blocks, block_size, 0, s>>>(level, nelements, stride, elements, fff, x, y);
    } else {
        my_kernel_<<<n_blocks, block_size, 0>>>(level, nelements, stride, elements, fff, x, y);
    }

    SFEM_DEBUG_SYNCHRONIZE();
    return SFEM_SUCCESS;
}

extern int cu_affine_sshex8_laplacian_apply(
        const int level,
        const ptrdiff_t nelements,
        const ptrdiff_t stride,  // Stride for elements and fff
        const ptrdiff_t interior_start,
        const idx_t *const SFEM_RESTRICT elements,
        const void *const SFEM_RESTRICT fff,
        const enum RealType real_type_xy,
        const void *const x,
        void *const y,
        void *stream) {
    switch (real_type_xy) {
        case SFEM_REAL_DEFAULT: {
            return cu_affine_sshex8_laplacian_apply_tpl(level,
                                                              nelements,
                                                              stride,
                                                              interior_start,
                                                              elements,
                                                              (cu_jacobian_t *)fff,
                                                              (real_t *)x,
                                                              (real_t *)y,
                                                              stream);
        }
        case SFEM_FLOAT32: {
            return cu_affine_sshex8_laplacian_apply_tpl(level,
                                                              nelements,
                                                              stride,
                                                              interior_start,
                                                              elements,
                                                              (cu_jacobian_t *)fff,
                                                              (float *)x,
                                                              (float *)y,
                                                              stream);
        }
        case SFEM_FLOAT64: {
            return cu_affine_sshex8_laplacian_apply_tpl(level,
                                                              nelements,
                                                              stride,
                                                              interior_start,
                                                              elements,
                                                              (cu_jacobian_t *)fff,
                                                              (double *)x,
                                                              (double *)y,
                                                              stream);
        }
        default: {
            fprintf(stderr,
                    "[Error] cu_tet4_laplacian_apply: not implemented for type %s (code %d)\n",
                    real_type_to_string(real_type_xy),
                    real_type_xy);
            assert(0);
            return SFEM_FAILURE;
        }
    }
}

#undef B_
#undef my_kernel_
