#include "hip/hip_runtime.h"
#include "cu_sshex8_laplacian.h"

#include "cu_hex8_laplacian_inline.hpp"
#include "cu_sshex8_inline.hpp"
#include "sfem_cuda_base.h"

// #define CU_AFFINE_SSHEX8_LAPLACIAN_APPLY_USE_ELEMENTAL_MATRIX

template <typename real_t>
__global__ void cu_affine_sshex8_laplacian_apply_kernel(const int                        level,
                                                        const ptrdiff_t                  nelements,
                                                        const ptrdiff_t                  stride,  // Stride for elements and fff
                                                        const idx_t *const SFEM_RESTRICT elements,
                                                        const cu_jacobian_t *const SFEM_RESTRICT fff,
                                                        const real_t *const SFEM_RESTRICT        x,
                                                        real_t *const SFEM_RESTRICT              y) {
#ifndef NDEBUG
    const int nxe = cu_sshex8_nxe(level);
#endif

    for (ptrdiff_t e = blockIdx.x * blockDim.x + threadIdx.x; e < nelements; e += blockDim.x * gridDim.x) {
#ifdef CU_AFFINE_SSHEX8_LAPLACIAN_APPLY_USE_ELEMENTAL_MATRIX
        scalar_t laplacian_matrix[8 * 8];
        // Build operator
        {
            scalar_t       sub_fff[6];
            const scalar_t h = 1. / level;
            cu_hex8_sub_fff_0(stride, &fff[e], h, sub_fff);
            cu_hex8_laplacian_matrix_fff_integral(sub_fff, laplacian_matrix);
        }
#else
        scalar_t sub_fff[6];
        {
            const scalar_t h = 1. / level;
            cu_hex8_sub_fff_0(stride, &fff[e], h, sub_fff);
        }
#endif

        // Iterate over sub-elements
        for (int zi = 0; zi < level; zi++) {
            for (int yi = 0; yi < level; yi++) {
                for (int xi = 0; xi < level; xi++) {
                    assert(cu_sshex8_lidx(level, xi + 1, yi + 1, zi + 1) < nxe);

                    int ev[8] = {// Bottom
                                 elements[cu_sshex8_lidx(level, xi, yi, zi) * stride + e],
                                 elements[cu_sshex8_lidx(level, xi + 1, yi, zi) * stride + e],
                                 elements[cu_sshex8_lidx(level, xi + 1, yi + 1, zi) * stride + e],
                                 elements[cu_sshex8_lidx(level, xi, yi + 1, zi) * stride + e],
                                 // Top
                                 elements[cu_sshex8_lidx(level, xi, yi, zi + 1) * stride + e],
                                 elements[cu_sshex8_lidx(level, xi + 1, yi, zi + 1) * stride + e],
                                 elements[cu_sshex8_lidx(level, xi + 1, yi + 1, zi + 1) * stride + e],
                                 elements[cu_sshex8_lidx(level, xi, yi + 1, zi + 1) * stride + e]};

                    scalar_t element_u[8];
                    for (int d = 0; d < 8; d++) {
                        element_u[d] = x[ev[d]];
                    }

                    scalar_t element_vector[8];

#ifdef CU_AFFINE_SSHEX8_LAPLACIAN_APPLY_USE_ELEMENTAL_MATRIX
                    for (int i = 0; i < 8; i++) {
                        element_vector[i] = 0;
                    }

                    for (int i = 0; i < 8; i++) {
                        const scalar_t *const row = &laplacian_matrix[i * 8];
                        const scalar_t        ui  = element_u[i];
                        assert(ui == ui);
                        for (int j = 0; j < 8; j++) {
                            assert(row[j] == row[j]);
                            element_vector[j] += ui * row[j];
                        }
                    }
#else
                    cu_hex8_laplacian_apply_fff_integral(sub_fff, element_u, element_vector);
#endif

                    for (int d = 0; d < 8; d++) {
                        assert(element_vector[d] == element_vector[d]);
                        atomicAdd(&y[ev[d]], element_vector[d]);
                    }
                }
            }
        }
    }
}

#include "cu_sshex8_laplacian_variants.hpp"
#include "cu_sshex8_laplacian_warp.hpp"

template <typename T>
static int cu_affine_sshex8_laplacian_apply_tpl(const int                                level,
                                                const ptrdiff_t                          nelements,
                                                const ptrdiff_t                          stride,  // Stride for elements and fff
                                                const ptrdiff_t                          interior_start,
                                                const idx_t *const SFEM_RESTRICT         elements,
                                                const cu_jacobian_t *const SFEM_RESTRICT fff,
                                                const T *const                           x,
                                                T *const                                 y,
                                                void                                    *stream) {
    SFEM_DEBUG_SYNCHRONIZE();

    int SFEM_HEX8_SHARED_MEM_KERNEL = 1;
    SFEM_READ_ENV(SFEM_HEX8_SHARED_MEM_KERNEL, atoi);

    if (SFEM_HEX8_SHARED_MEM_KERNEL) {
        switch (level) {
            case 4: {
                return cu_affine_sshex8_laplacian_apply_warp_tpl<T, 4>(
                        nelements, stride, interior_start, elements, fff, x, y, stream);
            }
            case 6: {
                return cu_affine_sshex8_laplacian_apply_warp_tpl<T, 6>(
                        nelements, stride, interior_start, elements, fff, x, y, stream);
            }
            case 8: {
                return cu_affine_sshex8_laplacian_apply_warp_tpl<T, 8>(
                        nelements, stride, interior_start, elements, fff, x, y, stream);
            }
            default:
                break;
        }
    }

    int SFEM_HEX8_LOCAL_MEM_KERNEL = 1;
    SFEM_READ_ENV(SFEM_HEX8_LOCAL_MEM_KERNEL, atoi);
    if (SFEM_HEX8_LOCAL_MEM_KERNEL) {
        switch (level) {
            case 2: {
                return acu_affine_sshex8_laplacian_apply_local_mem_tpl<T, 2>(
                        nelements, stride, interior_start, elements, fff, x, y, stream);
            }
#if 0  // Disabled to reduce compilation times (warp level version is the fastest)
            case 4: {
                return acu_affine_sshex8_laplacian_apply_local_mem_tpl<T, 4>(
                        nelements, stride, interior_start, elements, fff, x, y, stream);
            }
            case 6: {
                return acu_affine_sshex8_laplacian_apply_local_mem_tpl<T, 6>(
                        nelements, stride, interior_start, elements, fff, x, y, stream);
            }
            case 8: {
                return acu_affine_sshex8_laplacian_apply_local_mem_tpl<T, 8>(
                        nelements, stride, interior_start, elements, fff, x, y, stream);
            }
#endif
            case 16: {
                return acu_affine_sshex8_laplacian_apply_local_mem_tpl<T, 16>(
                        nelements, stride, interior_start, elements, fff, x, y, stream);
            }

            default:
                break;
        }
    }

    // Hand tuned
    int block_size = 128;
#ifdef SFEM_USE_OCCUPANCY_MAX_POTENTIAL
    {
        int min_grid_size;
        hipOccupancyMaxPotentialBlockSize(&min_grid_size, &block_size, cu_affine_sshex8_laplacian_apply_kernel<T>, 0, 0);
    }
#endif  // SFEM_USE_OCCUPANCY_MAX_POTENTIAL

    const ptrdiff_t n_blocks = MAX(ptrdiff_t(1), (nelements + block_size - 1) / block_size);

    if (stream) {
        hipStream_t s = *static_cast<hipStream_t *>(stream);
        cu_affine_sshex8_laplacian_apply_kernel<<<n_blocks, block_size, 0, s>>>(level, nelements, stride, elements, fff, x, y);
    } else {
        cu_affine_sshex8_laplacian_apply_kernel<<<n_blocks, block_size, 0>>>(level, nelements, stride, elements, fff, x, y);
    }

    SFEM_DEBUG_SYNCHRONIZE();
    return SFEM_SUCCESS;
}

extern int cu_affine_sshex8_laplacian_apply(const int                        level,
                                            const ptrdiff_t                  nelements,
                                            const ptrdiff_t                  stride,  // Stride for elements and fff
                                            const ptrdiff_t                  interior_start,
                                            const idx_t *const SFEM_RESTRICT elements,
                                            const void *const SFEM_RESTRICT  fff,
                                            const enum RealType              real_type_xy,
                                            const void *const                x,
                                            void *const                      y,
                                            void                            *stream) {
    switch (real_type_xy) {
        case SFEM_REAL_DEFAULT: {
            return cu_affine_sshex8_laplacian_apply_tpl(
                    level, nelements, stride, interior_start, elements, (cu_jacobian_t *)fff, (real_t *)x, (real_t *)y, stream);
        }
        case SFEM_FLOAT32: {
            return cu_affine_sshex8_laplacian_apply_tpl(
                    level, nelements, stride, interior_start, elements, (cu_jacobian_t *)fff, (float *)x, (float *)y, stream);
        }
        case SFEM_FLOAT64: {
            return cu_affine_sshex8_laplacian_apply_tpl(
                    level, nelements, stride, interior_start, elements, (cu_jacobian_t *)fff, (double *)x, (double *)y, stream);
        }
        default: {
            fprintf(stderr,
                    "[Error] cu_tet4_laplacian_apply: not implemented for type %s (code %d)\n",
                    real_type_to_string(real_type_xy),
                    real_type_xy);
            assert(0);
            return SFEM_FAILURE;
        }
    }
}
