#include "hip/hip_runtime.h"
#include "cu_hex8_linear_elasticity.h"

#include "sfem_cuda_base.h"

#include "cu_hex8_linear_elasticity_inline.hpp"
#include "cu_hex8_linear_elasticity_matrix_inline.hpp"

#include <stdio.h>

template <typename T>
__global__ void cu_affine_hex8_linear_elasticity_apply_kernel(const ptrdiff_t nelements,
                                                              const ptrdiff_t stride,  // Stride for elements and fff
                                                              const idx_t *const SFEM_RESTRICT         elements,
                                                              const cu_jacobian_t *const SFEM_RESTRICT g_jacobian_adjugate,
                                                              const cu_jacobian_t *const SFEM_RESTRICT g_jacobian_determinant,
                                                              const T                                  mu,
                                                              const T                                  lambda,
                                                              const ptrdiff_t                          u_stride,
                                                              const T *const SFEM_RESTRICT             g_ux,
                                                              const T *const SFEM_RESTRICT             g_uy,
                                                              const T *const SFEM_RESTRICT             g_uz,
                                                              const ptrdiff_t                          out_stride,
                                                              T *const SFEM_RESTRICT                   g_outx,
                                                              T *const SFEM_RESTRICT                   g_outy,
                                                              T *const SFEM_RESTRICT                   g_outz) {
    static const int n_qp  = 2;
    static const T   qx[2] = {0.2113248654, 0.7886751346};
    static const T   qw[2] = {1. / 2, 1. / 2};

    // static const int n_qp = 3;
    // static const T qx[3] = {0.1127016654, 1. / 2, 0.8872983346};
    // static const T qw[3] = {0.2777777778, 0.4444444444, 0.2777777778};

    for (ptrdiff_t e = blockIdx.x * blockDim.x + threadIdx.x; e < nelements; e += blockDim.x * gridDim.x) {
        idx_t ev[8];

        // Sub-geometry
        T adjugate[9];

        T ux[8];
        T uy[8];
        T uz[8];

        T outx[8];
        T outy[8];
        T outz[8];

        for (int d = 0; d < 8; d++) {
            outx[d] = 0;
            outy[d] = 0;
            outz[d] = 0;
        }

        // Copy over jacobian adjugate
        {
            const cu_jacobian_t *const jacobian_adjugate = &g_jacobian_adjugate[e];
            for (int i = 0; i < 9; i++) {
                adjugate[i] = jacobian_adjugate[i * nelements];
            }
        }

        const T jacobian_determinant = g_jacobian_determinant[e];

        for (int v = 0; v < 8; ++v) {
            ev[v] = elements[v * nelements + e];
        }

        for (int v = 0; v < 8; ++v) {
            const ptrdiff_t idx = ev[v] * u_stride;
            ux[v]               = g_ux[idx];
            uy[v]               = g_uy[idx];
            uz[v]               = g_uz[idx];

            assert(ux[v] == ux[v]);
            assert(uy[v] == uy[v]);
            assert(uz[v] == uz[v]);
        }

        for (int kz = 0; kz < n_qp; kz++) {
            for (int ky = 0; ky < n_qp; ky++) {
                for (int kx = 0; kx < n_qp; kx++) {
                    cu_hex8_linear_elasticity_apply_adj<T, T>(mu,
                                                              lambda,
                                                              adjugate,
                                                              jacobian_determinant,
                                                              qx[kx],
                                                              qx[ky],
                                                              qx[kz],
                                                              qw[kx] * qw[ky] * qw[kz],
                                                              ux,
                                                              uy,
                                                              uz,
                                                              outx,
                                                              outy,
                                                              outz);
                }
            }
        }

        for (int v = 0; v < 8; v++) {
            const ptrdiff_t idx = ev[v] * out_stride;
            assert(outx[v] == outx[v]);
            assert(outy[v] == outy[v]);
            assert(outz[v] == outz[v]);

            atomicAdd(&g_outx[idx], outx[v]);
            atomicAdd(&g_outy[idx], outy[v]);
            atomicAdd(&g_outz[idx], outz[v]);
        }
    }
}

template <typename T>
int cu_affine_hex8_linear_elasticity_apply_tpl(const ptrdiff_t                          nelements,
                                               const ptrdiff_t                          stride,  // Stride for elements and fff
                                               const idx_t *const SFEM_RESTRICT         elements,
                                               const cu_jacobian_t *const SFEM_RESTRICT jacobian_adjugate,
                                               const cu_jacobian_t *const SFEM_RESTRICT jacobian_determinant,
                                               const real_t                             mu,
                                               const real_t                             lambda,
                                               const ptrdiff_t                          u_stride,
                                               const T *const SFEM_RESTRICT             ux,
                                               const T *const SFEM_RESTRICT             uy,
                                               const T *const SFEM_RESTRICT             uz,
                                               const ptrdiff_t                          out_stride,
                                               T *const SFEM_RESTRICT                   outx,
                                               T *const SFEM_RESTRICT                   outy,
                                               T *const SFEM_RESTRICT                   outz,
                                               void                                    *stream) {
    SFEM_DEBUG_SYNCHRONIZE();

    int block_size = 128;
#ifdef SFEM_USE_OCCUPANCY_MAX_POTENTIAL
    {
        int min_grid_size;
        hipOccupancyMaxPotentialBlockSize(&min_grid_size, &block_size, cu_affine_hex8_linear_elasticity_apply_kernel<T>, 0, 0);
    }
#endif  // SFEM_USE_OCCUPANCY_MAX_POTENTIAL

    ptrdiff_t n_blocks = MAX(ptrdiff_t(1), (nelements + block_size - 1) / block_size);

    if (stream) {
        hipStream_t s = *static_cast<hipStream_t *>(stream);
        cu_affine_hex8_linear_elasticity_apply_kernel<T><<<n_blocks, block_size, 0, s>>>(nelements,
                                                                                         stride,
                                                                                         elements,
                                                                                         jacobian_adjugate,
                                                                                         jacobian_determinant,
                                                                                         mu,
                                                                                         lambda,
                                                                                         u_stride,
                                                                                         ux,
                                                                                         uy,
                                                                                         uz,
                                                                                         out_stride,
                                                                                         outx,
                                                                                         outy,
                                                                                         outz);
    } else {
        cu_affine_hex8_linear_elasticity_apply_kernel<T><<<n_blocks, block_size, 0>>>(nelements,
                                                                                      stride,
                                                                                      elements,
                                                                                      jacobian_adjugate,
                                                                                      jacobian_determinant,
                                                                                      mu,
                                                                                      lambda,
                                                                                      u_stride,
                                                                                      ux,
                                                                                      uy,
                                                                                      uz,
                                                                                      out_stride,
                                                                                      outx,
                                                                                      outy,
                                                                                      outz);
    }

    SFEM_DEBUG_SYNCHRONIZE();
    return SFEM_SUCCESS;
}

extern int cu_affine_hex8_linear_elasticity_apply(const ptrdiff_t                  nelements,
                                                  const ptrdiff_t                  stride,  // Stride for elements and fff
                                                  const idx_t *const SFEM_RESTRICT elements,
                                                  const void *const SFEM_RESTRICT  jacobian_adjugate,
                                                  const void *const SFEM_RESTRICT  jacobian_determinant,
                                                  const real_t                     mu,
                                                  const real_t                     lambda,
                                                  const enum RealType              real_type,
                                                  const ptrdiff_t                  u_stride,
                                                  const void *const SFEM_RESTRICT  ux,
                                                  const void *const SFEM_RESTRICT  uy,
                                                  const void *const SFEM_RESTRICT  uz,
                                                  const ptrdiff_t                  out_stride,
                                                  void *const SFEM_RESTRICT        outx,
                                                  void *const SFEM_RESTRICT        outy,
                                                  void *const SFEM_RESTRICT        outz,
                                                  void                            *stream) {
    switch (real_type) {
        case SFEM_REAL_DEFAULT: {
            return cu_affine_hex8_linear_elasticity_apply_tpl(nelements,
                                                              stride,
                                                              elements,
                                                              (cu_jacobian_t *)jacobian_adjugate,
                                                              (cu_jacobian_t *)jacobian_determinant,
                                                              mu,
                                                              lambda,
                                                              u_stride,
                                                              (real_t *)ux,
                                                              (real_t *)uy,
                                                              (real_t *)uz,
                                                              out_stride,
                                                              (real_t *)outx,
                                                              (real_t *)outy,
                                                              (real_t *)outz,
                                                              stream);
        }
        case SFEM_FLOAT32: {
            return cu_affine_hex8_linear_elasticity_apply_tpl(nelements,
                                                              stride,
                                                              elements,
                                                              (cu_jacobian_t *)jacobian_adjugate,
                                                              (cu_jacobian_t *)jacobian_determinant,
                                                              mu,
                                                              lambda,
                                                              u_stride,
                                                              (float *)ux,
                                                              (float *)uy,
                                                              (float *)uz,
                                                              out_stride,
                                                              (float *)outx,
                                                              (float *)outy,
                                                              (float *)outz,
                                                              stream);
        }
        case SFEM_FLOAT64: {
            return cu_affine_hex8_linear_elasticity_apply_tpl(nelements,
                                                              stride,
                                                              elements,
                                                              (cu_jacobian_t *)jacobian_adjugate,
                                                              (cu_jacobian_t *)jacobian_determinant,
                                                              mu,
                                                              lambda,
                                                              u_stride,
                                                              (double *)ux,
                                                              (double *)uy,
                                                              (double *)uz,
                                                              out_stride,
                                                              (double *)outx,
                                                              (double *)outy,
                                                              (double *)outz,
                                                              stream);
        }
        default: {
            SFEM_ERROR(
                    "[Error] cu_affine_hex8_linear_elasticity_apply: not implemented for type %s "
                    "(code %d)\n",
                    real_type_to_string(real_type),
                    real_type);
            return SFEM_FAILURE;
        }
    }
}

template <typename T>
__global__ void cu_affine_hex8_linear_elasticity_bsr_kernel(const ptrdiff_t nelements,
                                                            const ptrdiff_t stride,  // Stride for elements and fff
                                                            const idx_t *const SFEM_RESTRICT         elements,
                                                            const cu_jacobian_t *const SFEM_RESTRICT g_jacobian_adjugate,
                                                            const cu_jacobian_t *const SFEM_RESTRICT g_jacobian_determinant,
                                                            const T                                  mu,
                                                            const T                                  lambda,
                                                            const count_t *const SFEM_RESTRICT       rowptr,
                                                            const idx_t *const SFEM_RESTRICT         colidx,
                                                            T *const SFEM_RESTRICT                   values) {
    static const int n_qp  = 2;
    static const T   qx[2] = {0.2113248654, 0.7886751346};
    static const T   qw[2] = {1. / 2, 1. / 2};

    // static const int n_qp = 3;
    // static const T qx[3] = {0.1127016654, 1. / 2, 0.8872983346};
    // static const T qw[3] = {0.2777777778, 0.4444444444, 0.2777777778};

    for (ptrdiff_t e = blockIdx.x * blockDim.x + threadIdx.x; e < nelements; e += blockDim.x * gridDim.x) {
        idx_t ev[8];

        // Sub-geometry
        T adjugate[9];

        // Copy over jacobian adjugate
        {
            const cu_jacobian_t *const jacobian_adjugate = &g_jacobian_adjugate[e];
            for (int i = 0; i < 9; i++) {
                adjugate[i] = jacobian_adjugate[i * nelements];
            }
        }

        const T jacobian_determinant = g_jacobian_determinant[e];

        for (int v = 0; v < 8; ++v) {
            ev[v] = elements[v * nelements + e];
        }

        T block[9];
        for (int d = 0; d < 9; d++) {
            block[d] = 0;
        }

        for (int i = 0; i < 8; i++) {
            const int          lenrow   = rowptr[ev[i] + 1] - rowptr[ev[i]];
            const idx_t *const row      = &colidx[rowptr[ev[i]]];
            T *const           g_blocks = &values[rowptr[ev[i]] * 9];

            int ks[8];
            cu_hex8_find_cols(ev, row, lenrow, ks);

            for (int j = 0; j < 8; j++) {
                T *const g_block = &g_blocks[ks[j] * 9];

                for (int kz = 0; kz < n_qp; kz++) {
                    for (int ky = 0; ky < n_qp; ky++) {
                        for (int kx = 0; kx < n_qp; kx++) {
                            T trial_grad[3], test_grad[3];
                            cu_hex8_ref_shape_grad(i, qx[kx], qx[ky], qx[kz], test_grad);
                            cu_hex8_ref_shape_grad(j, qx[kx], qx[ky], qx[kz], trial_grad);
                            cu_linear_elasticity_matrix_block(mu,
                                                              lambda,
                                                              adjugate,
                                                              jacobian_determinant,
                                                              qw[kx] * qw[ky] * qw[kz],
                                                              trial_grad,
                                                              test_grad,
                                                              block);
                        }
                    }
                }

                for (int d = 0; d < 9; d++) {
                    atomicAdd(&g_block[d], block[d]);
                }
            }
        }
    }
}

template <typename T>
int cu_affine_hex8_linear_elasticity_bsr_tpl(const ptrdiff_t                          nelements,
                                             const ptrdiff_t                          stride,
                                             const idx_t *const SFEM_RESTRICT         elements,
                                             const cu_jacobian_t *const SFEM_RESTRICT jacobian_adjugate,
                                             const cu_jacobian_t *const SFEM_RESTRICT jacobian_determinant,
                                             const real_t                             mu,
                                             const real_t                             lambda,
                                             const count_t *const SFEM_RESTRICT       rowptr,
                                             const idx_t *const SFEM_RESTRICT         colidx,
                                             T *const SFEM_RESTRICT                   values,
                                             void                                    *stream) {
    SFEM_DEBUG_SYNCHRONIZE();

    int block_size = 128;
#ifdef SFEM_USE_OCCUPANCY_MAX_POTENTIAL
    {
        int min_grid_size;
        hipOccupancyMaxPotentialBlockSize(&min_grid_size, &block_size, cu_affine_hex8_linear_elasticity_bsr_kernel<T>, 0, 0);
    }
#endif  // SFEM_USE_OCCUPANCY_MAX_POTENTIAL

    ptrdiff_t n_blocks = MAX(ptrdiff_t(1), (nelements + block_size - 1) / block_size);

    if (stream) {
        hipStream_t s = *static_cast<hipStream_t *>(stream);
        cu_affine_hex8_linear_elasticity_bsr_kernel<T><<<n_blocks, block_size, 0, s>>>(
                nelements, stride, elements, jacobian_adjugate, jacobian_determinant, mu, lambda, rowptr, colidx, values);
    } else {
        cu_affine_hex8_linear_elasticity_bsr_kernel<T><<<n_blocks, block_size, 0>>>(
                nelements, stride, elements, jacobian_adjugate, jacobian_determinant, mu, lambda, rowptr, colidx, values);
    }

    SFEM_DEBUG_SYNCHRONIZE();
    return SFEM_SUCCESS;
}

extern int cu_affine_hex8_linear_elasticity_bsr(const ptrdiff_t                    nelements,
                                                const ptrdiff_t                    stride,
                                                const idx_t *const SFEM_RESTRICT   elements,
                                                const void *const SFEM_RESTRICT    jacobian_adjugate,
                                                const void *const SFEM_RESTRICT    jacobian_determinant,
                                                const real_t                       mu,
                                                const real_t                       lambda,
                                                const enum RealType                real_type,
                                                const count_t *const SFEM_RESTRICT rowptr,
                                                const idx_t *const SFEM_RESTRICT   colidx,
                                                void *const SFEM_RESTRICT          values,
                                                void                              *stream) {
    switch (real_type) {
        case SFEM_REAL_DEFAULT: {
            return cu_affine_hex8_linear_elasticity_bsr_tpl(nelements,
                                                            stride,
                                                            elements,
                                                            (cu_jacobian_t *)jacobian_adjugate,
                                                            (cu_jacobian_t *)jacobian_determinant,
                                                            mu,
                                                            lambda,
                                                            rowptr,
                                                            colidx,
                                                            (real_t *)values,
                                                            stream);
        }
        case SFEM_FLOAT32: {
            return cu_affine_hex8_linear_elasticity_bsr_tpl(nelements,
                                                            stride,
                                                            elements,
                                                            (cu_jacobian_t *)jacobian_adjugate,
                                                            (cu_jacobian_t *)jacobian_determinant,
                                                            mu,
                                                            lambda,
                                                            rowptr,
                                                            colidx,
                                                            (float *)values,
                                                            stream);
        }
        case SFEM_FLOAT64: {
            return cu_affine_hex8_linear_elasticity_bsr_tpl(nelements,
                                                            stride,
                                                            elements,
                                                            (cu_jacobian_t *)jacobian_adjugate,
                                                            (cu_jacobian_t *)jacobian_determinant,
                                                            mu,
                                                            lambda,
                                                            rowptr,
                                                            colidx,
                                                            (double *)values,
                                                            stream);
        }
        default: {
            SFEM_ERROR(
                    "[Error] cu_affine_hex8_linear_elasticity_bsr_tpl: not implemented for "
                    "type %s "
                    "(code %d)\n",
                    real_type_to_string(real_type),
                    real_type);
            return SFEM_FAILURE;
        }
    }
}

template <typename T>
__global__ void cu_affine_hex8_linear_elasticity_block_diag_sym_kernel(const ptrdiff_t                 nelements,
                                                                       const ptrdiff_t                 stride,
                                                                       idx_t *const SFEM_RESTRICT      elements,
                                                                       const cu_jacobian_t *const SFEM_RESTRICT jacobian_adjugate,
                                                                       const cu_jacobian_t *const SFEM_RESTRICT jacobian_determinant,
                                                                       const T                         mu,
                                                                       const T                         lambda,
                                                                       const ptrdiff_t                 out_stride,
                                                                       T *const                        out0,
                                                                       T *const                        out1,
                                                                       T *const                        out2,
                                                                       T *const                        out3,
                                                                       T *const                        out4,
                                                                       T *const                        out5) {
    static const int n_qp  = 2;
    static const T   qx[2] = {0.2113248654, 0.7886751346};
    static const T   qw[2] = {1. / 2, 1. / 2};

    for (ptrdiff_t e = blockIdx.x * blockDim.x + threadIdx.x; e < nelements; e += blockDim.x * gridDim.x) {
        idx_t ev[8];
        for (int v = 0; v < 8; ++v) {
            ev[v] = elements[v * stride + e];
        }

        T adjugate[9];

        // Copy over jacobian adjugate
        {
            for (int i = 0; i < 9; i++) {
                adjugate[i] = jacobian_adjugate[i * stride + e];
            }
        }

        const T determinant = jacobian_determinant[e];

        // Assemble the diagonal part of the matrix
        for (int edof_i = 0; edof_i < 8; edof_i++) {
            T element_matrix[6] = {0, 0, 0, 0, 0, 0};
            for (int zi = 0; zi < n_qp; zi++) {
                for (int yi = 0; yi < n_qp; yi++) {
                    for (int xi = 0; xi < n_qp; xi++) {
                        T test_grad[3];
                        cu_hex8_ref_shape_grad(edof_i, qx[xi], qx[yi], qx[zi], test_grad);
                        cu_linear_elasticity_matrix_sym<T>(mu,
                                                           lambda,
                                                           adjugate,
                                                           determinant,
                                                           test_grad,
                                                           test_grad,
                                                           qw[xi] * qw[yi] * qw[zi],
                                                           element_matrix);
                    }
                }
            }

            // local to global
            const ptrdiff_t idx = ev[edof_i] * out_stride;
            atomicAdd(&out0[idx], element_matrix[0]);
            atomicAdd(&out1[idx], element_matrix[1]);
            atomicAdd(&out2[idx], element_matrix[2]);
            atomicAdd(&out3[idx], element_matrix[3]);
            atomicAdd(&out4[idx], element_matrix[4]);
            atomicAdd(&out5[idx], element_matrix[5]);
        }
    }
}

template <typename T>
int cu_affine_hex8_linear_elasticity_block_diag_sym_tpl(const ptrdiff_t                 nelements,
                                                        const ptrdiff_t                 stride,
                                                        idx_t *const SFEM_RESTRICT      elements,
                                                        const cu_jacobian_t *const SFEM_RESTRICT jacobian_adjugate,
                                                        const cu_jacobian_t *const SFEM_RESTRICT jacobian_determinant,
                                                        const real_t                    mu,
                                                        const real_t                    lambda,
                                                        const ptrdiff_t                 out_stride,
                                                        T *const                        out0,
                                                        T *const                        out1,
                                                        T *const                        out2,
                                                        T *const                        out3,
                                                        T *const                        out4,
                                                        T *const                        out5,
                                                        void                           *stream) {
    SFEM_DEBUG_SYNCHRONIZE();

    int block_size = 128;
#ifdef SFEM_USE_OCCUPANCY_MAX_POTENTIAL
    {
        int min_grid_size;
        hipOccupancyMaxPotentialBlockSize(
                &min_grid_size, &block_size, cu_affine_hex8_linear_elasticity_block_diag_sym_kernel<T>, 0, 0);
    }
#endif  // SFEM_USE_OCCUPANCY_MAX_POTENTIAL

    ptrdiff_t n_blocks = MAX(ptrdiff_t(1), (nelements + block_size - 1) / block_size);

    if (stream) {
        hipStream_t s = *static_cast<hipStream_t *>(stream);
        cu_affine_hex8_linear_elasticity_block_diag_sym_kernel<T><<<n_blocks, block_size, 0, s>>>(nelements,
                                                                                                  stride,
                                                                                                  elements,
                                                                                                  jacobian_adjugate,
                                                                                                  jacobian_determinant,
                                                                                                  mu,
                                                                                                  lambda,
                                                                                                  out_stride,
                                                                                                  out0,
                                                                                                  out1,
                                                                                                  out2,
                                                                                                  out3,
                                                                                                  out4,
                                                                                                  out5);
    } else {
        cu_affine_hex8_linear_elasticity_block_diag_sym_kernel<T><<<n_blocks, block_size, 0>>>(nelements,
                                                                                               stride,
                                                                                               elements,
                                                                                               jacobian_adjugate,
                                                                                               jacobian_determinant,
                                                                                               mu,
                                                                                               lambda,
                                                                                               out_stride,
                                                                                               out0,
                                                                                               out1,
                                                                                               out2,
                                                                                               out3,
                                                                                               out4,
                                                                                               out5);
    }

    SFEM_DEBUG_SYNCHRONIZE();
    return SFEM_SUCCESS;
}

extern int cu_affine_hex8_linear_elasticity_block_diag_sym(const ptrdiff_t                 nelements,
                                                           const ptrdiff_t                 stride,
                                                           idx_t *const SFEM_RESTRICT      elements,
                                                           const void *const SFEM_RESTRICT jacobian_adjugate,
                                                           const void *const SFEM_RESTRICT jacobian_determinant,
                                                           const real_t                    mu,
                                                           const real_t                    lambda,
                                                           const ptrdiff_t                 out_stride,
                                                           const enum RealType             real_type,
                                                           void *const                     out0,
                                                           void *const                     out1,
                                                           void *const                     out2,
                                                           void *const                     out3,
                                                           void *const                     out4,
                                                           void *const                     out5,
                                                           void                           *stream) {
    switch (real_type) {
        case SFEM_REAL_DEFAULT: {
            return cu_affine_hex8_linear_elasticity_block_diag_sym_tpl(nelements,
                                                                       stride,
                                                                       elements,
                                                                       (cu_jacobian_t *)jacobian_adjugate,
                                                                       (cu_jacobian_t *)jacobian_determinant,
                                                                       mu,
                                                                       lambda,
                                                                       out_stride,
                                                                       (real_t *)out0,
                                                                       (real_t *)out1,
                                                                       (real_t *)out2,
                                                                       (real_t *)out3,
                                                                       (real_t *)out4,
                                                                       (real_t *)out5,
                                                                       stream);
        }
        case SFEM_FLOAT32: {
            return cu_affine_hex8_linear_elasticity_block_diag_sym_tpl(nelements,
                                                                       stride,
                                                                       elements,
                                                                       (cu_jacobian_t *)jacobian_adjugate,
                                                                       (cu_jacobian_t *)jacobian_determinant,
                                                                       mu,
                                                                       lambda,
                                                                       out_stride,
                                                                       (float *)out0,
                                                                       (float *)out1,
                                                                       (float *)out2,
                                                                       (float *)out3,
                                                                       (float *)out4,
                                                                       (float *)out5,
                                                                       stream);
        }
        case SFEM_FLOAT64: {
            return cu_affine_hex8_linear_elasticity_block_diag_sym_tpl(nelements,
                                                                       stride,
                                                                       elements,
                                                                       (cu_jacobian_t *)jacobian_adjugate,
                                                                       (cu_jacobian_t *)jacobian_determinant,
                                                                       mu,
                                                                       lambda,
                                                                       out_stride,
                                                                       (double *)out0,
                                                                       (double *)out1,
                                                                       (double *)out2,
                                                                       (double *)out3,
                                                                       (double *)out4,
                                                                       (double *)out5,
                                                                       stream);
        }
        default: {
            SFEM_ERROR(
                    "[Error] cu_affine_hex8_linear_elasticity_block_diag_sym: not implemented for "
                    "type %s "
                    "(code %d)\n",
                    real_type_to_string(real_type),
                    real_type);
            return SFEM_FAILURE;
        }
    }
}
