#include "hip/hip_runtime.h"
#include "cu_hex8_linear_elasticity.h"

#include "sfem_cuda_base.h"

#include "cu_hex8_linear_elasticity_inline.hpp"
#include "cu_hex8_linear_elasticity_matrix_inline.hpp"

#include <stdio.h>

template <typename T>
__global__ void cu_affine_hex8_linear_elasticity_apply_kernel(
        const ptrdiff_t nelements,
        const ptrdiff_t stride,  // Stride for elements and fff
        const idx_t *const SFEM_RESTRICT elements,
        const cu_jacobian_t *const SFEM_RESTRICT g_jacobian_adjugate,
        const cu_jacobian_t *const SFEM_RESTRICT g_jacobian_determinant,
        const T mu,
        const T lambda,
        const ptrdiff_t u_stride,
        const T *const SFEM_RESTRICT g_ux,
        const T *const SFEM_RESTRICT g_uy,
        const T *const SFEM_RESTRICT g_uz,
        const ptrdiff_t out_stride,
        T *const SFEM_RESTRICT g_outx,
        T *const SFEM_RESTRICT g_outy,
        T *const SFEM_RESTRICT g_outz) {
    static const int n_qp = 2;
    static const T qx[2] = {0.2113248654, 0.7886751346};
    static const T qw[2] = {1. / 2, 1. / 2};

    // static const int n_qp = 3;
    // static const T qx[3] = {0.1127016654, 1. / 2, 0.8872983346};
    // static const T qw[3] = {0.2777777778, 0.4444444444, 0.2777777778};

    for (ptrdiff_t e = blockIdx.x * blockDim.x + threadIdx.x; e < nelements;
         e += blockDim.x * gridDim.x) {
        idx_t ev[8];

        // Sub-geometry
        T adjugate[9];

        T ux[8];
        T uy[8];
        T uz[8];

        T outx[8];
        T outy[8];
        T outz[8];

        for (int d = 0; d < 8; d++) {
            outx[d] = 0;
            outy[d] = 0;
            outz[d] = 0;
        }

        // Copy over jacobian adjugate
        {
            const cu_jacobian_t *const jacobian_adjugate = &g_jacobian_adjugate[e];
            for (int i = 0; i < 9; i++) {
                adjugate[i] = jacobian_adjugate[i * nelements];
            }
        }

        const T jacobian_determinant = g_jacobian_determinant[e];

        for (int v = 0; v < 8; ++v) {
            ev[v] = elements[v * nelements + e];
        }

        for (int v = 0; v < 8; ++v) {
            const ptrdiff_t idx = ev[v] * u_stride;
            ux[v] = g_ux[idx];
            uy[v] = g_uy[idx];
            uz[v] = g_uz[idx];

            assert(ux[v] == ux[v]);
            assert(uy[v] == uy[v]);
            assert(uz[v] == uz[v]);
        }

        for (int kz = 0; kz < n_qp; kz++) {
            for (int ky = 0; ky < n_qp; ky++) {
                for (int kx = 0; kx < n_qp; kx++) {
                    cu_hex8_linear_elasticity_apply_adj<T, T>(mu,
                                                              lambda,
                                                              adjugate,
                                                              jacobian_determinant,
                                                              qx[kx],
                                                              qx[ky],
                                                              qx[kz],
                                                              qw[kx] * qw[ky] * qw[kz],
                                                              ux,
                                                              uy,
                                                              uz,
                                                              outx,
                                                              outy,
                                                              outz);
                }
            }
        }

        for (int v = 0; v < 8; v++) {
            const ptrdiff_t idx = ev[v] * out_stride;
            assert(outx[v] == outx[v]);
            assert(outy[v] == outy[v]);
            assert(outz[v] == outz[v]);

            atomicAdd(&g_outx[idx], outx[v]);
            atomicAdd(&g_outy[idx], outy[v]);
            atomicAdd(&g_outz[idx], outz[v]);
        }
    }
}

template <typename T>
int cu_affine_hex8_linear_elasticity_apply_tpl(
        const ptrdiff_t nelements,
        const ptrdiff_t stride,  // Stride for elements and fff
        const idx_t *const SFEM_RESTRICT elements,
        const cu_jacobian_t *const SFEM_RESTRICT jacobian_adjugate,
        const cu_jacobian_t *const SFEM_RESTRICT jacobian_determinant,
        const real_t mu,
        const real_t lambda,
        const ptrdiff_t u_stride,
        const T *const SFEM_RESTRICT ux,
        const T *const SFEM_RESTRICT uy,
        const T *const SFEM_RESTRICT uz,
        const ptrdiff_t out_stride,
        T *const SFEM_RESTRICT outx,
        T *const SFEM_RESTRICT outy,
        T *const SFEM_RESTRICT outz,
        void *stream) {
    int block_size = 128;
#ifdef SFEM_USE_OCCUPANCY_MAX_POTENTIAL
    {
        int min_grid_size;
        hipOccupancyMaxPotentialBlockSize(&min_grid_size,
                                           &block_size,
                                           cu_affine_hex8_linear_elasticity_apply_kernel<T>,
                                           0,
                                           0);
    }
#endif  // SFEM_USE_OCCUPANCY_MAX_POTENTIAL

    ptrdiff_t n_blocks = MAX(ptrdiff_t(1), (nelements + block_size - 1) / block_size);

    if (stream) {
        hipStream_t s = *static_cast<hipStream_t *>(stream);
        cu_affine_hex8_linear_elasticity_apply_kernel<T>
                <<<n_blocks, block_size, 0, s>>>(nelements,
                                                 stride,
                                                 elements,
                                                 jacobian_adjugate,
                                                 jacobian_determinant,
                                                 mu,
                                                 lambda,
                                                 u_stride,
                                                 ux,
                                                 uy,
                                                 uz,
                                                 out_stride,
                                                 outx,
                                                 outy,
                                                 outz);
    } else {
        cu_affine_hex8_linear_elasticity_apply_kernel<T>
                <<<n_blocks, block_size, 0>>>(nelements,
                                              stride,
                                              elements,
                                              jacobian_adjugate,
                                              jacobian_determinant,
                                              mu,
                                              lambda,
                                              u_stride,
                                              ux,
                                              uy,
                                              uz,
                                              out_stride,
                                              outx,
                                              outy,
                                              outz);
    }

    SFEM_DEBUG_SYNCHRONIZE();
    return SFEM_SUCCESS;
}

extern int cu_affine_hex8_linear_elasticity_apply(
        const ptrdiff_t nelements,
        const ptrdiff_t stride,  // Stride for elements and fff
        const idx_t *const SFEM_RESTRICT elements,
        const void *const SFEM_RESTRICT jacobian_adjugate,
        const void *const SFEM_RESTRICT jacobian_determinant,
        const real_t mu,
        const real_t lambda,
        const enum RealType real_type,
        const ptrdiff_t u_stride,
        const void *const SFEM_RESTRICT ux,
        const void *const SFEM_RESTRICT uy,
        const void *const SFEM_RESTRICT uz,
        const ptrdiff_t out_stride,
        void *const SFEM_RESTRICT outx,
        void *const SFEM_RESTRICT outy,
        void *const SFEM_RESTRICT outz,
        void *stream) {
    switch (real_type) {
        case SFEM_REAL_DEFAULT: {
            return cu_affine_hex8_linear_elasticity_apply_tpl(nelements,
                                                              stride,
                                                              elements,
                                                              (cu_jacobian_t *)jacobian_adjugate,
                                                              (cu_jacobian_t *)jacobian_determinant,
                                                              mu,
                                                              lambda,
                                                              u_stride,
                                                              (real_t *)ux,
                                                              (real_t *)uy,
                                                              (real_t *)uz,
                                                              out_stride,
                                                              (real_t *)outx,
                                                              (real_t *)outy,
                                                              (real_t *)outz,
                                                              stream);
        }
        case SFEM_FLOAT32: {
            return cu_affine_hex8_linear_elasticity_apply_tpl(nelements,
                                                              stride,
                                                              elements,
                                                              (cu_jacobian_t *)jacobian_adjugate,
                                                              (cu_jacobian_t *)jacobian_determinant,
                                                              mu,
                                                              lambda,
                                                              u_stride,
                                                              (float *)ux,
                                                              (float *)uy,
                                                              (float *)uz,
                                                              out_stride,
                                                              (float *)outx,
                                                              (float *)outy,
                                                              (float *)outz,
                                                              stream);
        }
        case SFEM_FLOAT64: {
            return cu_affine_hex8_linear_elasticity_apply_tpl(nelements,
                                                              stride,
                                                              elements,
                                                              (cu_jacobian_t *)jacobian_adjugate,
                                                              (cu_jacobian_t *)jacobian_determinant,
                                                              mu,
                                                              lambda,
                                                              u_stride,
                                                              (double *)ux,
                                                              (double *)uy,
                                                              (double *)uz,
                                                              out_stride,
                                                              (double *)outx,
                                                              (double *)outy,
                                                              (double *)outz,
                                                              stream);
        }
        default: {
            fprintf(stderr,
                    "[Error] cu_affine_hex8_linear_elasticity_apply: not implemented for type %s "
                    "(code %d)\n",
                    real_type_to_string(real_type),
                    real_type);
            assert(0);
            return SFEM_FAILURE;
        }
    }
}

template <typename T>
__global__ void cu_affine_hex8_linear_elasticity_bsr_kernel(
        const ptrdiff_t nelements,
        const ptrdiff_t stride,  // Stride for elements and fff
        const idx_t *const SFEM_RESTRICT elements,
        const cu_jacobian_t *const SFEM_RESTRICT g_jacobian_adjugate,
        const cu_jacobian_t *const SFEM_RESTRICT g_jacobian_determinant,
        const T mu,
        const T lambda,
        const count_t *const SFEM_RESTRICT rowptr,
        const idx_t *const SFEM_RESTRICT colidx,
        T *const SFEM_RESTRICT values) {
    static const int n_qp = 2;
    static const T qx[2] = {0.2113248654, 0.7886751346};
    static const T qw[2] = {1. / 2, 1. / 2};


    // static const int n_qp = 3;
    // static const T qx[3] = {0.1127016654, 1. / 2, 0.8872983346};
    // static const T qw[3] = {0.2777777778, 0.4444444444, 0.2777777778};

    for (ptrdiff_t e = blockIdx.x * blockDim.x + threadIdx.x; e < nelements;
         e += blockDim.x * gridDim.x) {
        idx_t ev[8];

        // Sub-geometry
        T adjugate[9];

        // Copy over jacobian adjugate
        {
            const cu_jacobian_t *const jacobian_adjugate = &g_jacobian_adjugate[e];
            for (int i = 0; i < 9; i++) {
                adjugate[i] = jacobian_adjugate[i * nelements];
            }
        }

        const T jacobian_determinant = g_jacobian_determinant[e];

        for (int v = 0; v < 8; ++v) {
            ev[v] = elements[v * nelements + e];
        }

        T block[9];
        for (int d = 0; d < 9; d++) {
            block[d] = 0;
        }

        for (int i = 0; i < 8; i++) {
            const int lenrow = rowptr[ev[i] + 1] - rowptr[ev[i]];
            const idx_t *const row = &colidx[rowptr[ev[i]]];
            T *const g_blocks = &values[rowptr[ev[i]] * 9];

            int ks[8];
            cu_hex8_find_cols(ev, row, lenrow, ks);

            for (int j = 0; j < 8; j++) {
                T *const g_block = &g_blocks[ks[j] * 9];

                for (int kz = 0; kz < n_qp; kz++) {
                    for (int ky = 0; ky < n_qp; ky++) {
                        for (int kx = 0; kx < n_qp; kx++) {
                            T trial_grad[3], test_grad[3];
                            cu_hex8_ref_shape_grad(i, qx[kx], qx[ky], qx[kz], test_grad);
                            cu_hex8_ref_shape_grad(j, qx[kx], qx[ky], qx[kz], trial_grad);
                            cu_linear_elasticity_matrix_block(mu,
                                                              lambda,
                                                              adjugate,
                                                              jacobian_determinant,
                                                              qw[kx] * qw[ky] * qw[kz],
                                                              trial_grad,
                                                              test_grad,
                                                              block);
                        }
                    }
                }

                for (int d = 0; d < 9; d++) {
                    atomicAdd(&g_block[d], block[d]);
                }
            }
        }
    }
}

template <typename T>
int cu_affine_hex8_linear_elasticity_bsr_tpl(
        const ptrdiff_t nelements,
        const ptrdiff_t stride,
        const idx_t *const SFEM_RESTRICT elements,
        const cu_jacobian_t *const SFEM_RESTRICT jacobian_adjugate,
        const cu_jacobian_t *const SFEM_RESTRICT jacobian_determinant,
        const real_t mu,
        const real_t lambda,
        const count_t *const SFEM_RESTRICT rowptr,
        const idx_t *const SFEM_RESTRICT colidx,
        T *const SFEM_RESTRICT values,
        void *stream) {
    int block_size = 128;
#ifdef SFEM_USE_OCCUPANCY_MAX_POTENTIAL
    {
        int min_grid_size;
        hipOccupancyMaxPotentialBlockSize(
                &min_grid_size, &block_size, cu_affine_hex8_linear_elasticity_bsr_kernel<T>, 0, 0);
    }
#endif  // SFEM_USE_OCCUPANCY_MAX_POTENTIAL

    ptrdiff_t n_blocks = MAX(ptrdiff_t(1), (nelements + block_size - 1) / block_size);

    if (stream) {
        hipStream_t s = *static_cast<hipStream_t *>(stream);
        cu_affine_hex8_linear_elasticity_bsr_kernel<T>
                <<<n_blocks, block_size, 0, s>>>(nelements,
                                                 stride,
                                                 elements,
                                                 jacobian_adjugate,
                                                 jacobian_determinant,
                                                 mu,
                                                 lambda,
                                                 rowptr,
                                                 colidx,
                                                 values);
    } else {
        cu_affine_hex8_linear_elasticity_bsr_kernel<T>
                <<<n_blocks, block_size, 0>>>(nelements,
                                              stride,
                                              elements,
                                              jacobian_adjugate,
                                              jacobian_determinant,
                                              mu,
                                              lambda,
                                              rowptr,
                                              colidx,
                                              values);
    }

    SFEM_DEBUG_SYNCHRONIZE();
    return SFEM_SUCCESS;
}

extern int cu_affine_hex8_linear_elasticity_bsr(const ptrdiff_t nelements,
                                                const ptrdiff_t stride,
                                                const idx_t *const SFEM_RESTRICT elements,
                                                const void *const SFEM_RESTRICT jacobian_adjugate,
                                                const void *const SFEM_RESTRICT
                                                        jacobian_determinant,
                                                const real_t mu,
                                                const real_t lambda,
                                                const enum RealType real_type,
                                                const count_t *const SFEM_RESTRICT rowptr,
                                                const idx_t *const SFEM_RESTRICT colidx,
                                                void *const SFEM_RESTRICT values,
                                                void *stream) {
    switch (real_type) {
        case SFEM_REAL_DEFAULT: {
            return cu_affine_hex8_linear_elasticity_bsr_tpl(nelements,
                                                            stride,
                                                            elements,
                                                            (cu_jacobian_t *)jacobian_adjugate,
                                                            (cu_jacobian_t *)jacobian_determinant,
                                                            mu,
                                                            lambda,
                                                            rowptr,
                                                            colidx,
                                                            (real_t *)values,
                                                            stream);
        }
        case SFEM_FLOAT32: {
            return cu_affine_hex8_linear_elasticity_bsr_tpl(nelements,
                                                            stride,
                                                            elements,
                                                            (cu_jacobian_t *)jacobian_adjugate,
                                                            (cu_jacobian_t *)jacobian_determinant,
                                                            mu,
                                                            lambda,
                                                            rowptr,
                                                            colidx,
                                                            (float *)values,
                                                            stream);
        }
        case SFEM_FLOAT64: {
            return cu_affine_hex8_linear_elasticity_bsr_tpl(nelements,
                                                            stride,
                                                            elements,
                                                            (cu_jacobian_t *)jacobian_adjugate,
                                                            (cu_jacobian_t *)jacobian_determinant,
                                                            mu,
                                                            lambda,
                                                            rowptr,
                                                            colidx,
                                                            (double *)values,
                                                            stream);
        }
        default: {
            fprintf(stderr,
                    "[Error] cu_affine_hex8_linear_elasticity_bsr_tpl: not implemented for "
                    "type %s "
                    "(code %d)\n",
                    real_type_to_string(real_type),
                    real_type);
            assert(0);
            return SFEM_FAILURE;
        }
    }
}
